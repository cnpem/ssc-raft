#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define PI 3.141592653589793238462643383279502884

#define TPBX 16
#define TPBY 16
#define TPBZ 4
#define TPBE 256

extern "C" {
  __global__ void radon_kernel(float *output, float *input,
			       int sizeImage, int nrays, int nangles,
			       int blockSize, float a)
  {
    int tx = threadIdx.x + blockIdx.x*blockDim.x; 
    int ty = threadIdx.y + blockIdx.y*blockDim.y; 
    int tz = threadIdx.z + blockIdx.z*blockDim.z;
  
    if ( (tx<nrays) && (ty < nangles) && (tz<blockSize)  ){
 
      int k, X, Y;

      float s, x, y, linesum, ctheta, stheta, theta, t;  
      float dt = 2.0*a/(nrays-1);
      float dtheta = PI/(nangles-1);
 
      theta = ty * dtheta;
      ctheta =cosf(theta);
      stheta =sinf(theta);
      
      t = - a + tx * dt; 
      
      linesum = 0;
      for( k = 0; k < nrays; k++ ) {
	s = - a + k * dt;
	x = t * ctheta - s * stheta;
	y = t * stheta + s * ctheta;
	X = (int) ((x + 1)/dt);
	Y = (int) ((y + 1)/dt);	 
	if ((X >= 0) & (X<sizeImage) & (Y>=0) & (Y<sizeImage) )
	  linesum += input[ tz * sizeImage * sizeImage + Y * sizeImage + X ];
      }
      //output[tz * nrays * nangles + tx * nangles + ty] = linesum * dt;	
      output[tz * nrays * nangles + ty * nrays + tx] = linesum * dt;	
    }
  }
}


extern "C" {
  __global__ void radon_local_kernel(float* output, float *input,
				     int sizeImage, int nrays, int nangles,
				     int blockSize, int centerx, int centery)
  {
    int tx = threadIdx.x + blockIdx.x*blockDim.x; 
    int ty = threadIdx.y + blockIdx.y*blockDim.y; 
    int tz = threadIdx.z + blockIdx.z*blockDim.z;
  
    if ( (tx<nrays) && (ty < nangles) && (tz<blockSize)  ){
      
      int k, X, Y;

      float a, s, x, y, linesum, ctheta, stheta, theta, t;
      float dxy = 2.0/(sizeImage-1); //, ds = 2.0*sqrtf(2.0)/(nrays-1);
      float dtheta = PI/(nangles-1);
      float xc, yc, T, S;

      a = ( nrays/2.0 ) * dxy;
    
      theta = ty * dtheta;
      ctheta =cosf(theta);
      stheta =sinf(theta);
      
      t = - a + tx * dxy;
      
      xc = -1 + centerx * dxy;
      yc = -1 + centery * dxy;
      T =  xc * ctheta + yc * stheta;
      S = -xc * stheta + yc * ctheta;
      
      linesum = 0;
      for( k = 0; k < sizeImage; k++ ) {
	s = - 1 + k * dxy;
	x = (t-T) * ctheta - (s-S) * stheta;
	y = (t-T) * stheta + (s-S) * ctheta;
	X = (int) ((x + 1)/dxy);
	Y = (int) ((y + 1)/dxy);
	if ((X >= 0) & (X<sizeImage) & (Y>=0) & (Y<sizeImage) )
	  linesum += input[ tz * sizeImage * sizeImage + Y * sizeImage + X ];
      }

      output[tz * nrays * nangles + ty * nrays + tx] = linesum * dxy;
    }
  }
}
                              


extern "C" {
  void radonp_gpu(float* h_output, float* h_input, int sizeImage, int nrays, int nangles, int device, int blocksize, float a)
  {
    hipSetDevice(device);
    
    float *d_output, *d_input;
    
    // Allocate GPU buffers for the output sinogram
    hipMalloc(&d_output, sizeof(float) * nrays * nangles * blocksize);
    hipMalloc(&d_input, sizeof(float) * sizeImage * sizeImage * blocksize);
    hipMemcpy(d_input, h_input, sizeof(float) * sizeImage * sizeImage * blocksize, hipMemcpyHostToDevice);	
    
    //

    dim3 threadsPerBlock(TPBX,TPBY,TPBZ);
    dim3 gridBlock((int)ceil((nrays)/threadsPerBlock.x)+1,
		   (int)ceil((nangles)/threadsPerBlock.y)+1,
		   (int)ceil(blocksize/threadsPerBlock.z)+1);
        
    radon_kernel<<<gridBlock, threadsPerBlock>>>(d_output, d_input, sizeImage, nrays, nangles, blocksize, a);
    
    hipDeviceSynchronize();
    
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(h_output, d_output, sizeof(float) * nrays * nangles * blocksize, hipMemcpyDeviceToHost);
    
    //cudaFree(d_input);
    hipFree(d_output);
    hipFree(d_input);
    hipDeviceReset();

    return;
  }
}

extern "C" {
  float radonp_ray(float *h_output, float* h_input,
		   int sizeImage, int nrays, int nangles,
		   float a, int i, int j)
  {
    int k, X, Y;
    float s, x, y, linesum, ctheta, stheta, theta, t;  
    float dt = 2.0*a/(nrays-1), ds = 2.0*sqrtf(2.0)/(nrays-1);
    float dtheta = PI/(nangles-1);
    float output;
    
    theta = i*dtheta;
    ctheta =cosf(theta);
    stheta =sinf(theta);
    
    t = - a + j * dt; 
    
    linesum = 0;
    for( k = 0; k < nrays; k++ )
      {
	s = -sqrtf(2.0) + k * ds; //- a + k * dt;
	x = t * ctheta - s * stheta;
	y = t * stheta + s * ctheta;
	X = (int) ((x + a)/dt);
	Y = (int) ((y + a)/dt);	     
	if ((X > -1) & (X<sizeImage) & (Y>-1) & (Y<sizeImage) ){
	  h_output[Y * sizeImage + X] = 1;
	  linesum += h_input[ Y * sizeImage + X ];
	}
      }
    output = linesum * dt;

    return output;
  }
}


extern "C" {
  void radonp_local_gpu(float* h_output, float* h_input, int sizeImage,
			int nrays, int nangles, int device, int blocksize, float a,
			int centerx, int centery)

  {
    hipSetDevice(device);

    float *d_output, *d_input;

    // Allocate GPU buffers for the output sinogram
    hipMalloc(&d_output, sizeof(float) * nrays * nangles * blocksize );
    hipMalloc(&d_input, sizeof(float) * sizeImage * sizeImage * blocksize );
    hipMemcpy(d_input, h_input, sizeof(float) * sizeImage * sizeImage * blocksize, hipMemcpyHostToDevice);

    //
    dim3 threadsPerBlock(TPBX,TPBY,TPBZ);
    dim3 gridBlock((int)ceil((nrays)/threadsPerBlock.x)+1,
		   (int)ceil((nangles)/threadsPerBlock.y)+1,
		   (int)ceil(blocksize/threadsPerBlock.z)+1);

    radon_local_kernel<<<gridBlock, threadsPerBlock>>>(d_output, d_input, sizeImage, nrays, nangles, blocksize, centerx, centery);

    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(h_output, d_output, sizeof(float) * nrays * nangles * blocksize, hipMemcpyDeviceToHost);

    //cudaFree(d_input);
    hipFree(d_output);
    hipFree(d_input);
    hipDeviceReset();

    return;
  }
}
