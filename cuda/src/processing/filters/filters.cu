#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
#include "common/opt.hpp"
#include "processing/filters.hpp"
#include "common/complex.hpp"

extern "C"{
	
__global__ void fbp_filtering_C2C(Filter filter, 
    complex *kernel, dim3 size)
	{
        int i  = blockIdx.x*blockDim.x + threadIdx.x;
        int j  = blockIdx.y*blockDim.y + threadIdx.y;
        int k  = blockIdx.z*blockDim.z + threadIdx.z;

        size_t index = IND(i,j,k,size.x,size.y);
        
        if ( i >= size.x) return;
        
        float w =  2.0f * fminf( i, size.x - i ) / (float)size.x;
        // float w =  2.0f * i / (float)size.x;

        float expoent = 2.0f * float(M_PI)/(float)(size.x) * filter.axis_offset * i;

        w = filter.apply( w );

        complex aux;
        
        if ( filter.type == Filter::EType::differential){ 
            aux.x = 0.0;
            aux.y = - w;
        }else{
            aux.x = w;
            aux.y = 0.0;
        }

        kernel[index] *= exp1j(- expoent ) * aux;
        
	}

    __global__ void fbp_filtering_R2C2R(Filter filter, 
    complex *kernel, dim3 size)
	{
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        int j = blockIdx.y*blockDim.y + threadIdx.y;
        int k = blockIdx.z*blockDim.z + threadIdx.z;

        size_t index = IND(i,j,k,size.x,size.y);
        
        if ( i >= size.x) return;

        float w =  i / (float)size.x;

        float expoent = 2.0f * float(M_PI)/(float)( 2 * size.x - 2) * filter.axis_offset * i;

        w = filter.apply( w );

        complex aux;
        
        if ( filter.type == Filter::EType::differential){ 
            aux.x = 0.0;
            aux.y = - w;
        }else{
            aux.x = w;
            aux.y = 0.0;
        }

        kernel[index] *= exp1j(- expoent ) * aux;
	}

    void convolution_Real_C2C_1D(GPU gpus, hipfftComplex *data, 
    dim3 size, Filter filter)
	{
        dim3 threadsPerBlock(TPBX,TPBY,TPBZ);
        dim3 gridBlock( (int)ceil( size.x / threadsPerBlock.x ) + 1, 
                        (int)ceil( size.y / threadsPerBlock.y ) + 1, 
                        (int)ceil( size.z / threadsPerBlock.z ) + 1);

        HANDLE_FFTERROR(hipfftExecC2C(gpus.mplan, data, data, HIPFFT_FORWARD));

        // opt::fftshift1D<<<gridBlock,threadsPerBlock>>>(data,size);
                
        fbp_filtering_C2C<<<gridBlock,threadsPerBlock>>>(filter, (complex*)data, size);

        HANDLE_FFTERROR(hipfftExecC2C(gpus.mplan, data, data, HIPFFT_BACKWARD));

        // opt::fftshift1D<<<gridBlock,threadsPerBlock>>>(data,size);
	}

    void convolution_R2C_C2R_1D(GPU gpus, float *data, 
    dim3 size, Filter filter)
	{
        size_t nfft = opt::get_total_points(size);

        hipfftComplex *fft = opt::allocGPU<hipfftComplex>(nfft);

        dim3 threadsPerBlock(TPBX,TPBY,1);
        dim3 gridBlock( (int)ceil( size.x / threadsPerBlock.x ) + 1, 
                        (int)ceil( size.y / threadsPerBlock.y ) + 1, 
                        1);
              
        HANDLE_FFTERROR(hipfftExecR2C(gpus.mplan, data, fft));

        // opt::fftshift1D<<<gridBlock,threadsPerBlock>>>(fft,size);
                
        fbp_filtering_R2C2R<<<gridBlock,threadsPerBlock>>>(filter, (complex*)fft, size);

        HANDLE_FFTERROR(hipfftExecC2R(gpus.mplanI, fft, data));

        // float scale = (float)( 1.0f / size.x );
        // opt::scale<<<gridBlock,threadsPerBlock>>>(data, size, scale);

        // opt::fftshift1D<<<gridBlock,threadsPerBlock>>>(data,size);

        HANDLE_ERROR(hipFree(fft));
	}

    void filterFBP_1(GPU gpus, Filter filter, 
    float *tomogram, dim3 size, dim3 size_pad, dim3 pad)
	{	
        /* int dim = { 1, 2 }
            1: if plan 1D multiples cuffts
            2: if plan 2D multiples cuffts */
        int dim = 1; 

        dim3 gridBlock( (int)ceil( size_pad.x / gpus.BT.x ) + 1, 
                        (int)ceil( size_pad.y / gpus.BT.y ) + 1, 
                        (int)ceil( size_pad.z / gpus.BT.z ) + 1);


        opt::MPlanFFT(&gpus.mplan, dim, size_pad, HIPFFT_C2C);

        size_t npad = opt::get_total_points(size_pad);
        float scale = (float)( 1.0f / size_pad.x );

        hipfftComplex *dataPadded = opt::allocGPU<hipfftComplex>(npad);

        opt::paddR2C<<<gridBlock,gpus.BT>>>(tomogram, dataPadded, size, pad, 0.0f);

		convolution_Real_C2C_1D(gpus, dataPadded, size_pad, filter);

        opt::remove_paddC2R<<<gridBlock,gpus.BT>>>(dataPadded, tomogram, size, pad);

        opt::scale<<<gridBlock,gpus.BT>>>(tomogram, size, scale);

        HANDLE_ERROR(hipFree(dataPadded));
		HANDLE_FFTERROR(hipfftDestroy(gpus.mplan));
	}

	void filterFBP(GPU gpus, Filter filter, 
    float *tomogram, dim3 size, dim3 size_pad, dim3 pad)
	{	
        /* int dim = { 1, 2 }
            1: if plan 1D multiples cuffts
            2: if plan 2D multiples cuffts */
        // int dim = 1; 

        dim3 threadsPerBlock(TPBX,TPBY,TPBZ);
        dim3 gridBlock( (int)ceil( size_pad.x / TPBX ) + 1, 
                        (int)ceil( size_pad.y / TPBY ) + 1, 
                        (int)ceil( size_pad.z / TPBZ ) + 1);

        dim3 fft_size = dim3( size_pad.x / 2 + 1, size.y, 1 );

        size_t npad = opt::get_total_points(size_pad);
        float scale = (float)( 1.0f / size_pad.x );

        // opt::MPlanFFT(&gpus.mplan , dim, size_pad, HIPFFT_R2C);
        // opt::MPlanFFT(&gpus.mplanI, dim, size_pad, HIPFFT_C2R);

		hipfftPlan1d(&gpus.mplan , size_pad.x, HIPFFT_R2C, size_pad.y);
		hipfftPlan1d(&gpus.mplanI, size_pad.x, HIPFFT_C2R, size_pad.y);

        float *dataPadded = opt::allocGPU<float>(npad);

        opt::paddR2R<<<gridBlock,threadsPerBlock>>>(tomogram, dataPadded, size, pad, 0.0f);

        size_t offset; 
        for( int k = 0; k < size.z; k++){  
            
            offset = (size_t)k * size_pad.x * size_pad.y;

            convolution_R2C_C2R_1D( gpus, dataPadded + offset, fft_size, filter);
        }
        
        opt::remove_paddR2R<<<gridBlock,threadsPerBlock>>>(dataPadded, tomogram, size, pad);

        // opt::scale<<<gridBlock,threadsPerBlock>>>(tomogram, size, scale);

        HANDLE_ERROR(hipFree(dataPadded));
		HANDLE_FFTERROR(hipfftDestroy(gpus.mplan));
        HANDLE_FFTERROR(hipfftDestroy(gpus.mplanI));
	}
}

extern "C" {

	void SinoFilter(float* sino, size_t nrays, size_t nangles, size_t blocksize, int csino, bool bRampFilter, Filter reg, bool bShiftCenter, float* sintable)
	{	
		cImage fft(nrays/2+1,nangles);
		// cImage fft2(nrays/2+1,nangles);

		// printf("FILTER: %ld %ld %ld %ld \n",nrays,nangles,blocksize,nrays/2+1);

		hipfftHandle plan_r2c, plan_c2r;
		hipfftPlan1d(&plan_r2c, nrays, HIPFFT_R2C, nangles);
		hipfftPlan1d(&plan_c2r, nrays, HIPFFT_C2R, nangles);
		
		dim3 blk = fft.ShapeBlock();
		dim3 thr = fft.ShapeThread();

		// printf("Enter sino filter \n ");

		for(int k=0; k<blocksize; k++)
		{
			HANDLE_FFTERROR( hipfftExecR2C(plan_r2c, sino+k*nrays*nangles, fft.gpuptr) );

			if(bRampFilter)
				BandFilterReg<<<blk,thr>>>(fft.gpuptr, nrays/2+1, csino, bShiftCenter, sintable, reg);
			else
				std::cout << __FILE__ << " " << __LINE__ << " " << "Auto reg missing!" << std::endl;

			HANDLE_FFTERROR( hipfftExecC2R(plan_c2r, fft.gpuptr, sino+k*nrays*nangles) );
		}
		
		hipfftDestroy(plan_r2c);
		hipfftDestroy(plan_c2r);
	}

	__global__ void BandFilterReg(complex* vec, size_t sizex, int icenter, bool bShiftCenter, float* sintable, Filter mfilter)
	{
		int tx = blockIdx.x * blockDim.x + threadIdx.x;
		int ty = blockIdx.y * blockDim.y + threadIdx.y;

		float rampfilter = float(tx) / (float)sizex;
		rampfilter = mfilter.apply(rampfilter);

		float fcenter = 1.0f - (bShiftCenter ? (sintable[ty]) : 0);
		fcenter = -2*float(M_PI)/float(2*sizex-2) * fcenter * icenter;

		if(tx < sizex)
			vec[ty*sizex + tx] *= exp1j(fcenter * tx) * rampfilter;
	}

	void Highpass(rImage& x, float wid)
	{
		size_t sizex = x.sizex;
		size_t sizey = x.sizey;

		cImage fourier(sizex/2+1, sizey);
		hipfftHandle planrc;
		hipfftHandle plancr;

		// Optimize allocation
		hipfftPlan1d(&planrc, sizex, HIPFFT_R2C, sizey);
		hipfftPlan1d(&plancr, sizex, HIPFFT_C2R, sizey);

		for(size_t bz=0; bz<x.sizez; bz++)
		{
			hipfftExecR2C(planrc, x.gpuptr + sizex*sizey*bz, fourier.gpuptr);
			KFilter<<<dim3((sizex/2+32)/32,sizey),32>>>(fourier.gpuptr, sizex, wid);
			hipfftExecC2R(plancr, fourier.gpuptr, x.gpuptr + sizex*sizey*bz);
		}

		hipfftDestroy(planrc);
		hipfftDestroy(plancr);
	}

	__global__ void KFilter(complex* x, size_t sizex, float wid)
	{
		const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

		if(idx > sizex/2)
			return;

		float xs = float(idx)*wid/sizex;
		x[blockIdx.y*(sizex/2+1) + idx] *= (1.0f - expf(-20.0f*xs*xs))/sizex;
	}

	

	__device__ complex DeltaFilter(complex* img, int sizeimage, float fx, float fy)
	{
		fx = fminf(fx, sizeimage/2-1E-4f);
		int ix = int(fx);
		int iy = int(fy);

		float a = fx-ix;
		float b = fy-iy;

		const int h2n = sizeimage/2+1;

		return  img[(iy%sizeimage)*h2n + ix]*(1-a)*(1-b) +
				img[((iy+1)%sizeimage)*h2n + ix]*(1-a)*b +
				img[(iy%sizeimage)*h2n + (ix+1)]*a*(1-b) +
				img[((iy+1)%sizeimage)*h2n + (ix+1)]*a*b;
	}
	

	__global__ void BandFilterC2C(complex* vec, size_t sizex, int center, Filter mfilter = Filter())
	{
		int tx = blockIdx.x * blockDim.x + threadIdx.x;
		int ty = blockIdx.y * blockDim.y + threadIdx.y;

		float rampfilter = 2.0f*fminf(tx,sizex-tx)/(float)sizex;
		rampfilter = mfilter.apply(rampfilter);

		if(tx < sizex)
			vec[ty*sizex + tx] *= exp1j(-2*float(M_PI)/float(sizex) * center * tx) * rampfilter;
	}

    __global__ void SetX(complex* out, float* in, int sizex)
    {
        /* Float to Complex (imaginary part zero)*/
        size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
        size_t ty = blockIdx.y + gridDim.y * blockIdx.z;
        
        if(tx < sizex)
        {
            out[ty*sizex + tx].x = in[ty*sizex + tx];
            out[ty*sizex + tx].y = 0;
        }
    }

    __global__ void GetX(float* out, complex* in, int sizex)
    {
        /* Complex (real part) to Float */
        size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
        size_t ty = blockIdx.y + gridDim.y * blockIdx.z;
        
        if(tx < sizex)
            out[ty*sizex + tx] = in[ty*sizex + tx].x;
    }

    __global__ void GetXBST(void* out, complex* in, size_t sizex, float threshold, EType::TypeEnum raftDataType, int rollxy)
    {
        size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
        size_t ty = blockIdx.y + blockDim.y * blockIdx.z;
        
        if(tx >= sizex)
            return;
        
        float fpixel = (in[ty*sizex + tx].x)/float(sizex);
        BasicOps::set_pixel(out, fpixel, tx, ty, sizex, threshold, raftDataType);
    }

    void BSTFilter(hipfftHandle plan,
            complex* filtersino, float* sinoblock,
            size_t nrays, size_t nangles, int csino, Filter reg, hipStream_t stream) {

        dim3 filterblock((nrays+255)/256,nangles,1);
        dim3 filterthread(256,1,1);

        SetX<<<filterblock,filterthread, 0, stream>>>(filtersino, sinoblock, nrays);

        HANDLE_FFTERROR(hipfftExecC2C(plan, filtersino, filtersino, HIPFFT_FORWARD));

        BandFilterC2C<<<filterblock,filterthread, 0, stream>>>(filtersino, nrays, csino, reg);

        HANDLE_FFTERROR(hipfftExecC2C(plan, filtersino, filtersino, HIPFFT_BACKWARD));

        GetX<<<filterblock,filterthread, 0, stream>>>(sinoblock, filtersino, nrays);

        //hipMemset(sinoblock, 0, nrays*nangles*4);
    }

}

__host__ __device__ inline float Filter::apply(float input)
{
	float param = 0.0f;
    float aux   = 0.0f;
    
	if (type == EType::gaussian)
	{
		input *= exp(-0.693f * reg * input * input) / (1.0f + paganin * input * input);
	}
	else if (type == EType::lorentz)
	{
		input *= 1.0 / ( ( 1.0f + reg * input * input ) * (1.0f + paganin * input * input) );
	}
	else if (type == EType::cosine)
	{
		input *= cosf(float(M_PI) * 0.5f * input) / (1.0f + paganin * input * input);
	}
	else if (type == EType::rectangle)
	{
		param = fmaxf(input * reg * float(M_PI) * 0.5f, 1E-4f);
		input *= ( sinf(param) / param ) / (1.0f + paganin * input * input);
	}
	else if (type == EType::hann)
	{
		input *= 0.5f + 0.5f * cosf(2.0f * float(M_PI) * input) / (1.0f + paganin * input * input);
	}
	else if (type == EType::hamming)
	{
		input *= (0.54f + 0.46f * cosf(2.0f * float(M_PI) * input)) / (1.0f + paganin * input * input);
	}
	else if (type == EType::ramp)
	{
		input *= 1.0f / (1.0f + paganin * input * input);
	}
    else if (type == EType::differential)
	{
		input = 1.0f / ( 2.0f * float(M_PI) * SIGN(input) );
	}
    else if (type == EType::none)
	{
		input = 1.0f;
	}

	return input;
}




