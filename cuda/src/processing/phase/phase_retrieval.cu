#include "hip/hip_runtime.h"
#include "processing/filters.hpp"
#include "processing/processing.hpp"
#include "common/opt.hpp"

extern "C" {

    void setPhaseParameters(CFG *configs, float *parameters_float, int *parameters_int)
    {
        /* Set Tomogram (or detector) variables */
        configs->tomo.size     = dim3(parameters_int[0],parameters_int[1],parameters_int[2]);  

        /* Set padding */
        
        /* Pad is the integer number such that the total padding is = ( pad + 1 ) * dimension 
        Example: 
            - Data have dimension on x-axis of nx = 2048;
            - The padx = 1;
            - The new dimension is nx_pad = nx * (1 + padx) = 4096
        */
        configs->tomo.pad      = dim3(parameters_int[3],parameters_int[4],0); //dim3(parameters_int[3],parameters_int[4],parameters_int[5]);

        /* Padsize is the final dimension with padding. 
        Example:
            - Data have dimension on x-axis of nx = 2048 and padx = 1
            - padsizex = nx_pad = nx * (1 + padx) = 4096
            - See Pad example above. 
        */
        configs->tomo.padsize = dim3(configs->tomo.size.x * ( 1 + configs->tomo.pad.x),configs->tomo.size.y * ( 1 + configs->tomo.pad.y),configs->tomo.size.z);

        /* GPU blocksize */
        configs->blocksize = parameters_int[7];

        /* Set Phase  */
		configs->phase_type                = parameters_int[6]; /* Phase type */
		configs->delta_beta                = parameters_float[0]; /* Phase  delta/beta parameter */

        /* Set Geometry */
        // configs->geometry.geometry         = parameters_int[9];
        configs->geometry.detector_pixel_x = parameters_float[1];
        configs->geometry.detector_pixel_y = parameters_float[2];
        configs->geometry.energy           = parameters_float[3];
        configs->geometry.z2x              = parameters_float[4];
        configs->geometry.z2y              = parameters_float[5];
        configs->geometry.magnitude_x      = parameters_float[6];
        configs->geometry.magnitude_y      = parameters_float[7];
        configs->geometry.wavelenght       = ( plank * vc ) / configs->geometry.energy;

        /* Set magnitude [(z1+z2)/z1] according to the beam geometry */
		// switch (configs->geometry.geometry){
		// 	case 0: /* Parallel */	
		// 		configs->geometry.magnitude_x = 1.0;
		// 		configs->geometry.magnitude_y = 1.0;
		// 		break;
		// 	case 1: /* Conebeam */
		// 		// configs->geometry.magnitude_x = ( configs->geometry.z1x + configs->geometry.z2x ) / configs->geometry.z1x;
		// 		// configs->geometry.magnitude_y = ( configs->geometry.z1y + configs->geometry.z2y ) / configs->geometry.z1y;
		// 		break;
		// 	case 2: /* Fanbeam */		
		// 		// configs->geometry.magnitude_x = ( configs->geometry.z1x + configs->geometry.z2x ) / configs->geometry.z1x;
		// 		configs->geometry.magnitude_y = 1.0;
		// 		break;
		// 	default:
		// 		// printf("Parallel case as default! \n");
		// 		configs->geometry.magnitude_x = 1.0;
		// 		configs->geometry.magnitude_y = 1.0;
		// 		break;
		// }

        configs->geometry.obj_pixel_x = configs->geometry.detector_pixel_x / configs->geometry.magnitude_x;
        configs->geometry.obj_pixel_y = configs->geometry.detector_pixel_y / configs->geometry.magnitude_y;

        configs->geometry.z2x /= configs->geometry.magnitude_x;
        configs->geometry.z2y /= configs->geometry.magnitude_y;

        /* Compute memory in bytes of a single frame for Measurements and its padded version for FFT */
        configs->tomo.lenght_memory_bytes     = static_cast<float>(sizeof(float)) * configs->tomo.size.x;
        configs->tomo.width_memory_bytes      = static_cast<float>(sizeof(float)) * configs->tomo.size.y;

        configs->tomo.frame_memory_bytes      = configs->tomo.lenght_memory_bytes * configs->tomo.width_memory_bytes;
        configs->tomo.frame_padd_memory_bytes = static_cast<float>(sizeof(float)) * configs->tomo.padsize.x * configs->tomo.padsize.y;

        /* Compute total memory used of Phase Filter method on a single frame */
        configs->total_required_mem_per_frame_bytes = (
                configs->tomo.frame_memory_bytes // Projection
                ); 
    }

    void printPhaseParameters(CFG *configs)
    {
        printf("Tomo size: %d, %d, %d \n",configs->tomo.size.x,configs->tomo.size.y,configs->tomo.size.z);
        printf("Tomo Pad: %d, %d, %d \n",configs->tomo.pad.x,configs->tomo.pad.y,configs->tomo.pad.z);
        printf("Tomo Padsize: %d, %d, %d \n",configs->tomo.padsize.x,configs->tomo.padsize.y,configs->tomo.padsize.z);
        printf("Phase type: %d \n", configs->phase_type);
        printf("Phase delta / beta: %e \n", configs->delta_beta);
        printf("z2: %e \n", configs->geometry.z2x);
        printf("pixeldet: %e \n", configs->geometry.detector_pixel_x);
        printf("energy: %e \n", configs->geometry.energy);
        printf("magn: %e \n", configs->geometry.magnitude_x );
    }

	void applyPhase(CFG configs, GPU gpus, 
    float *projections, dim3 tomo, dim3 tomo_pad)
	{
		switch (configs.phase_type){
			case 0:
				/* Paganin */
				_paganin_gpu(configs, gpus, projections, tomo, tomo_pad, configs.tomo.pad);
				break;
            case 1:
				/* Paganin tomopy */
				_paganin_gpu_tomopy(configs, gpus, projections, tomo, tomo_pad, configs.tomo.pad);
				break;
            case 2:
				/* Paganin v0 */
				_paganin_gpu_v0(configs, gpus, projections, tomo, tomo_pad, configs.tomo.pad);
				break;
			default:
                // printf("Using default Paganin phase filter. \n");
				_paganin_gpu(configs, gpus, projections, tomo, tomo_pad, configs.tomo.pad);
				break;
		}	

	}

	void getPhase(CFG configs, GPU gpus, 
    float *projections, dim3 size, dim3 size_pad)
	{	
		/* Plan for Fourier transform - cufft */
		int n[] = {(int)size_pad.x,(int)size_pad.x};
		HANDLE_FFTERROR(hipfftPlanMany(&gpus.mplan, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, size.z));

		applyPhase(configs, gpus, projections, size, size_pad);
	
		// hipDeviceSynchronize();

		/* Destroy plan */
		HANDLE_FFTERROR(hipfftDestroy(gpus.mplan));
	}

	void getPhaseGPU(CFG configs, GPU gpus, 
	float *projections, int sizez, int ngpu)
	{	
		/* Initialize GPU device */
		HANDLE_ERROR(hipSetDevice(ngpu));

        /* Projection data sizes */
        int nrays      = configs.tomo.size.x;
        int nslices    = configs.tomo.size.y;
        int nrayspad   = configs.tomo.padsize.x;
        int nslicespad = configs.tomo.padsize.y;

		int i; 
        int blocksize = configs.blocksize;

        if ( blocksize == 0 ){
            int blocksize_aux  = compute_GPU_blocksize(sizez, configs.total_required_mem_per_slice_bytes, true, A100_MEM);
            blocksize          = min(sizez, blocksize_aux);
        }

        int ind_block = (int)ceil( (float) sizez / blocksize );

		float *dprojections = opt::allocGPU<float>((size_t) nrays * nslices * blocksize);

        		/* Plan for Fourier transform - cufft */
		int n[] = {nrays,nslices};
		HANDLE_FFTERROR(hipfftPlanMany(&gpus.mplan, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, blocksize));

		/* Loop for each batch of size 'batch' in threads */
		int ptr = 0, subblock; size_t ptr_block = 0;

		for (i = 0; i < ind_block; i++){

			subblock    = min(configs.tomo.size.z - ptr, blocksize);
			ptr_block = (size_t)nrays * nslices * ptr;

			/* Update pointer */
			ptr = ptr + subblock;

            if( subblock != blocksize){
				HANDLE_FFTERROR(hipfftDestroy(gpus.mplan));
				HANDLE_FFTERROR(hipfftPlanMany(&gpus.mplan, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, subblock));
			}

            opt::CPUToGPU<float>(projections + ptr_block, dprojections, 
                            (size_t)nrays * nslices * subblock);

			getPhase( configs, gpus, projections,
                    dim3(nrays, nslices, subblock), 
                    dim3(nrayspad, nslicespad, subblock)
                    );

			opt::GPUToCPU<float>(projections + ptr_block, dprojections, 
                                (size_t)nrays * nslices * subblock);

		}
		// HANDLE_ERROR(hipDeviceSynchronize());
        
        /* Destroy plan */
		HANDLE_FFTERROR(hipfftDestroy(gpus.mplan));

        /* Free memory */
		HANDLE_ERROR(hipFree(dprojections));

	}

    void getPhaseMultiGPU(int *gpus, int ngpus, 
    float *projections, float *paramf, int *parami)
	{	
		int i, Maxgpudev;
		
		/* Multiples devices */
		HANDLE_ERROR(hipGetDeviceCount(&Maxgpudev));

		/* If devices input are larger than actual devices on GPU, exit */
		for(i = 0; i < ngpus; i++) 
			assert(gpus[i] < Maxgpudev && "Invalid device number.");

		CFG configs; DIM tomo; GPU gpu_parameters;

        setPhaseParameters(&configs, paramf, parami);

        setGPUParameters(&gpu_parameters, configs.tomo.padsize, ngpus, gpus);

        printPhaseParameters(&configs);
        printGPUParameters(&gpu_parameters);

		int subvolume = (tomo.size.z + ngpus - 1) / ngpus;
		int subblock, ptr = 0; size_t ptr_volume = 0;

		if (ngpus == 1){ /* 1 device */

			getPhaseGPU(configs, gpu_parameters, projections, subvolume, gpus[0]);

		}else{
		/* Launch async Threads for each device.
			Each device solves a block of 'nrays * nangles' size.
		*/
			// See future c++ async launch
			std::vector<std::future<void>> threads = {};
            threads.reserve(ngpus);

			for (i = 0; i < ngpus; i++){
				
				subblock   = min(tomo.size.z - ptr, subvolume);
				ptr_volume = (size_t)tomo.size.x * tomo.size.y * ptr;

				/* Update pointer */
				ptr = ptr + subblock;
				
				threads.push_back( std::async(  std::launch::async, 
												getPhaseGPU,
                                                configs, 
												gpu_parameters, 
												projections + ptr_volume, 
												subblock, gpus[i]
												));		

			}
			// Log("Synchronizing all threads...\n");
		
			for (i = 0; i < ngpus; i++)
				threads[i].get();
		}	

		HANDLE_ERROR(hipDeviceSynchronize());
	}

	// void setPhaseFilterKernel(CFG *configs, GPU gpus, 
    // float *kernel, int phase_type, float phase_reg,
	// dim3 size_pad)
	// {	
	// 	// hipblasHandle_t handle = NULL;
    //     // hipblasCreate(&handle);
    //     // hipblasStatus_t stat;

	// 	/* Compute phase filter kernel */ 
	// 	switch (phase_type){
	// 			case 0:
	// 				/* code */
	// 				printf("No filter was selected!");
	// 				break;
	// 			case 1:
	// 				/* code */
	// 				paganinKernel<<<gpus.Grd,gpus.BT>>>(kernel, phase_reg, size_pad);
	// 				break;
	// 			case 2:
	// 				/* code */
	// 				bronnikovKernel<<<gpus.Grd,gpus.BT>>>(kernel, phase_reg, size_pad);
	// 				break;
	// 			case 3:
	// 				/* code */
	// 				bornKernel<<<gpus.Grd,gpus.BT>>>(kernel, phase_reg, size_pad);
	// 				break;
	// 			case 4:
	// 				/* code */
	// 				rytovKernel<<<gpus.Grd,gpus.BT>>>(kernel, phase_reg, size_pad);
	// 				break;
	// 			default:
	// 				printf("Using default Paganin phase filter. \n");
	// 				paganinKernel<<<gpus.Grd,gpus.BT>>>(kernel, phase_reg, size_pad);
	// 				break;
	// 		}

    //     /* Normalize kernel by maximum value */ 
 	// 	// int max;
    //     // stat = hipblasIsamax(handle, (int)size_pad.x * size_pad.y, kernel, 1, &max);

    //     // if (stat != HIPBLAS_STATUS_SUCCESS)
    //     //     printf("Cublas Max failed\n");

	// 	// float maximum;
	// 	// HANDLE_ERROR(hipMemcpy(&maximum, kernel + max, sizeof(float), hipMemcpyDeviceToHost));
    //     // Normalize<<<gpus.Grd,gpus.BT>>>(kernel, maximum, size_pad);

	// 	// hipblasDestroy(handle);
	// }
}

