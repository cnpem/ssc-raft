#include "hip/hip_runtime.h"
#include "../../../inc/filters.h"
#include "../../../inc/common/ffts.h"

extern "C"{
	
	void filterFBP(GPU gpus, Filter filter, float *tomogram, hipfftComplex *filter_kernel, dim3 size, dim3 size_pad)
	{	
		int n[] = {(int)size_pad.x};
		HANDLE_FFTERROR(hipfftPlanMany(&gpus.mplan1dC2C, 1, n, n, 1, size_pad.x, n, 1, size_pad.x, HIPFFT_C2C, (long long int)size.y * size.z));

		_fbp_filter<<<gpus.Grd,gpus.BT>>>(filter, filter_kernel, size_pad);

		convolution_mplan1DR2R(gpus, tomogram, filter_kernel, 0.0f, size, size_pad);

		hipfftDestroy(gpus.mplan1dC2C);
	}

	__global__ void _fbp_filter(Filter filter, hipfftComplex *kernel, dim3 size)
	{
        size_t i = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k = blockIdx.z*blockDim.z + threadIdx.z;
        size_t ind   = size.x * j + i;
		size_t index = size.y * k * size.x + ind;

		if( i > size.x ) return;

		float dt   =        2.0f / (float)size.x;
		float wMax =        1.0f / ( 2.0f * dt );
		float dw   = 2.0f * wMax / (float)size.x;

		/* Reciprocal grid */ 
        float w = - wMax + i * dw;

		w = filter.apply( w );
	
		kernel[index] = ( exp1j(- 2.0f * float(M_PI) * filter.axis_offset * w ) * w );
	}
}

extern "C" {

	void convolution_mplan2DR2R(GPU gpus, float *data, float *kernel, float pad_value, dim3 size, dim3 size_pad)
	{
        dim3 pad_size = dim3(size_pad.x, size_pad.y, size.z);
        size_t npad   = (size_t)size_pad.x * size_pad.y * size.z;

        hipfftComplex *dataPadded;

		HANDLE_ERROR(hipMalloc((void **)&dataPadded, sizeof(hipfftComplex) * npad ));

        padding<<<gpus.Grd,gpus.BT>>>(data, dataPadded, pad_value, size, pad_size);
        
        HANDLE_FFTERROR(hipfftExecC2C(gpus.mplan, dataPadded, dataPadded, HIPFFT_FORWARD));
        
        ProdComplexFloat<<<gpus.Grd,gpus.BT>>>(dataPadded, kernel, dataPadded, pad_size);	
        
        HANDLE_FFTERROR(hipfftExecC2C(gpus.mplan, dataPadded, dataPadded, HIPFFT_BACKWARD));
        
        fftNormalize2d<<<gpus.Grd,gpus.BT>>>(dataPadded, pad_size);

        recuperate_padding<<<gpus.Grd,gpus.BT>>>(dataPadded, data, size, pad_size);

		hipFree(dataPadded);
	}

	void convolution_mplan1DR2R(GPU gpus, float *data, hipfftComplex *kernel, float pad_value, dim3 size, dim3 size_pad)
	{
        dim3 pad_size = dim3(size_pad.x, size.y, size.z);
        size_t npad   = (size_t)size_pad.x * size.y * size.z;

        hipfftComplex *dataPadded;

		HANDLE_ERROR(hipMalloc((void **)&dataPadded, sizeof(hipfftComplex) * npad ));

        padding<<<gpus.Grd,gpus.BT>>>(data, dataPadded, pad_value, size, pad_size);
        
        HANDLE_FFTERROR(hipfftExecC2C(gpus.mplan, dataPadded, dataPadded, HIPFFT_FORWARD));
        
        ProdComplexComplex<<<gpus.Grd,gpus.BT>>>(dataPadded, kernel, dataPadded, pad_size);	
        
        HANDLE_FFTERROR(hipfftExecC2C(gpus.mplan, dataPadded, dataPadded, HIPFFT_BACKWARD));
        
        fftNormalize1d<<<gpus.Grd,gpus.BT>>>(dataPadded, pad_size);

        recuperate_padding<<<gpus.Grd,gpus.BT>>>(dataPadded, data, size, pad_size);

		hipFree(dataPadded);
	}

    

    __global__ void ProdComplexFloat(hipfftComplex *a, float *b, hipfftComplex *ans, dim3 size)
    {
        size_t i = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k = blockIdx.z*blockDim.z + threadIdx.z;
        size_t ind   = size.x * j + i;
        size_t index = size.y * k * size.x + ind;

        if ( (i >= size.x) || (j >= size.y) || (k >= size.z) ) return;
        ans[index].x = a[index].x * b[ind];	
        ans[index].y = a[index].y * b[ind];	
    }

	__global__ void ProdComplexComplex(hipfftComplex *a, hipfftComplex *b, hipfftComplex *ans, dim3 size)
    {
        size_t i = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k = blockIdx.z*blockDim.z + threadIdx.z;
        size_t ind   = size.x * j + i;
        size_t index = size.y * k * size.x + ind;

        if ( (i >= size.x) || (j >= size.y) || (k >= size.z) ) return;
        ans[index] = ComplexMult(a[index],b[ind]);
    }

    __global__ void fftshiftKernel(float *c, dim3 size)
    {
        int shift;
        int N = ( (size.x * size.y) + size.x ) / 2 ;	
        int M = ( (size.x * size.y) - size.x ) / 2 ;	
        float temp;
        size_t i = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k = blockIdx.z*blockDim.z + threadIdx.z;
        int index; 

        if ( (i >= size.x) || (j >= size.y) || (k >= size.z) ) return;
        
        if ( i < ( size.x / 2 ) ){	
            if ( j < ( size.y / 2 ) ){	
                index = size.x * (k*size.y + j)  + i;
                shift = index + N;
                temp 	 = c[index];	
                c[index] = c[shift];	
                c[shift] = temp;
            }
        }else{
            if ( j < ( size.y / 2 ) ){
                index = size.x * (k*size.y + j)  + i;
                shift = index + M;
                temp 	 = c[index];	
                c[index] = c[shift];	
                c[shift] = temp;
            }
        }
    }

    __global__ void Normalize(float *a, float b, dim3 size)
    {
        size_t i = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k = blockIdx.z*blockDim.z + threadIdx.z;
        
        size_t index = size.x * j + i;
        
        if ( (i >= size.x) || (j >= size.y) || (k >= 1) ) return;
        
        a[index] = a[index] / b; 
    }

	void SinoFilter(float* sino, size_t nrays, size_t nangles, size_t blocksize, int csino, bool bRampFilter, Filter reg, bool bShiftCenter, float* sintable)
	{	
		cImage fft(nrays/2+1,nangles);
		// cImage fft2(nrays/2+1,nangles);

		// printf("FILTER: %ld %ld %ld %ld \n",nrays,nangles,blocksize,nrays/2+1);

		hipfftHandle plan_r2c, plan_c2r;
		hipfftPlan1d(&plan_r2c, nrays, HIPFFT_R2C, nangles);
		hipfftPlan1d(&plan_c2r, nrays, HIPFFT_C2R, nangles);
		
		dim3 blk = fft.ShapeBlock();
		dim3 thr = fft.ShapeThread();

		// printf("Enter sino filter \n ");

		for(int k=0; k<blocksize; k++)
		{
			HANDLE_FFTERROR( hipfftExecR2C(plan_r2c, sino+k*nrays*nangles, fft.gpuptr) );

			if(bRampFilter)
				BandFilterReg<<<blk,thr>>>(fft.gpuptr, nrays/2+1, csino, bShiftCenter, sintable, reg);
			else
				std::cout << __FILE__ << " " << __LINE__ << " " << "Auto reg missing!" << std::endl;

			HANDLE_FFTERROR( hipfftExecC2R(plan_c2r, fft.gpuptr, sino+k*nrays*nangles) );
		}
		
		hipfftDestroy(plan_r2c);
		hipfftDestroy(plan_c2r);
	}

	__global__ void BandFilterReg(complex* vec, size_t sizex, int icenter, bool bShiftCenter, float* sintable, Filter mfilter)
	{
		int tx = blockIdx.x * blockDim.x + threadIdx.x;
		int ty = blockIdx.y * blockDim.y + threadIdx.y;

		float rampfilter = float(tx) / (float)sizex;
		rampfilter = mfilter.apply(rampfilter);
		// printf("band filter value: %lf \n ",rampfilter);

		float fcenter = 1.0f - (bShiftCenter ? (sintable[ty]) : 0);
		fcenter = -2*float(M_PI)/float(2*sizex-2) * fcenter * icenter;

		if(tx < sizex)
			vec[ty*sizex + tx] *= exp1j(fcenter * tx) * rampfilter;
	}

	void Highpass(rImage& x, float wid)
	{
		size_t sizex = x.sizex;
		size_t sizey = x.sizey;

		cImage fourier(sizex/2+1, sizey);
		hipfftHandle planrc;
		hipfftHandle plancr;

		// Optimize allocation
		hipfftPlan1d(&planrc, sizex, HIPFFT_R2C, sizey);
		hipfftPlan1d(&plancr, sizex, HIPFFT_C2R, sizey);

		for(size_t bz=0; bz<x.sizez; bz++)
		{
			hipfftExecR2C(planrc, x.gpuptr + sizex*sizey*bz, fourier.gpuptr);
			KFilter<<<dim3((sizex/2+32)/32,sizey),32>>>(fourier.gpuptr, sizex, wid);
			hipfftExecC2R(plancr, fourier.gpuptr, x.gpuptr + sizex*sizey*bz);
		}

		hipfftDestroy(planrc);
		hipfftDestroy(plancr);
	}

	__global__ void KFilter(complex* x, size_t sizex, float wid)
	{
		const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

		if(idx > sizex/2)
			return;

		float xs = float(idx)*wid/sizex;
		x[blockIdx.y*(sizex/2+1) + idx] *= (1.0f - expf(-20.0f*xs*xs))/sizex;
	}

	

	__device__ complex DeltaFilter(complex* img, int sizeimage, float fx, float fy)
	{
		fx = fminf(fx, sizeimage/2-1E-4f);
		int ix = int(fx);
		int iy = int(fy);

		float a = fx-ix;
		float b = fy-iy;

		const int h2n = sizeimage/2+1;

		return  img[(iy%sizeimage)*h2n + ix]*(1-a)*(1-b) +
				img[((iy+1)%sizeimage)*h2n + ix]*(1-a)*b +
				img[(iy%sizeimage)*h2n + (ix+1)]*a*(1-b) +
				img[((iy+1)%sizeimage)*h2n + (ix+1)]*a*b;
	}
	
	// inline __global__ void SetX(complex* out, float* in, int sizex)
	// {
	// 	size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
	// 	size_t ty = blockIdx.y + gridDim.y * blockIdx.z;
		
	// 	if(tx < sizex)
	// 	{
	// 		out[ty*sizex + tx].x = in[ty*sizex + tx];
	// 		out[ty*sizex + tx].y = 0;
	// 	}
	// }
	
	// inline __global__ void GetX(float* out, complex* in, int sizex)
	// {
	// 	size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
	// 	size_t ty = blockIdx.y + gridDim.y * blockIdx.z;
		
	// 	if(tx < sizex)
	// 		out[ty*sizex + tx] = (in[ty*sizex + tx].x)/sizex;
	// }
	
	// inline __global__ void GetXBST(void* out, complex* in, size_t sizex, float threshold, EType::TypeEnum raftDataType, int rollxy)
	// {
	// 	size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
	// 	size_t ty = blockIdx.y + blockDim.y * blockIdx.z;
		
	// 	if(tx >= sizex)
	// 		return;
		
	// 	float fpixel = (in[ty*sizex + tx].x)/float(sizex);
	// 	 BasicOps::set_pixel(out, fpixel, tx, ty, sizex, threshold, raftDataType);
	// }

	__global__ void BandFilterC2C(complex* vec, size_t sizex, int center, Filter mfilter = Filter())
	{
		int tx = blockIdx.x * blockDim.x + threadIdx.x;
		int ty = blockIdx.y * blockDim.y + threadIdx.y;

		float rampfilter = 2.0f*fminf(tx,sizex-tx)/(float)sizex;
		rampfilter = mfilter.apply(rampfilter);

		if(tx < sizex)
			vec[ty*sizex + tx] *= exp1j(-2*float(M_PI)/float(sizex) * center * tx) * rampfilter;
	}

	// void BSTFilter(hipfftHandle plan, complex* filtersino, float* sinoblock, size_t nrays, size_t nangles, int csino, Filter reg)
	// {

	// 	dim3 filterblock((nrays+255)/256,nangles,1);
	// 	dim3 filterthread(256,1,1);

	// 	SetX<<<filterblock,filterthread>>>(filtersino, sinoblock, nrays);
			
	// 	HANDLE_FFTERROR(hipfftExecC2C(plan, filtersino, filtersino, HIPFFT_FORWARD));
			
	// 	BandFilterC2C<<<filterblock,filterthread>>>(filtersino, nrays, csino, reg);
			
	// 	HANDLE_FFTERROR(hipfftExecC2C(plan, filtersino, filtersino, HIPFFT_BACKWARD));
		
	// 	GetX<<<filterblock,filterthread>>>(sinoblock, filtersino, nrays);

	// 	//hipMemset(sinoblock, 0, nrays*nangles*4);
	// }

}

__host__ __device__ inline float Filter::apply(float input)
{
	float param = 0.0f;

	if (type == EType::gaussian)
	{
		input *= exp(-0.693f * reg * input * input);
		input /= (1.0f + paganin * input * input);
	}
	else if (type == EType::lorentz)
	{
		input /= 1.0f + reg * input * input;
		input /= (1.0f + paganin * input * input);
	}
	else if (type == EType::cosine)
	{
		input *= cosf(float(M_PI) * 0.5f * input);
		input /= (1.0f + paganin * input * input);
	}
	else if (type == EType::rectangle)
	{
		param = fmaxf(input * reg * float(M_PI) * 0.5f, 1E-4f);
		input *= sinf(param) / param;
		input /= (1.0f + paganin * input * input);
	}
	else if (type == EType::hann)
	{
		input *= 0.5f + 0.5f * cosf(2.0f * float(M_PI) * input);
		input /= (1.0f + paganin * input * input);
	}
	else if (type == EType::hamming)
	{
		input *= (0.54f + 0.46f * cosf(2.0f * float(M_PI) * input));
		input /= (1.0f + paganin * input * input);
	}
	else if (type == EType::ramp)
	{
		input /= (1.0f + paganin * input * input);
	}

	return input;
}

Convolution::Plan(hipfftHandle _mplan, hipfftHandle _implan, int _dim, dim3 _pad, Convolution::fftType _type) :
mplan(mplan), implan(_implan), dim(_dim), pad(_pad), typefft((fftType)_type)
{	
    int sizeArray[dim];

    if ( dim == 1 ) 
        sizeArray[dim] = {(int)pad.x};
    else
        sizeArray[dim] = {(int)pad.x,(int)pad.y};

    int rank      = dim;
    int inembed   = sizeArray;
    int istride   = 1;
    int idist     = pad.x;
    int onembed   = sizeArray;
    int ostride   = 1;
    int odist     = pad.x;
    size_t _batch = pad.z;
	
    switch (typefft){
	    case 0:
            HANDLE_FFTERROR(hipfftPlanMany(&mplan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));
            break;
	    case 1:
            HANDLE_FFTERROR(hipfftPlanMany(&mplan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));
            HANDLE_FFTERROR(hipfftPlanMany(&implan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2R, batch));
            break;
	    case 2:
            HANDLE_FFTERROR(hipfftPlanMany(&mplan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch));
            HANDLE_FFTERROR(hipfftPlanMany(&implan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2R, batch));
		    break;
	    case 3:
            HANDLE_FFTERROR(hipfftPlanMany(&mplan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch));
            HANDLE_FFTERROR(hipfftPlanMany(&implan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));
		    break;
	    case 4:
            HANDLE_FFTERROR(hipfftPlanMany(&mplan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2R, batch));
            HANDLE_FFTERROR(hipfftPlanMany(&implan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch));
		    break;
	    default:
            HANDLE_FFTERROR(hipfftPlanMany(&mplan, rank, sizeArray, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));
		    break;
	}
}

Convolution::~Plan(hipfftHandle _mplan, hipfftHandle _implan, Convolution::fftType _type) :
mplan(mplan), implan(_implan), typefft((fftType)_type)
{
    switch (typefft){
	    case 0:
            HANDLE_FFTERROR(hipfftDestroy(mplan));
            break;
	    case 1:
            HANDLE_FFTERROR(hipfftDestroy(mplan));
            HANDLE_FFTERROR(hipfftDestroy(implan));
            break;
	    case 2:
            HANDLE_FFTERROR(hipfftDestroy(mplan));
            HANDLE_FFTERROR(hipfftDestroy(implan));
            break;
	    case 3:
            HANDLE_FFTERROR(hipfftDestroy(mplan));
            HANDLE_FFTERROR(hipfftDestroy(implan));
            break;
	    case 4:
            HANDLE_FFTERROR(hipfftDestroy(mplan));
            HANDLE_FFTERROR(hipfftDestroy(implan));
            break;
	    default:
            HANDLE_FFTERROR(hipfftDestroy(mplan));
		    break;
	}
}

template<typename Type1, typename Type2, typename Type3>
void Convolution::convolve(GPU gpus, Type1 *input, Type2 *kernel, Type3 *output)
{
    fftType _fft  = typefft;  
    padType _pad_forward = Convolution::setPad<Type1>(_fft);
    padType _pad_inverse = Convolution::setPad<Type2>(_fft);
    
    switch (_pad_forward){
	    case 0:
            hipfftComplex* padgpuptr = Opt::allocGPU<hipfftComplex>(n_pad);
            Convolution::padding<hipfftComplex,hipfftComplex>(gpus, input, padgpuptr);
		    break;
	    case 1:
            hipfftComplex* padgpuptr = Opt::allocGPU<hipfftComplex>(n_pad);
            Convolution::padding<float,hipfftComplex>(gpus, input, padgpuptr);
		    break;
        case 2:
            float* padgpuptr = Opt::allocGPU<float>(n_pad);
            Convolution::padding<hipfftComplex,float>(gpus, input, padgpuptr);
		    break;
        case 3:
            float* padgpuptr = Opt::allocGPU<float>(n_pad);
            Convolution::padding<float,float>(gpus, input, padgpuptr);
		    break;
	    default:
            hipfftComplex* padgpuptr = Opt::allocGPU<hipfftComplex>(n_pad);
            Convolution::padding<Type1,hipfftComplex>(gpus, input, padgpuptr);
		    break;
	}

    switch (_fft){
	    case 0:
            HANDLE_FFTERROR(hipfftExecC2C(mplan, padgpuptr, padgpuptr, HIPFFT_FORWARD));
		    break;
	    case 1:
            HANDLE_FFTERROR(cufftExecR2R(mplan, padgpuptr, padgpuptr, HIPFFT_FORWARD));
		    break;
	    default:
            HANDLE_FFTERROR(hipfftExecC2C(mplan, padgpuptr, padgpuptr, HIPFFT_FORWARD));
		    break;
	}

    Opt::pointTopointProd<Type3, Type2>(gpus, Type1 *a, Type2 *b, Type1 *ans, pad, dim3 sizeb)

    switch (_fft){
	    case 0:	
            HANDLE_FFTERROR(hipfftExecC2C(mplan, padgpuptr, padgpuptr, HIPFFT_BACKWARD));
            Convolution::remove_padding<hipfftComplex, hipfftComplex>(gpus, padgpuptr, gpuptr);
		    break;
	    case 1:
            HANDLE_FFTERROR(cufftExecR2R(implan, padgpuptr, padgpuptr, HIPFFT_BACKWARD));
            Convolution::remove_padding<float, float>(gpus, padgpuptr, gpuptr);
		    break;

	    default:
            HANDLE_FFTERROR(hipfftExecC2C(implan, padgpuptr, padgpuptr, HIPFFT_BACKWARD));
            Convolution::remove_padding<hipfftComplex, hipfftComplex>(gpus, padgpuptr, gpuptr);
		    break;
	}
    
    switch (_pad_inverse){
	    case 0:
            Convolution::remove_padding<hipfftComplex,hipfftComplex>(gpus, padgpuptr, output, _pad_inverse);
		    break;
	    case 1:
            Convolution::padding<float,hipfftComplex>(gpus, input, padgpuptr);
		    break;
        case 2:
            Convolution::padding<hipfftComplex,float>(gpus, input, padgpuptr);
		    break;
        case 3:
            Convolution::padding<float,float>(gpus, input, padgpuptr);
		    break;
	    default:
            hipfftComplex* padgpuptr = Opt::allocGPU<hipfftComplex>(n_pad);
            Convolution::padding<Type1,hipfftComplex>(gpus, input, padgpuptr);
		    break;
	}

    HANDLE_ERROR(hipFree(padgpuptr));

    if (type == fftType::C2R_R2C || type == fftType::R2C_C2R)
        HANDLE_ERROR(hipFree(ipadgpuptr));    
}
