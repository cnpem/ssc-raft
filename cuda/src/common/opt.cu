#include "hip/hip_runtime.h"
#include <hip/driver_types.h>

#include <chrono>
#include <cstddef>
#include <cstdlib>
#include <cstring>
#include <thread>
#include <utility>
#include <vector>
#include <sys/mman.h>

#include "common/complex.hpp"
#include "common/configs.hpp"
#include "common/logerror.hpp"
#include "common/opt.hpp"

/*============================================================================*/
/* namespace opt (in 'inc/commons/opt.hpp') functions definitions */


void opt::flip_x(float *data, int sizex, int sizey, int sizez) {
    const size_t sizexy = sizex * sizey;

    for (size_t k = 0; k < sizez; ++k) {
        for (size_t j = 0; j < sizey; ++j) {
            float* row = data + j * sizex + k * sizexy;
            for (size_t i = 0; i < (sizex/2); ++i) {
                std::swap(row[i], row[sizex - 1 - i]);
            }
        }
    }
}

void _transpose_zyx2xyz_worker(float* out, float* in,
        size_t i, size_t sizex, size_t sizey, size_t sizez) {
    const size_t sizexy = sizex * sizey;
    const size_t sizezy = sizez * sizey;
    for (size_t j = 0; j < sizey; ++j) {
            for (size_t k = 0; k < sizez; ++k) {
                out[i * sizezy + j * sizez + k] = in[k * sizexy + j * sizex + i];
            }
        }
}

void opt::transpose_cpu_zyx2xyz(float *data, int sizex, int sizey, int sizez) {
    const size_t sizexy = sizex * sizey;
    const size_t sizezy = sizez * sizey;
    const size_t sizexyz = size_t(sizex) * size_t(sizey) * size_t(sizez);

    float *temp = (float *)aligned_alloc(64, sizeof(float) * sizexyz);

    mlock(data, sizeof(float) * sizexyz);

    std::vector<std::thread> threads;
    threads.reserve(sizex);

    for(size_t i = 0; i < sizex; ++i) {
        threads.emplace_back(_transpose_zyx2xyz_worker,
                temp, data, i, sizex, sizey, sizez);
    }

    for(auto& t: threads) {
        t.join();
    }

    memcpy(data, temp, sizeof(float) * sizexyz);

    munlock(data, sizeof(float) * sizexyz);

    free(temp);
}

void opt::MPlanFFT(hipfftHandle *mplan, int RANK, dim3 DATASIZE, hipfftType FFT_TYPE) {
    /* rank:
    Dimensionality of the transform: 1D (1), 2D (2) or 3D (3) cufft */

    /* FFT_TYPE: The transform data type
        HIPFFT_R2C, HIPFFT_C2R, HIPFFT_C2C
    */

    /* Array of size rank, describing the size of each dimension,
    n[0] being the size of the outermost and
    n[rank-1] innermost (contiguous) dimension of a transform. */
    int *n = (int *)malloc(RANK);
    n[0] = (int)DATASIZE.y;

    int idist = DATASIZE.x; /* Input data distance between batches */
    int odist = DATASIZE.x; /* Output data distance between batches */

    if (FFT_TYPE == HIPFFT_C2R) idist = idist / 2 + 1;

    if (FFT_TYPE == HIPFFT_R2C) odist = odist / 2 + 1;

    int batch = DATASIZE.y * DATASIZE.z; /* Number of batched executions */

    if (RANK >= 2) {
        n[1] = (int)DATASIZE.x;
        batch = DATASIZE.z;
        idist *= DATASIZE.y;
        odist *= DATASIZE.y;
    } else if (RANK >= 3) {
        n[2] = (int)DATASIZE.z;
        idist *= DATASIZE.z;
        odist *= DATASIZE.z;
        batch = 1;
    }

    int *inembed = n, *onembed = n; /* Input/Output size with pitch (ignored for 1D transforms).
    If set to NULL all other advanced data layout parameters are ignored. */
    int istride = 1, ostride = 1;         /* Distance between two successive input/output elements. */

    HANDLE_FFTERROR(hipfftPlanMany(mplan, RANK, n, inembed, istride, idist, onembed, ostride, odist, FFT_TYPE, batch));
}

__global__ void setSinCosTable(float *sintable, float *costable, float *angles, int nangles) {
    size_t k = blockIdx.x * blockDim.x + threadIdx.x;

    if ((k >= nangles)) return;

    sintable[k] = __sinf(angles[k]);
    costable[k] = __cosf(angles[k]);
}

void getLog(float *data, dim3 size, hipStream_t stream) {
    dim3 threadsPerBlock(TPBX, TPBY, TPBZ);
    dim3 gridBlock((int)ceil(size.x / threadsPerBlock.x) + 1, (int)ceil(size.y / threadsPerBlock.y) + 1,
                   (int)ceil(size.z / threadsPerBlock.z) + 1);

    Klog<<<gridBlock, threadsPerBlock, 0, stream>>>(data, size);
}

static __global__ void Klog(float *data, dim3 size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    if ((i >= size.x) || (j >= size.y) || (k >= size.z)) return;

    size_t index = IND(i, j, k, size.x, size.y);

    data[index] = -logf(data[index]);
}


dim3 opt::setGridBlock(dim3 size) {
    dim3 gridBlock((int)ceil(size.x / TPBX) + 1, (int)ceil(size.y / TPBY) + 1, (int)ceil(size.z / TPBZ) + 1);

    return gridBlock;
}

__global__ void opt::scale(hipfftComplex *data, dim3 size, float scale) {

    size_t i = blockIdx.x*blockDim.x + threadIdx.x; 
    size_t j = blockIdx.y*blockDim.y + threadIdx.y; 
    size_t k = blockIdx.z*blockDim.z + threadIdx.z;

    size_t index = size.y * size.x * k + size.x * j + i; 

    if( (i >= size.x) || (j >= size.y) || (k >= size.z)) return;  

    data[index].x = data[index].x / scale;
    data[index].y = data[index].y / scale;
}

