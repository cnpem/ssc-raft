#include "hip/hip_runtime.h"
#include <hip/driver_types.h>

#include <chrono>
#include <cstddef>
#include <cstdlib>
#include <cstring>
#include <thread>
#include <utility>
#include <vector>

#include "common/complex.hpp"
#include "common/configs.hpp"
#include "common/logerror.hpp"
#include "common/opt.hpp"

/*============================================================================*/
/* namespace opt (in 'inc/commons/opt.hpp') functions definitions */


void opt::flip_x(float *data, int sizex, int sizey, int sizez) {
    const size_t sizexy = sizex * sizey;

    for (size_t k = 0; k < sizez; ++k) {
        for (size_t j = 0; j < sizey; ++j) {
            float* row = data + j * sizex + k * sizexy;
            for (size_t i = 0; i < (sizex/2); ++i) {
                std::swap(row[i], row[sizex - 1 - i]);
            }
        }
    }
}

//this implementation is still quite naive, but can be faster than numpy for big arrays
void opt::transpose_cpu(float *data, int sizex, int sizey, int sizez) {
    const size_t sizexy = sizex * sizey;
    const size_t sizexz = sizex * sizez;
    const size_t sizexyz = size_t(sizex) * size_t(sizey) * size_t(sizez);

    float *temp = (float *)aligned_alloc(64, sizeof(float) * sizexyz);

    for (size_t j = 0; j < sizey; ++j) {
        for (size_t k = 0; k < sizez; ++k) {
            memcpy(temp + j * sizexz + k * sizex, data + j * sizex + k * sizexy,
                    sizeof(float) * sizex);
        }
    }

    memcpy(data, temp, sizeof(float) * sizexyz);

    free(temp);
}

void opt::MPlanFFT(hipfftHandle *mplan, int RANK, dim3 DATASIZE, hipfftType FFT_TYPE) {
    /* rank:
    Dimensionality of the transform: 1D (1), 2D (2) or 3D (3) cufft */

    /* FFT_TYPE: The transform data type
        HIPFFT_R2C, HIPFFT_C2R, HIPFFT_C2C
    */

    /* Array of size rank, describing the size of each dimension,
    n[0] being the size of the outermost and
    n[rank-1] innermost (contiguous) dimension of a transform. */
    int *n = (int *)malloc(RANK);
    n[0] = (int)DATASIZE.x;

    int idist = DATASIZE.x; /* Input data distance between batches */
    int odist = DATASIZE.x; /* Output data distance between batches */

    if (FFT_TYPE == HIPFFT_C2R) idist = idist / 2 + 1;

    if (FFT_TYPE == HIPFFT_R2C) odist = odist / 2 + 1;

    int batch = DATASIZE.y * DATASIZE.z; /* Number of batched executions */

    if (RANK >= 2) n[1] = (int)DATASIZE.y;
    batch = DATASIZE.z;
    idist *= DATASIZE.y;
    odist *= DATASIZE.y;

    if (RANK >= 3) {
        n[2] = (int)DATASIZE.z;
        idist *= DATASIZE.z;
        odist *= DATASIZE.z;
        batch = 1;
    }

    int *inembed = NULL, *onembed = NULL; /* Input/Output size with pitch (ignored for 1D transforms).
    If set to NULL all other advanced data layout parameters are ignored. */
    int istride = 1, ostride = 1;         /* Distance between two successive input/output elements. */

    HANDLE_FFTERROR(hipfftPlanMany(mplan, RANK, n, inembed, istride, idist, onembed, ostride, odist, FFT_TYPE, batch));
}

__global__ void setSinCosTable(float *sintable, float *costable, float *angles, int nangles) {
    size_t k = blockIdx.x * blockDim.x + threadIdx.x;

    if ((k >= nangles)) return;

    sintable[k] = __sinf(angles[k]);
    costable[k] = __cosf(angles[k]);
}

void getLog(float *data, dim3 size, hipStream_t stream) {
    dim3 threadsPerBlock(TPBX, TPBY, TPBZ);
    dim3 gridBlock((int)ceil(size.x / threadsPerBlock.x) + 1, (int)ceil(size.y / threadsPerBlock.y) + 1,
                   (int)ceil(size.z / threadsPerBlock.z) + 1);

    Klog<<<gridBlock, threadsPerBlock, 0, stream>>>(data, size);
}

static __global__ void Klog(float *data, dim3 size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    if ((i >= size.x) || (j >= size.y) || (k >= size.z)) return;

    size_t index = IND(i, j, k, size.x, size.y);

    data[index] = -logf(data[index]);
}


dim3 opt::setGridBlock(dim3 size) {
    dim3 gridBlock((int)ceil(size.x / TPBX) + 1, (int)ceil(size.y / TPBY) + 1, (int)ceil(size.z / TPBZ) + 1);

    return gridBlock;
}

__global__ void opt::scale(hipfftComplex *data, dim3 size, float scale) {

    size_t i = blockIdx.x*blockDim.x + threadIdx.x; 
    size_t j = blockIdx.y*blockDim.y + threadIdx.y; 
    size_t k = blockIdx.z*blockDim.z + threadIdx.z;

    size_t index = size.y * size.x * k + size.x * j + i; 

    if( (i >= size.x) || (j >= size.y) || (k >= size.z)) return;  

    data[index].x = data[index].x / scale;
    data[index].y = data[index].y / scale;
}

