#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>
#include <cstdio>
#include "common/configs.hpp"
#include "common/opt.hpp"
#include "pipelines/pipeline.hpp"
#include "geometries/parallel/em.hpp"
#include "geometries/parallel/fbp.hpp"
#include "geometries/parallel/bst.hpp"
#include "processing/processing.hpp"

using std::thread;

extern "C"{
    void ReconstructionPipeline(float *obj, float *data,
            float *flats, float *darks, float *angles,
            float *parameters_float, int *parameters_int, int *flags,
            int *gpus, int ngpus)
    {
        int i, Maxgpu;
        int total_number_of_processes;

        /* Multiples devices */
        hipGetDeviceCount(&Maxgpu);

        /* If devices input are larger than actual devices on GPU, exit */
        for(i = 0; i < ngpus; i++)
            assert(gpus[i] < Maxgpu && "Invalid device number.");

        CFG configs; GPU gpu_parameters;

        setReconstructionParameters(&configs, parameters_float, parameters_int, flags);

        setGPUParameters(&gpu_parameters, configs.tomo.padsize, ngpus, gpus);

        /* Set total number of processes to be sent to the GPUs */
        total_number_of_processes = getTotalProcesses(configs, BYTES_TO_GB * getTotalDeviceMemory(), configs.tomo.size.z, true);

        /* Set processes pipeline for different geometries */
        Process *process = setProcesses(configs, gpu_parameters, total_number_of_processes);

        // clock_t b_begin = clock();

        _setReconstructionPipeline(&configs, process, gpu_parameters,
                obj, data, flats, darks, angles,
                total_number_of_processes);

        HANDLE_ERROR(hipGetLastError());

        /* Free process (array of structs) */
        free(process);
    }
}


extern "C"{
    void _setReconstructionPipeline(CFG *configs, Process *process, GPU gpus,
            float *obj, float *data, float *flats, float *darks,
            float *angles, int total_number_of_processes)
    {

        configs->tomo.batchsize    = dim3(   configs->tomo.size.x, configs->tomo.size.y, process->tomobatch_size);
        configs->tomo.padbatchsize = dim3(configs->tomo.padsize.x, configs->tomo.padsize.y, process->tomobatch_size);
        configs->obj.batchsize     = dim3(    configs->obj.size.x,  configs->obj.size.y,  process->objbatch_size);

        std::vector<thread> threads_pipeline;

        for (int p = 0; p < total_number_of_processes; ++p) {

            threads_pipeline.emplace_back(  thread(
                        _ReconstructionProcessPipeline,
                        (*configs), process[p],
                        gpus, obj, data, flats, darks,
                        angles
                        ));

            if (p % gpus.ngpus == gpus.ngpus - 1) {
                for (int g = 0; g < gpus.ngpus; ++g) {
                    threads_pipeline[g].join();
                    hipSetDevice(g);
                    hipDeviceSynchronize();
                }
                threads_pipeline.clear();
            }


        }
    }
}


extern "C" {

    void _ReconstructionProcessPipeline(CFG configs, Process process, GPU gpus,
            float *obj, float *data, float *flats, float *darks, float *angles)
    {

        /* Initialize GPU device */
        HANDLE_ERROR(hipSetDevice(process.index_gpu));

        /* Local GPUs Pointers: allocation */
        WKP *workspace = allocateWorkspace(configs, process.tomobatch_size, process.objbatch_size);

        /* Copy data from host to device */
        HANDLE_ERROR(hipMemcpy(workspace->angles, angles, configs.tomo.size.y * sizeof(float), hipMemcpyHostToDevice));


        HANDLE_ERROR(hipMemcpy(workspace->tomo,  data + process.tomoptr_index,
                    process.tomoptr_size * sizeof(float), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(workspace->flat, flats + process.tomo_index_z * configs.tomo.size.x,
                    process.tomobatch_size * configs.tomo.size.x * sizeof(float), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(workspace->dark, darks + process.tomo_index_z * configs.tomo.size.x,
                    process.tomobatch_size * configs.tomo.size.x * sizeof(float), hipMemcpyHostToDevice));

        /* Enter Reconstruction Pipeline */
        _ReconstructionPipeline(configs, workspace, gpus);

        /* Copy Reconstructed data from device to host */
        HANDLE_ERROR(hipMemcpy(&obj[process.objptr_index], workspace->obj, process.objptr_size * sizeof(float), hipMemcpyDeviceToHost));

        freeWorkspace(workspace, configs);

        // hipDeviceSynchronize();
    }
}


extern "C"{
    void _ReconstructionPipeline(CFG configs, WKP *workspace, GPU gpus)
    {
        if( configs.flags.do_flat_dark_correction )
            getBackgroundCorrection(gpus, workspace->tomo, workspace->flat, workspace->dark,
                    configs.tomo.batchsize, configs.numflats);

        if( configs.flags.do_flat_dark_log )
            getLog(workspace->tomo, configs.tomo.batchsize);

        printf("Do rings with: lambda: %f rings_block: %d\n",
                configs.rings_lambda, configs.rings_block);

        if( configs.flags.do_rings )
            getTitarenkoRings(gpus, workspace->tomo,
                    configs.tomo.batchsize, configs.rings_lambda,
                    configs.rings_block);


        printf("Do rotation? %d\n", configs.flags.do_rotation);

        if ( configs.flags.do_rotation) {
            printf("do rotation_auto_offset: %d\n", configs.flags.do_rotation_auto_offset);
            const int rotation_axis_offset = configs.flags.do_rotation_auto_offset ?
                 getCentersino(workspace->tomo, workspace->tomo,
                         workspace->dark, workspace->flat,
                         configs.tomo.size.x, configs.tomo.size.y) :
                 configs.rotation_axis_offset;
            printf("deviation: %d\n", rotation_axis_offset);
            getCorrectRotationAxis(workspace->tomo, workspace->tomo,
                    configs.tomo.batchsize, rotation_axis_offset);
        }

        getReconstructionMethods(configs, gpus, workspace);
    }
}

extern "C"{


    void getReconstructionMethods(CFG configs, GPU gpus, WKP *workspace)
    {
        switch (configs.reconstruction_method){
            case 0:
                /* FBP */
                getFBP( configs, gpus,
                        workspace->obj,
                        workspace->tomo,
                        workspace->angles,
                        configs.tomo.batchsize,
                        configs.tomo.padbatchsize,
                        configs.obj.batchsize);
                break;
            case 1:
                /* BST */
                getBST( configs, gpus,
                        workspace->obj,
                        workspace->tomo,
                        workspace->angles,
                        configs.tomo.batchsize,
                        configs.tomo.padbatchsize,
                        configs.obj.batchsize);
                break;
            case 2:
                /* EM RT eEM */
                get_eEM_RT( configs, gpus,
                        workspace->obj,
                        workspace->tomo,
                        workspace->angles,
                        configs.tomo.batchsize.z);
                break;
            case 3:
                /* EM RT tEM */
                // get_tEM_RT( configs, gpus,
                //             workspace->obj,
                //             workspace->tomo,
                //             workspace->flat,
                //             workspace->angles,
                //             process.tomobatch_size);
                break;
            case 4:
                /* EM RT eEM TV */
                break;
            case 5:
                /* EM RT tEM TV */
                break;
            case 6:
                /* EM FST eEM */
                break;
            case 7:
                /* EM FST eEM TV*/
                break;
            case 8:
                /* EM FST tEM TV*/
            case 9:
                /* FDK */
                break;
            case 10:
                /* EM Conical eEM*/
                break;
            case 11:
                /* EM Conical tEM*/
                break;
            case 12:
                /* EM Conical eEM TV*/
                break;
            case 13:
                /* EM Conical tEM TV*/
                break;
            default:
                printf("No reconstruction method selected. Finshing run... \n");
                exit(EXIT_SUCCESS);
                break;
        }
    }

}

