#include "hip/hip_runtime.h"
#include "common/configs.hpp"

extern "C"{
    int compute_GPU_blocksize(float nslices, const float total_required_mem_per_slice,
    bool using_fft, float GPU_MEMORY) 
    {
        const float empiric_const = using_fft? 2.0 : 1.0; // the GPU needs some free memory to perform the FFTs.
        const float epsilon = 0.0;       // how much free memory we want to leave, in GB.
        
        // the values permitted for blocksize are powers of two.
        int raw_blocksize; // biggest blocksize feasible, although not necessarily: 
            // 1) a power of two; and 
            // 2) not a divisor of nslices (i.e., nslices % raw_blocksize != 0).
        int blocksize_exp = 1; // to store which power of 2 will be used. 
        int blocksize;

        std::cout << "Calculating blocksize..." << std::endl;

        raw_blocksize = static_cast<int>(
            -epsilon + (GPU_MEMORY)/(BYTES_TO_GB*total_required_mem_per_slice) );
        raw_blocksize = raw_blocksize/empiric_const;
        
        std::cout << "\t  Raw blocksize: " << raw_blocksize << std::endl;

        if (nslices < raw_blocksize) {
            blocksize = nslices;
        } else {
            while (raw_blocksize >> blocksize_exp) {
                blocksize_exp++;
            }
            blocksize_exp--;
            blocksize = 1 << blocksize_exp;
        }
        std::cout << "\t  Blocksize: " << blocksize << std::endl;
        return blocksize;
    }
}

extern "C"{
	Process *setProcesses(CFG configs, GPU gpus, int total_number_of_processes)
	{  
        int process_index;

        Process *process = (Process *)malloc(sizeof(Process) * total_number_of_processes);

        switch (configs.geometry.geometry){
            case 0:
                /* Parallel */
                for (process_index = 0; process_index < total_number_of_processes; process_index++)
                    setProcessParallel(configs, process, gpus, process_index, total_number_of_processes);
                break;
            case 1:
                /* Conebeam */
                for (process_index = 0; process_index < total_number_of_processes; process_index++)
                    setProcessConebeam(configs, process, gpus, process_index, total_number_of_processes);
                break;
            case 2:
                /* Fanbeam - the process division for Fanbeam geometry is the same as the Parallel one */
               for (process_index = 0; process_index < total_number_of_processes; process_index++)
                    setProcessParallel(configs, process, gpus, process_index, total_number_of_processes);
                break;
            default:
                printf("Nope.");
                break;
        }	

        return process; 
                
	}
}

extern "C"{
    void setProcessParallel(CFG configs, Process* process, GPU gpus, int index, int n_total_processes)
    {   
        /* Processes to parallelize the data z-axis by independent blocks */
        /* Declare variables */
        long long int  n_obj, n_tomo, ind_obj, ind_tomo;
        int ind, ind_max, block;  

        /* Set indexes */
        block    = (int) ( configs.tomo.size.z / n_total_processes ); 

        ind      = index * block;

        ind_max  = (int) std::min( ( index + 1 ) * block, (int)configs.tomo.size.z);

        /* Indexes for Reconstruction division - same as Tomogram division */
        n_obj    = (long long int) ( ind_max - ind ) * configs.obj.size.x * configs.obj.size.y;
        ind_obj  = (long long int)             ind   * configs.obj.size.x * configs.obj.size.y;

        /* Indexes for Tomogram division - same as Reconstruction division */
        n_tomo   = (long long int) ( ind_max - ind ) * configs.tomo.size.x * configs.tomo.size.y;
        ind_tomo = (long long int)             ind   * configs.tomo.size.x * configs.tomo.size.y;

        /* Set process struct */
        (*process).index          = index;
        (*process).index_gpu      = (int)gpus.gpus[index % gpus.ngpus]; 
        (*process).batch_index    = (int)index % gpus.ngpus;
        (*process).tomobatch_size = (int)( ind_max - ind );
        (*process).objbatch_size  = (int)( ind_max - ind );

        /* Tomogram division */
        (*process).tomo_index_z   = ind;
        (*process).tomoptr_index  = ind_tomo;
        (*process).tomoptr_size   = n_tomo;

        /* Reconstruction division */
        (*process).objptr_size    = n_obj;
        (*process).objptr_index   = ind_obj;
        
    }
}


extern "C"{
    void setProcessConebeam(CFG configs, Process* process, GPU gpus, int index, int n_total_processes)
    {   
        /* Processes to parallelize the data z-axis by independent blocks */
        
        /* Declare variables */
        int block;  

        /* Reconstruction */
        long long int n_obj, ind_obj;
        int indz, indz_max; 
        float posz, posz_max;
        
        /* Tomogram and reconstruction filter (v of vertical) */
        long long int n_tomo, ind_tomo, n_filter, ind_filter;
        int ind, ind_max, indz_filter;
        float pos, pos_max, lenght;

        /* Reconstruction (or object) variables */ 
        block    = (int) ( ( configs.obj.zslice1 - configs.obj.zslice0 ) / n_total_processes ); 
        
        indz        = configs.obj.zslice0 + index * block;
        
        indz_max    = (int) std::min( configs.obj.zslice0 + ( index + 1 ) * block, configs.obj.zslice1 ); 

        n_obj     = (long long int) ( indz_max -                indz ) * configs.obj.size.x * configs.obj.size.y;
        ind_obj   = (long long int) ( indz     - configs.obj.zslice0 ) * configs.obj.size.x * configs.obj.size.y;

        posz           = - configs.obj.Lz +     indz * configs.obj.dz;
        posz_max       = - configs.obj.Lz + indz_max * configs.obj.dz;
                
        lenght      = sqrtf( configs.obj.Lx * configs.obj.Lx + configs.obj.Ly * configs.obj.Ly );
        
        /* Tomogram (or detector) and filter (with padding) variables */
        float z12x  = configs.geometry.z1x + configs.geometry.z2x;
        // float z12y  = configs.geometry.z1y + configs.geometry.z2y;

        /* Tomogram */
        pos           = std::max(- configs.tomo.Lz, std::min( z12x * posz     / ( configs.geometry.z1x - lenght ), z12x *     posz / ( configs.geometry.z1x + lenght ) ) );
        pos_max       = std::min(+ configs.tomo.Lz, std::max( z12x * posz_max / ( configs.geometry.z1x + lenght ), z12x * posz_max / ( configs.geometry.z1x - lenght ) ) ); 

        ind        = std::max(                        0, (int) floor( ( pos     + configs.tomo.Lz ) / configs.tomo.dz ) );
        ind_max    = std::min( (int)configs.tomo.size.z, (int)  ceil( ( pos_max + configs.tomo.Lz ) / configs.tomo.dz ) );

        n_tomo      = (long long int) ( ind_max -                  ind ) * ( configs.tomo.size.x * configs.tomo.size.y );
        ind_tomo    = (long long int) ( ind     - configs.tomo.zslice0 ) * ( configs.tomo.size.x * configs.tomo.size.y );

        /* Set process struct */

        /* Reconstruction Filter */
        n_filter    = (long long int) ( ind_max -                 ind  ) * ( configs.tomo.padsize.x * configs.tomo.size.y );
        ind_filter  = (long long int) ( ind     - configs.tomo.zslice0 ) * ( configs.tomo.padsize.x * configs.tomo.size.y );
        indz_filter = (          int) (                       n_filter ) / ( configs.tomo.padsize.x * configs.tomo.size.y );

        /* Set process struct */
        (*process).index           = index;
        (*process).index_gpu       = (int)gpus.gpus[index % gpus.ngpus]; 
        (*process).batch_index     = (int)index % gpus.ngpus;
        (*process).tomobatch_size  = (int)(  ind_max -  ind );
        (*process).objbatch_size   = (int)( indz_max - indz );

        /* Tomogram division */
        (*process).tomo_index_z    = ind;
        (*process).tomoptr_index   = ind_tomo;
        (*process).tomoptr_size    = n_tomo;

        /* Reconstruction Filter division */
        (*process).filterptr_size  = n_filter;
        (*process).filterptr_index = ind_filter;
        (*process).filter_index_z  = indz_filter; 

        /* Reconstruction division */
        (*process).objptr_size     = n_obj;
        (*process).objptr_index    = ind_obj;
        (*process).obj_posz        = posz;
        (*process).tomo_posz       = - configs.tomo.Lz + ind * configs.tomo.dz;
    }
}

extern "C"{
    int getTotalProcesses(CFG configs, const float GPU_MEMORY, int sizeZ, bool using_fft)
    {
        int blocksizeMax = compute_GPU_blocksize((float)sizeZ, 
                                                configs.total_required_mem_per_slice,
                                                using_fft,
                                                GPU_MEMORY);   

        int n_total_processes = (int)( sizeZ / blocksizeMax );

        return n_total_processes;
    }
}