#include "hip/hip_runtime.h"
#include "../../../../inc/include.h"
#include "../../../../inc/common/types.hpp"
#include "../../../../inc/common/kernel_operators.hpp"
#include "../../../../inc/common/complex.hpp"
#include "../../../../inc/common/operations.hpp"
#include "../../../../inc/common/logerror.hpp"

# define vc 299792458           /* Velocity of Light [m/s] */ 
# define plank 4.135667662E-15  /* Plank constant [ev*s] */

extern "C" {

	void phase_filters(float *projections, float *paramf, size_t *parami, 
				int nrays, int nangles, int nslices,
				int *gpus, int ngpus)
	{	
		int i, Maxgpudev;
		
		/* Multiples devices */
		hipGetDeviceCount(&Maxgpudev);

		/* If devices input are larger than actual devices on GPU, exit */
		for(i = 0; i < ngpus; i++) 
			assert(gpus[i] < Maxgpudev && "Invalid device number.");

		PAR param;

		set_phase_filters_parameters(&param, paramf, parami, nrays, nslices, nangles);

		int subvolume = (nangles + ngpus - 1) / ngpus;
		int subblock, ptr = 0; 
		size_t ptr_volume = 0;

		if (ngpus == 1){ /* 1 device */

			_phase_filters_threads(param, projections, nrays, nangles, nslices, gpus[0]);

		}else{
		/* Launch async Threads for each device.
			Each device solves a block of 'nrays * nangles' size.
		*/
			// See future c++ async launch
			std::vector<std::future<void>> threads = {};

			for (i = 0; i < ngpus; i++){
				
				subblock   = min(nangles - ptr, subvolume);
				ptr_volume = (size_t)nrays * nslices * ptr;
				
				// printf("Sub[%d] = %d () \n",i,subblock,ptr);

				/* Update pointer */
				ptr = ptr + subblock;
				
				threads.push_back( std::async( std::launch::async, _phase_filters_threads, param, projections + ptr_volume, (size_t)nrays, (size_t)subblock, (size_t)nslices, gpus[i]));		

			}
		
			// Log("Synchronizing all threads...\n");
		
			for (i = 0; i < ngpus; i++)
				threads[i].get();
		}	

		hipDeviceSynchronize();
	}

	void _phase_filters_threads(PAR param, float *projections, size_t nrays, size_t nangles, size_t nslices, int ngpu)
	{	
		/* Initialize GPU device */
		HANDLE_ERROR(hipSetDevice(ngpu))

		size_t npad = param.Npadx * param.Npady;
		float *d_kernel;
		hipblasHandle_t handle = NULL;
        hipblasCreate(&handle);
        hipblasStatus_t stat;

		// Compute phase filter kernel
		HANDLE_ERROR(hipMalloc((void **)&d_kernel, sizeof(float) * npad ));

		switch ((int)param.filter){
				case 0:
					/* code */
					printf("No filter was selected!");
					break;
				case 1:
					/* code */
					paganinKernel<<<param.Grd,param.BT>>>(d_kernel, param, param.Npadx, param.Npady, nangles);
					break;
				case 2:
					/* code */
					bronnikovKernel<<<param.Grd,param.BT>>>(d_kernel, param, param.Npadx, param.Npady, nangles);
					break;
				case 3:
					/* code */
					bornKernel<<<param.Grd,param.BT>>>(d_kernel, param, param.Npadx, param.Npady, nangles);
					break;
				case 4:
					/* code */
					rytovKernel<<<param.Grd,param.BT>>>(d_kernel, param, param.Npadx, param.Npady, nangles);
					break;

				default:
					paganinKernel<<<param.Grd,param.BT>>>(d_kernel, param, param.Npadx, param.Npady, nangles);
					break;
			}

        // Normalize kernel by maximum value
 		int max;
        stat = hipblasIsamax(handle, (int)npad, d_kernel, 1, &max);

        if (stat != HIPBLAS_STATUS_SUCCESS)
            printf("Cublas Max failed\n");

		float maximum;
		HANDLE_ERROR(hipMemcpy(&maximum, d_kernel + max, sizeof(float), hipMemcpyDeviceToHost));
        Normalize<<<param.Grd,param.BT>>>(d_kernel, maximum, param.Npadx, param.Npady);

		hipDeviceSynchronize();

		size_t bz; 
		param.blocksize = min(nangles,param.blocksize);

		// printf("Filter: %d \n",(int)param.filter);
		// printf("Dims: %ld, %ld, %ld \n",param.Npadx,param.Npady,param.blocksize);
		// printf("Dims: %e, %e, %e, %e, %e, %e \n",param.z1x,param.z1y,param.z2x,param.z2y,param.energy,param.lambda);
		
		/* Plan for Fourier transform - cufft */
		int n[] = {(int)param.Npadx,(int)param.Npady};
		HANDLE_FFTERROR(hipfftPlanMany(&param.mplan, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, (int)param.blocksize));

		size_t zblock = param.blocksize;

		size_t ind_block = (size_t)ceil( (float) nangles / param.blocksize );
		int ptr = 0;
		size_t ptr_block = 0;

		/* Loop for each batch of size 'batch' in threads */

		// printf("GPU: %d; zblock = %ld; indBlock = %ld \n", ngpu, zblock, ind_block);

		for (bz = 0; bz < ind_block; bz++){

			zblock    = min(nangles - ptr, param.blocksize);
			ptr_block = (size_t)nrays * nslices * ptr;

			// printf("zblock[%d,%ld] = %ld, ptr = %d (%d) \n", ngpu, bz, zblock, ptr, nangles - ptr);

			/* Update pointer */
			ptr = ptr + zblock;

			if( zblock != param.blocksize){

				HANDLE_FFTERROR(hipfftDestroy(param.mplan));

				HANDLE_FFTERROR(hipfftPlanMany(&param.mplan, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, (int)zblock));
			}
				
			switch ((int)param.filter){
				case 0:
					/* code */
					printf("No filter was selected!");
					break;
				case 1:
					/* code */
					_paganin_gpu(param, projections + ptr_block, d_kernel, nrays, zblock, nslices);
					break;
				case 2:
					/* code */
					_bronnikov_gpu(param, projections + ptr_block, d_kernel, nrays, zblock, nslices);
					break;
				case 3:
					/* code */
					_born_gpu(param, projections + ptr_block, d_kernel, nrays, zblock, nslices);
					break;
				case 4:
					/* code */
					_rytov_gpu(param, projections + ptr_block, d_kernel, nrays, zblock, nslices);
					break;

				default:
					printf("Using paganin as default phase filter!");
					_paganin_gpu(param, projections + ptr_block, d_kernel, nrays, zblock, nslices);
					break;
			}	
		}
		hipDeviceSynchronize();

		/* Destroy plan */
		HANDLE_FFTERROR(hipfftDestroy(param.mplan));
		hipFree(d_kernel);
		hipblasDestroy(handle);

		hipDeviceSynchronize();
	}

	void set_phase_filters_parameters(PAR *param, float *paramf, size_t *parami, size_t sizex, size_t sizey, size_t sizez)
	{
		/* Initialize paganin parameters */
		param->z1x       = paramf[0]; // z1x;
		param->z1y       = paramf[1]; // z1y;
		param->z2x       = paramf[2]; // z2x;
		param->z2y       = paramf[3]; // z2y;
		param->energy    = paramf[4]; // energy;
		param->alpha     = paramf[5]; // alpha;		

		/* Dimensions */
		param->padx      = parami[0]; // padx;
		param->pady      = parami[1]; // pady;
		param->blocksize = parami[2]; // blocksize;

		param->filter    = (int)parami[3]; // filter type;

		param->lambda    = ( plank * vc ) / param->energy;
		param->wave      = ( 2.0 * float(M_PI) ) / param->lambda;

		param->Npadx     = sizex + 2.0 * param->padx; 
		param->Npady     = sizey + 2.0 * param->pady; 


		/* GPUs */
		/* Initialize Device sizes variables */
		size_t Nsx      = 16;
		size_t Nsy      = 16; 
		size_t Nsz      = 1;

		param->BT       = dim3(Nsx,Nsy,Nsz);
        const int bx    = ( param->Npadx + Nsx - 1 ) / Nsx;	
		const int by    = ( param->Npady + Nsy - 1 ) / Nsy;
		const int bz    = ( sizez        + Nsz - 1 ) / Nsz;
		param->Grd      = dim3(bx,by,bz);
	}

}

