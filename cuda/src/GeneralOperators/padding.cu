#include "hip/hip_runtime.h"
#include "../../inc/include.h"
#include "../../inc/common/types.hpp"
#include "../../inc/common/kernel_operators.hpp"
#include "../../inc/common/complex.hpp"
#include "../../inc/common/operations.hpp"
#include "../../inc/common/logerror.hpp"


extern "C"{

    __global__ void padding(float *in, hipfftComplex *inpadded, float value, size_t sizex, size_t sizey, size_t sizez, size_t padsizex, size_t padsizey)
    {
        size_t Npadx = sizex + 2 * padsizex;
        size_t Npady = sizey + 2 * padsizey;

        int i      = blockIdx.x*blockDim.x + threadIdx.x;
        int j      = blockIdx.y*blockDim.y + threadIdx.y;
        int k      = blockIdx.z*blockDim.z + threadIdx.z;

        int ii     = (int)( i - padsizex );
        int jj     = (int)( j - padsizey );

        long long int index  = sizex * k * sizey + sizex * jj + ii;
        long long int indpad = Npadx * k * Npady + Npadx *  j +  i;

        if ( (i >= Npadx) || (j >= Npady) || (k >= sizez) ) return;

        inpadded[indpad].x = value;
        inpadded[indpad].y = 0.0;

        if ( (ii < 0) || (ii >= sizex) || (jj < 0) || (jj >= sizey) || (k >= sizez) ) return;

        inpadded[indpad].x = in[index];

    }

    __global__ void recuperate_padding(hipfftComplex *inpadded, float *in, size_t sizex, size_t sizey, size_t sizez, size_t padsizex, size_t padsizey)
    {
        size_t Npadx = sizex + 2 * padsizex;
        size_t Npady = sizey + 2 * padsizey;

        int i      = blockIdx.x*blockDim.x + threadIdx.x;
        int j      = blockIdx.y*blockDim.y + threadIdx.y;
        int k      = blockIdx.z*blockDim.z + threadIdx.z;
        
        int ii     = (int)( i - padsizex );
        int jj     = (int)( j - padsizey );

        long long int index  = sizex * k * sizey + sizex * jj + ii;
        long long int indpad = Npadx * k * Npady + Npadx *  j +  i;

        if ( (ii < 0) || (ii >= sizex) || (jj < 0) || (jj >= sizey) || (k >= sizez) ) return;

        in[index] = inpadded[indpad].x;
        
    }

}