#include "hip/hip_runtime.h"
#include "../../inc/include.h"
#include "../../inc/common/kernel_operators.hpp"
#include "../../inc/common/complex.hpp"
#include "../../inc/common/types.hpp"
#include "../../inc/common/operations.hpp"
#include "../../inc/common/logerror.hpp"


extern "C"{
   static __global__ void KFlatDarkTransposelog(float* in, float* dark, float* flat, dim3 size, int numflats)
	{  
      // Supports 2 flats only
		size_t idx = threadIdx.x + blockIdx.x*blockDim.x;
		
		if(idx < size.x && blockIdx.y < size.y)
		{
			float dk = dark[blockIdx.y * size.x + idx];
			float ft = flat[blockIdx.y * size.x + idx];
			
         if(numflats > 1){
				float interp = float(blockIdx.z+1)/float(size.z+1);
				ft = ft*(1.0f-interp) + interp*(float)flat[size.x*size.y + blockIdx.y * size.x + idx];
			}

			size_t line = size.y*blockIdx.z + blockIdx.y;
			// size_t coll = blockIdx.y * size.z + blockIdx.z;
			
			// out[coll * size.x + idx] = -log( fmaxf(in[line * size.x + idx] - dk, 0.5f) / fmaxf(ft-dk,0.5f) );
			in[line * size.x + idx] = -log( fmaxf(in[line * size.x + idx] - dk, 0.5f) / fmaxf(ft-dk,0.5f) );

		}
	}

	static __global__ void KFlatDarkTranspose(float* in, float* dark, float* flat, dim3 size, int numflats)
	{  
      // Supports 2 flats only
		size_t idx = threadIdx.x + blockIdx.x*blockDim.x;
		
		if(idx < size.x && blockIdx.y < size.y)
		{
			float dk = dark[blockIdx.y * size.x + idx];
			float ft = flat[blockIdx.y * size.x + idx];
			
         if(numflats > 1){
				float interp = float(blockIdx.z+1)/float(size.z+1);
				ft = ft*(1.0f-interp) + interp*(float)flat[size.x*size.y + blockIdx.y * size.x + idx];
			}

			size_t line = size.y*blockIdx.z + blockIdx.y;
			// size_t coll = blockIdx.y * size.z + blockIdx.z;
			
			// out[coll * size.x + idx] = -log( fmaxf(in[line * size.x + idx] - dk, 0.5f) / fmaxf(ft-dk,0.5f) );
			in[line * size.x + idx] = -log( fmaxf(in[line * size.x + idx] - dk, 0.5f) / fmaxf(ft-dk,0.5f) );

		}
	}

	void flatdarktranspose_log_gpu(int gpu, float* frames, float* flat, float* dark, int nrays, int nslices, int nangles, int numflats)
	{	
      // Supports 2 flats max
      hipSetDevice(gpu);

		int b;
      int blocksize = min(nslices,32);

      dim3 blocks = dim3(nrays,blocksize,nangles);
		blocks.x = (nrays+127) / 128;

		rImage data(nrays,blocksize,nangles);
		Image2D<float> cflat(nrays, blocksize, numflats);
		Image2D<float> cdark(nrays, blocksize);

      for(b = 0; b < nslices; b += blocksize){
			blocksize = min(blocksize,nslices-b);

         data.CopyFrom(frames + (size_t)b*nrays*nangles, 0, (size_t)nrays*nangles*blocksize);
			cflat.CopyFrom(flat + (size_t)b*nrays*numflats, 0, (size_t)nrays*numflats*blocksize);
			cdark.CopyFrom(dark + (size_t)b*nrays, 0, (size_t)nrays*blocksize);

         KFlatDarkTransposelog<<<blocks,128>>>(data.gpuptr, cdark.gpuptr, cflat.gpuptr, dim3(nrays,blocksize,nangles), cflat.sizez);

         data.CopyTo(frames + (size_t)b*nrays*nangles, 0, (size_t)nrays*nangles*blocksize);
      
      }

		hipDeviceSynchronize();
	}

	void flatdarktranspose_gpu(int gpu, float* frames, float* flat, float* dark, int nrays, int nslices, int nangles, int numflats)
	{	
      // Supports 2 flats max
      hipSetDevice(gpu);

		int b;
      int blocksize = min(nslices,32);

      dim3 blocks = dim3(nrays,blocksize,nangles);
		blocks.x = (nrays+127) / 128;

		rImage data(nrays,blocksize,nangles);
		Image2D<float> cflat(nrays, blocksize, numflats);
		Image2D<float> cdark(nrays, blocksize);

      for(b = 0; b < nslices; b += blocksize){
			blocksize = min(blocksize,nslices-b);

         data.CopyFrom(frames + (size_t)b*nrays*nangles, 0, (size_t)nrays*nangles*blocksize);
			cflat.CopyFrom(flat + (size_t)b*nrays*numflats, 0, (size_t)nrays*numflats*blocksize);
			cdark.CopyFrom(dark + (size_t)b*nrays, 0, (size_t)nrays*blocksize);

         KFlatDarkTranspose<<<blocks,128>>>(data.gpuptr, cdark.gpuptr, cflat.gpuptr, dim3(nrays,blocksize,nangles), cflat.sizez);

         data.CopyTo(frames + (size_t)b*nrays*nangles, 0, (size_t)nrays*nangles*blocksize);
      
      }

		hipDeviceSynchronize();
	}

	void flatdarktranspose_log_block(int* gpus, int ngpus, float* frames, float* flat, float* dark, int nrays, int nslices, int nangles, int numflats)
	{
		int t;
		int blockgpu = (nslices + ngpus - 1) / ngpus;
		
		// printf("Aqui\n");
		std::vector<std::future<void>> threads;

		// printf("Aqui2\n");

		// printf("valores cu: %d, %d,%d,%d %d %d\n",ngpus,nrays,nangles,nslices,numflats,blockgpu);

		for(t = 0; t < ngpus; t++){ 
			
			blockgpu = min(nslices - blockgpu * t, blockgpu);
			// printf("Aqui3 %d %d %ld %d %d\n",t,blockgpu,(size_t)t*blockgpu*nrays*nangles,nrays,nangles);

			threads.push_back(std::async( std::launch::async, flatdarktranspose_log_gpu, gpus[t], frames + (size_t)t*blockgpu*nrays*nangles, 
						flat + (size_t)t*blockgpu*nrays*numflats, dark + (size_t)t*blockgpu*nrays, nrays, blockgpu, nangles, numflats
						));
		}

		for(auto& t : threads)
			t.get();
	}

	void flatdarktranspose_block(int* gpus, int ngpus, float* frames, float* flat, float* dark, int nrays, int nslices, int nangles, int numflats)
	{
		int t;
		int blockgpu = (nslices + ngpus - 1) / ngpus;
		
		// printf("Aqui\n");
		std::vector<std::future<void>> threads;

		// printf("Aqui2\n");

		// printf("valores cu: %d, %d,%d,%d %d %d\n",ngpus,nrays,nangles,nslices,numflats,blockgpu);

		for(t = 0; t < ngpus; t++){ 
			
			blockgpu = min(nslices - blockgpu * t, blockgpu);
			// printf("Aqui3 %d %d %ld %d %d\n",t,blockgpu,(size_t)t*blockgpu*nrays*nangles,nrays,nangles);

			threads.push_back(std::async( std::launch::async, flatdarktranspose_gpu, gpus[t], frames + (size_t)t*blockgpu*nrays*nangles, 
						flat + (size_t)t*blockgpu*nrays*numflats, dark + (size_t)t*blockgpu*nrays, nrays, blockgpu, nangles, numflats
						));
		}

		for(auto& t : threads)
			t.get();
	}

	void _CPUReduceBLock16(float* out, uint16_t* frames, uint16_t* cflat, uint16_t* cdark, 
		size_t sizex, size_t sizey, size_t sizez, size_t block, int numflats, int tidx, int nthreads)
	{
		for(size_t z = (size_t)tidx; z < sizez; z += (size_t)nthreads)
		{
			for(size_t by = 0; by < block; by++)
			{
				for(size_t x = 0; x < sizex; x++)
				{
					size_t step = sizey/block;
					float val = 0;

					for(size_t fy = 0; fy < step; fy++)
					{
						float flat = cflat[(by*step+fy)*sizex + x];
						float dark = cdark[(by*step+fy)*sizex + x];

						if(numflats>1)
						{
							float interp = float(z+1)/float(sizez+1);
							flat = flat*(1.0f-interp) + interp*cflat[sizex*sizey + (by*step+fy)*sizex + x];
						}

						val += -logf( fmaxf(frames[z*sizex*sizey + (by*step+fy)*sizex + x]-dark,0.5f) / fmaxf(flat-dark,0.5f) );

					}
					out[by*sizez*sizex + sizex*z + x] = val;
				}
			}
		}
	}

	void CPUReduceBLock16(float* out, uint16_t* frames, uint16_t* cflat, uint16_t* cdark, 
	size_t sizex, size_t sizey, size_t sizez, size_t block, int numflats)
	{
		std::vector<std::future<void>> threads;
		for(int t=0; t<16; t++)
			threads.push_back( std::async(std::launch::async, _CPUReduceBLock16, out, frames, cflat, cdark, 
				sizex, sizey, sizez, block, numflats, t, 16) );

		for(int t=0; t<16; t++)
				threads[t].get();
	}
}

extern "C"{
	void flatdarkcpu(float* out, float* frames, float* cflat, float* cdark, 
		size_t sizex, size_t sizey, size_t sizez, size_t block, int numflats, int tidx, int nthreads)
	{
		for(size_t z = (size_t)tidx; z < sizez; z += (size_t)nthreads)
		{
			for(size_t by = 0; by < block; by++)
			{
				for(size_t x = 0; x < sizex; x++)
				{
					size_t step = sizey/block;
					float val = 0;

					for(size_t fy = 0; fy < step; fy++)
					{
						float flat = cflat[(by*step+fy)*sizex + x];
						float dark = cdark[(by*step+fy)*sizex + x];

						if(numflats>1)
						{
							float interp = float(z+1)/float(sizez+1);
							flat = flat*(1.0f-interp) + interp*cflat[sizex*sizey + (by*step+fy)*sizex + x];
						}

						val += fmaxf(frames[z*sizex*sizey + (by*step+fy)*sizex + x]-dark,0.5f) / fmaxf(flat-dark,0.5f);

					}
					out[by*sizez*sizex + sizex*z + x] = val;
				}
			}
		}
	}

	void flatdarkcpu_log(float* out, float* frames, float* cflat, float* cdark, 
		size_t sizex, size_t sizey, size_t sizez, size_t block, int numflats, int tidx, int nthreads)
	{
		for(size_t z = (size_t)tidx; z < sizez; z += (size_t)nthreads)
		{
			for(size_t by = 0; by < block; by++)
			{
				for(size_t x = 0; x < sizex; x++)
				{
					size_t step = sizey/block;
					float val = 0;

					for(size_t fy = 0; fy < step; fy++)
					{
						float flat = cflat[(by*step+fy)*sizex + x];
						float dark = cdark[(by*step+fy)*sizex + x];

						if(numflats>1)
						{
							float interp = float(z+1)/float(sizez+1);
							flat = flat*(1.0f-interp) + interp*cflat[sizex*sizey + (by*step+fy)*sizex + x];
						}

						val += -logf( fmaxf(frames[z*sizex*sizey + (by*step+fy)*sizex + x]-dark,0.5f) / fmaxf(flat-dark,0.5f) );

					}
					out[by*sizez*sizex + sizex*z + x] = val;
				}
			}
		}
	}

	void flatdarkcpu_block(float* out, float* frames, float* cflat, float* cdark, 
	size_t sizex, size_t sizey, size_t sizez, size_t block, int numflats)
	{
		std::vector<std::future<void>> threads;
		for(int t=0; t<16; t++)
			threads.push_back( std::async(std::launch::async, flatdarkcpu, out, frames, cflat, cdark, 
				sizex, sizey, sizez, block, numflats, t, 16) );

		for(int t=0; t<16; t++)
				threads[t].get();
	}

	void flatdarkcpu_log_block(float* out, float* frames, float* cflat, float* cdark, 
		size_t sizex, size_t sizey, size_t sizez, size_t block, int numflats)
		{
			std::vector<std::future<void>> threads;
			for(int t=0; t<16; t++)
				threads.push_back( std::async(std::launch::async, flatdarkcpu_log, out, frames, cflat, cdark, 
					sizex, sizey, sizez, block, numflats, t, 16) );
	
			for(int t=0; t<16; t++)
					threads[t].get();
		}
}