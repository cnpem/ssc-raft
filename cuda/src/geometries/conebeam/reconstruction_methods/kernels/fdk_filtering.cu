#include "hip/hip_runtime.h"
#include "include.hpp"
#include "geometries/conebeam/fdk.hpp"
#include "common/types.hpp"

extern "C"{
void copy_gpu_filter_fft(Lab lab, float* proj, float** c_proj, hipfftComplex** c_signal, float** W, Process process) {
    // long long int N = process.n_filter;
    long long int Npad = process.n_filter_pad;

    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    // printf(hipGetErrorString(hipGetLastError()));
    // printf("\n");

    //hipMalloc(c_signal, sizeof(hipfftComplex)*N);
    hipMalloc(c_signal, sizeof(hipfftComplex)*Npad);

    hipMalloc(c_proj, process.n_filter * sizeof(float));    
    hipMemcpy(*c_proj, &proj[process.idx_filter], process.n_filter * sizeof(float), hipMemcpyHostToDevice);

    // hipMalloc(W, lab.nh * sizeof(float));
    hipMalloc(W, (lab.nph) * sizeof(float));

    printf("Filter number: %d \n",lab.filter_type);
    switch (lab.filter_type){
        case 0:
            // No filter Applied
            break;
        case 1:
            // Gaussian
            filt_Gaussian<<< 1, 1>>>(lab, *W);
            break;
        case 2:
            // Lorentz
            filt_Lorentz<<< 1, 1>>>(lab, *W);
            break;
        case 3:
            // Cosine
            filt_Cosine<<< 1, 1>>>(lab, *W);
            break;
        case 4:
            // Rectangle
            filt_Rectangle<<< 1, 1>>>(lab, *W);
            break;
        case 5:
            // Hann
            filt_Hann<<< 1, 1>>>(lab, *W);
            break;
        case 6:
            // Hamming
            filt_Hamming<<< 1, 1>>>(lab, *W);
            break;
        case 7:
            // Ramp
            filt_Ramp<<< 1, 1>>>(lab, *W);
            break;
        default:
            // Ramp
            filt_Ramp<<< 1, 1>>>(lab, *W);

    }

    // Normalize kernel by maximum value
    // hipblasHandle_t handle = NULL;
    // hipblasCreate(&handle);
    // hipblasStatus_t stat;
    // int max;

    // int n_threads = NUM_THREADS;
    // long long int n_blocks  = N/n_threads + (N % n_threads == 0 ? 0:1);

    // stat = hipblasIsamax(handle, lab.nph, *W, 1, &max);

    // if (stat != HIPBLAS_STATUS_SUCCESS)
    //     printf("Cublas Max failed\n");

    // float maximum;
    // HANDLE_ERROR(hipMemcpy(&maximum, *W + max, sizeof(float), hipMemcpyDeviceToHost));
    // Normalize<<<n_blocks, n_threads>>>(*W, maximum, lab.nph, 1);

    hipDeviceSynchronize(); 
    // printf(hipGetErrorString(hipGetLastError()));
    // printf("\n");

 
    printf("GPU memory allocated...\n");

    clock_t end = clock();
    printf("Time copy_to_gpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);

    // hipblasDestroy(handle);
}}

extern "C"{
void copy_cpu_filter_fft(float* proj, float* c_proj, hipfftComplex* c_signal, float* c_W,  Process process) {
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    // printf(hipGetErrorString(hipGetLastError()));
    // printf("\n");

    long long int N = process.n_filter;   
    hipMemcpy(&proj[process.idx_filter], c_proj, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(c_proj);
    hipFree(c_signal);
    hipFree(c_W);

    clock_t end = clock();
    printf("Time copy_to_cpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_gpu_filter_conv(Lab lab, float* proj, float** c_proj, float** c_Q, Process process) {
    long long int N = process.n_filter;

    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    // printf(hipGetErrorString(hipGetLastError()));
    // printf("\n");

    hipMalloc(c_Q, sizeof(float)*N);

    hipMalloc(c_proj, process.n_filter * sizeof(float));    
    hipMemcpy(*c_proj, &proj[process.idx_filter], process.n_filter * sizeof(float), hipMemcpyHostToDevice);
 
    printf("GPU memory allocated...\n");

    hipDeviceSynchronize(); 
    // printf(hipGetErrorString(hipGetLastError()));
    // printf("\n");


    clock_t end = clock();
    printf("Time copy_to_gpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_cpu_filter_conv(float* proj, float* c_proj, float* c_Q, Process process) {
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    // printf(hipGetErrorString(hipGetLastError()));
    // printf("\n");

    long long int N = process.n_filter;   //lab.nbeta * lab.nv * lab.nh;
    hipMemcpy(&proj[process.idx_filter], c_Q, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(c_proj);
    hipFree(c_Q);

    clock_t end = clock();
    printf("Time copy_to_cpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}
