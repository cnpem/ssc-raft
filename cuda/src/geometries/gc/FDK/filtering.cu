#include "hip/hip_runtime.h"
#include "../../../../inc/include.h"
#include "../../../../inc/gp/rings.h"
#include "../../../../inc/gc/fdk.h"
#include "../../../../inc/common/types.hpp"



extern "C"{
void copy_gpu_filter_fft(Lab lab, float* proj, float** c_proj, hipfftComplex** c_signal, float** W, Process process) {
    long long int N = process.n_filter;

    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    hipMalloc(c_signal, sizeof(hipfftComplex)*N);

    hipMalloc(c_proj, process.n_filter * sizeof(float));    
    hipMemcpy(*c_proj, &proj[process.idx_filter], process.n_filter * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(W, lab.nh * sizeof(float));
    filt_W<<< 1, 1>>>(lab, *W);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

 
    printf("GPU memory allocated...\n");

    clock_t end = clock();
    printf("Time copy_to_gpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_cpu_filter_fft(float* proj, float* c_proj, hipfftComplex* c_signal, float* c_W,  Process process) {
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    long long int N = process.n_filter;                                         //lab.nbeta * lab.nv * lab.nh;
    hipMemcpy(&proj[process.idx_filter], c_proj, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(c_proj);
    hipFree(c_signal);
    hipFree(c_W);

    clock_t end = clock();
    printf("Time copy_to_cpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_gpu_filter_conv(Lab lab, float* proj, float** c_proj, float** c_Q, Process process) {
    long long int N = process.n_filter;

    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    hipMalloc(c_Q, sizeof(float)*N);

    hipMalloc(c_proj, process.n_filter * sizeof(float));    
    hipMemcpy(*c_proj, &proj[process.idx_filter], process.n_filter * sizeof(float), hipMemcpyHostToDevice);
 
    printf("GPU memory allocated...\n");

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");


    clock_t end = clock();
    printf("Time copy_to_gpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_cpu_filter_conv(float* proj, float* c_proj, float* c_Q, Process process) {
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    long long int N = process.n_filter;                                         //lab.nbeta * lab.nv * lab.nh;
    hipMemcpy(&proj[process.idx_filter], c_Q, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(c_proj);
    hipFree(c_Q);

    clock_t end = clock();
    printf("Time copy_to_cpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}
