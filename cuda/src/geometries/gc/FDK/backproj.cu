#include "hip/hip_runtime.h"
#include "../../../../inc/geometries/gc/fdk.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>

extern "C"{
__global__ void backproj(float* recon, float* proj, float* beta, Lab lab, Process process){

    long long int n = blockDim.x * blockIdx.x + threadIdx.x ;
    long long int idx;
    int i, j, k, m;

    float    x, y, z;
    float u, v, X, Z;
    float cosb, sinb, Q, L;

    int xi, zk;

    set_recon_idxs(n, &i, &j, &k, lab);
    x = -lab.x + i*lab.dx;
    y = -lab.y + j*lab.dy;
    z = process.z_ph + k*lab.dz;
	
    recon[n] = 0.0;

    L = sqrtf(x*x + y*y);
    // if( L <= lab.x && L <= lab.y){
    for(m = 0; m < lab.nbeta; m++){

        cosb = beta[m];
        sinb = beta[m+lab.nbeta];

        u = x*cosb - y*sinb;
        v = x*sinb + y*cosb;

        X = + lab.Dsd*u/(lab.D + v);
        Z = + lab.Dsd*z/(lab.D + v);    

        xi = (int) ((X + lab.h)/lab.dh);
        zk = (int) ((Z - process.z_det)/lab.dv);
	
        if( xi < 0) continue;             
        if( xi >= lab.nh) continue; 
        if( zk < 0) continue;             
        if( zk + process.zi >= lab.nv) continue; 

        idx = (long long int) zk*lab.nbeta*lab.nh + m*lab.nh + xi; 

        Q = proj[idx];   
        recon[n] = recon[n] + Q*__powf(lab.Dsd/(lab.D + v), 2);
        // recon[n] = recon[n] + Q*__powf(lab.Dsd/(lab.D + x*sinb - y*cosb), 2);
    }
    recon[n] = recon[n]*lab.dbeta / 2.0;
    // }
}}

extern "C"{
void copy_to_gpu_back(Lab lab, float* proj, float* recon, float *angles, float** c_proj, float** c_recon, float** c_beta, Process process) {
    long long int N,M;	
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    N = process.n_recon;
    M = process.n_proj;     //lab.nx * lab.ny * lab.nz;

    float *dangles;
    hipMalloc((void **)&dangles, lab.nbeta * sizeof(float));
    hipMemcpy(dangles, angles, lab.nbeta * sizeof(float), hipMemcpyHostToDevice);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    printf("Allocating gpu memory...");
    hipMalloc(c_recon, N * sizeof(float));

    hipMalloc(c_proj, M * sizeof(float));     
    hipMemcpy(*c_proj, &proj[process.idx_proj], M * sizeof(float), hipMemcpyHostToDevice);
 
    printf("GPU memory allocated...\n");
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    hipMalloc(c_beta, 2* lab.nbeta * sizeof(float));
    set_beta<<< 1, 1 >>>(lab,dangles,*c_beta);

    hipFree(dangles);

    clock_t end = clock();
    printf("Time copy_to_gpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_to_cpu_back(float* recon, float* c_proj, float* c_recon, float* c_beta, Process process) {
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");


    long long int N = process.n_recon;    //lab.nbeta * lab.nv * lab.nh;
    hipMemcpy(&recon[process.idx_recon], c_recon, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(c_proj);
    hipFree(c_recon);
    hipFree(c_beta);
    clock_t end = clock();
    printf("Time copy_to_cpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void backprojection(Lab lab, float* recon, float* proj, float* beta,  Process process) {
    long long int M;	
    long int n_blocks;
    int n_threads;

    M = process.n_recon;   //lab.nx * lab.ny * lab.nz;
    
    n_threads = NUM_THREADS;
    n_blocks  = M/n_threads + (M % n_threads == 0 ? 0:1);   
    

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    hipSetDevice(process.i_gpu);

    printf("\n Starting Backprojection: GPU %d \n", process.i_gpu);

    clock_t b_begin = clock();

    backproj<<<n_blocks, n_threads>>>(recon, proj, beta, lab, process);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    clock_t b_end = clock();
    printf("Time backproj: Gpu %d ---- %f \n",process.i_gpu, double(b_end - b_begin)/CLOCKS_PER_SEC);
}}


extern "C"{
__global__ void set_beta(Lab lab, float *dangles, float* beta){

    for(int m = 0; m < lab.nbeta; m++){
        // beta[m] = cosf(lab.dbeta*m);
        // beta[m + lab.nbeta] = sinf(lab.dbeta*m);
        // printf("beta[%d] = %e \n",m,lab.dbeta*m);
        beta[m] = cosf(dangles[m]);
        beta[m + lab.nbeta] = sinf(dangles[m]);
    }
}}



extern "C"{
__device__ void set_recon_idxs(long long int n, int* i, int*j, int* k, Lab lab) {
    long int nij, rem_ij;
    nij = lab.nx*lab.ny;
    *k = (n) / nij;    
    rem_ij = (n) % nij;
    *j = rem_ij / lab.nx;
    *i = rem_ij % lab.nx;
}}
