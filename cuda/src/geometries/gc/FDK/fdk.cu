#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>
#include "../../../../inc/gc/fdk.h"
#include <fstream>
#include <future>
#include <thread>
#include <time.h>
#include <vector>
#include <iostream>

using namespace std;

int main(int argc, char *argv[]) {
    return 0;
}


extern "C"{
void gpu_fdk(   Lab lab, Rings_ ring, float *recon, float *proj, 
                int* gpus, int ndevs, double *time){	

    int i, k, n_process;
    
    n_process = memory(lab, ndevs);
    printf("n_process = %d, n_gpus = %d \n", n_process, ndevs);

    Process* process = (Process*) malloc(sizeof(Process)*n_process);
    for(i = 0; i < n_process; i++) set_process(lab, i, &process[i], n_process, gpus, ndevs);


    printf("Filter:\n");
    clock_t f_begin = clock();
    k = 0;
    std::vector<thread> threads_filt;

    float* c_filter[ndevs];
    hipfftComplex* c_signal[ndevs];
    float* c_W[ndevs];

    while(k < n_process){

        if(k % ndevs == 0){
            for(i = 0; i < ndevs; i++) 
                copy_gpu_filter(lab, proj, &c_filter[i], &c_signal[i], &c_W[i], process[k+i]);   
            hipDeviceSynchronize();
        }

        threads_filt.emplace_back(thread( filtering, ring, lab, c_filter[k%ndevs], c_signal[k%ndevs], c_W[k%ndevs], process[k])) ;
        k = k+1;

        if(k % ndevs == 0){
            for(i = 0; i < ndevs; i++) threads_filt[i].join();
            threads_filt.clear();
            hipDeviceSynchronize();

            for(i = 0; i < ndevs; i++) 
                copy_cpu_filter(proj, c_filter[i], c_signal[i], c_W[i], process[k-ndevs+i]);            
        }
    }

    clock_t f_end = clock();
    time[0] = double(f_end - f_begin)/CLOCKS_PER_SEC;


    printf("Backproject:\n");
    clock_t b_begin = clock();
    k = 0;
    std::vector<thread> threads_back;
    float* c_proj[ndevs];
    float* c_recon[ndevs];

    while(k < n_process){

        if(k % ndevs == 0){
            for(i = 0; i < ndevs; i++)
                copy_to_gpu_back(lab, proj, recon, &c_proj[i], &c_recon[i], process[k+i]);   
            hipDeviceSynchronize();    
        }

        threads_back.emplace_back(thread( backprojection, lab, c_recon[k%ndevs], c_proj[k%ndevs], process[k])) ;
        k = k+1;

        if(k % ndevs == 0){
            for(i = 0; i < ndevs; i++) threads_back[i].join();
            threads_back.clear();
            hipDeviceSynchronize();
            for(i = 0; i < ndevs; i++)
                copy_to_cpu_back(recon, c_proj[i], c_recon[i], process[k-ndevs+i]); 
        }
    }

    clock_t b_end = clock();
    time[1] = double(b_end - b_begin)/CLOCKS_PER_SEC;

	free(process);
}}

extern "C"{
void copy_to_gpu_back(Lab lab, float* proj, float* recon, float** c_proj, float** c_recon, Process process) {
    long long int N,M;	
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    N = process.n_recon;
    M = process.n_proj;                                        //lab.nx * lab.ny * lab.nz;

    printf("Allocating gpu memory...");
    hipMalloc(c_recon, N * sizeof(float));

    hipMalloc(c_proj, M * sizeof(float));     
    hipMemcpy(*c_proj, &proj[process.idx_proj], M * sizeof(float), hipMemcpyHostToDevice);
 
    printf("GPU memory allocated...\n");
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    clock_t end = clock();
    printf("Time copy_to_gpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_to_cpu_back(float* recon, float* c_proj, float* c_recon,  Process process) {
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    long long int N = process.n_recon;                                         //lab.nbeta * lab.nv * lab.nh;
    hipMemcpy(&recon[process.idx_recon], c_recon, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(c_proj);
    hipFree(c_recon);
    clock_t end = clock();
    printf("Time copy_to_cpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void backprojection(Lab lab, float* recon, float* proj, Process process) {
    long long int M;	
    long int n_blocks;
    int n_threads;

    M = process.n_recon;                                        //lab.nx * lab.ny * lab.nz;
    
    n_threads = NUM_THREADS;
    n_blocks  = M/n_threads + (M % n_threads == 0 ? 0:1);   
    
    hipSetDevice(process.i_gpu);

    clock_t b_begin = clock();

    backproj<<<n_blocks, n_threads>>>(recon, proj, lab, process);

    clock_t b_end = clock();
    printf("Time backproj: Gpu %d ---- %f \n",process.i_gpu, double(b_end - b_begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_gpu_filter(Lab lab, float* proj, float** c_proj, hipfftComplex** c_signal, float** W, Process process) {
    int n = lab.nh;
    long long int batch = process.z_filter*lab.nbeta;
    long long int N = process.n_filter;

    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipMalloc(c_signal, sizeof(hipfftComplex)*N);

    hipMalloc(c_proj, process.n_filter * sizeof(float));    
    hipMemcpy(*c_proj, &proj[process.idx_filter], process.n_filter * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(W, lab.nh * sizeof(float));
    filt_W<<<1, 1>>>(lab, *W);
 
    printf("GPU memory allocated...\n");

    clock_t end = clock();
    printf("Time copy_to_gpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_cpu_filter(float* proj, float* c_proj, hipfftComplex* c_signal, float* c_W,  Process process) {
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    hipDeviceSynchronize(); 
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    long long int N = process.n_filter;                                         //lab.nbeta * lab.nv * lab.nh;
    hipMemcpy(&proj[process.idx_filter], c_proj, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(c_proj);
    hipFree(c_signal);
    hipFree(c_W);

    clock_t end = clock();
    printf("Time copy_to_cpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}


void filtering(Rings_ ring, Lab lab, float* proj, hipfftComplex* signal, float* W, Process process)
{
    // rings
    ringsgpu_fdk(process.i_gpu, proj, ring.nrays, ring.nangles, process.zi_filter, rings.lambda_rings, rings.ringblocks);

    // fdk()
    fft(lab, proj, signal, W, process)

    return
}



