#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>
#include "../../../../inc/gc/fdk.h"
#include <fstream>
#include <future>
#include <thread>
#include <time.h>
#include <vector>
#include <iostream>

using namespace std;

int main(int argc, char *argv[]) {
    return 0;
}


extern "C"{
void gpu_fdk(   Lab lab, float *recon, float *proj, 
                int* gpus, int ndevs, double *time){	

    int i, k, n_process;
    
    n_process = memory(lab, ndevs);
    printf("n_process = %d, n_gpus = %d \n", n_process, ndevs);

    Process* process = (Process*) malloc(sizeof(Process)*n_process);
    for(i = 0; i < n_process; i++) set_process(lab, i, &process[i], n_process, gpus, ndevs);


    printf("Filter:\n");
    clock_t f_begin = clock();
    k = 0;
    std::vector<thread> threads_filt;

    float* c_filter[ndevs];
    hipfftComplex* c_signal[ndevs];
    float* c_W[ndevs];

    while(k < n_process){

        if(k % ndevs == 0){
            for(i = 0; i < ndevs; i++) 
                copy_gpu_filter(lab, proj, &c_filter[i], &c_signal[i], &c_W[i], process[k+i]);   
            hipDeviceSynchronize();
        }

        threads_filt.emplace_back(thread( fft, lab, c_filter[k%ndevs], c_signal[k%ndevs], c_W[k%ndevs], process[k])) ;
        k = k+1;

        if(k % ndevs == 0){
            for(i = 0; i < ndevs; i++) threads_filt[i].join();
            threads_filt.clear();
            hipDeviceSynchronize();

            for(i = 0; i < ndevs; i++) 
                copy_cpu_filter(proj, c_filter[i], c_signal[i], c_W[i], process[k-ndevs+i]);            
        }
    }

    clock_t f_end = clock();
    time[0] = double(f_end - f_begin)/CLOCKS_PER_SEC;


    printf("Backproject:\n");
    clock_t b_begin = clock();
    k = 0;
    std::vector<thread> threads_back;
    float* c_proj[ndevs];
    float* c_recon[ndevs];

    while(k < n_process){

        if(k % ndevs == 0){
            for(i = 0; i < ndevs; i++)
                copy_to_gpu_back(lab, proj, recon, &c_proj[i], &c_recon[i], process[k+i]);   
            hipDeviceSynchronize();    
        }

        threads_back.emplace_back(thread( backprojection, lab, c_recon[k%ndevs], c_proj[k%ndevs], process[k])) ;
        k = k+1;

        if(k % ndevs == 0){
            for(i = 0; i < ndevs; i++) threads_back[i].join();
            threads_back.clear();
            hipDeviceSynchronize();
            for(i = 0; i < ndevs; i++)
                copy_to_cpu_back(recon, c_proj[i], c_recon[i], process[k-ndevs+i]); 
        }
    }

    clock_t b_end = clock();
    time[1] = double(b_end - b_begin)/CLOCKS_PER_SEC;

	free(process);
}}

extern "C"{
void copy_to_gpu_back(Lab lab, float* proj, float* recon, float** c_proj, float** c_recon, Process process) {
    long long int N,M;	
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    N = process.n_recon;
    M = process.n_proj;                                        //lab.nx * lab.ny * lab.nz;

    printf("Allocating gpu memory...");
    hipMalloc(c_recon, N * sizeof(float));

    hipMalloc(c_proj, M * sizeof(float));     
    hipMemcpy(*c_proj, &proj[process.idx_proj], M * sizeof(float), hipMemcpyHostToDevice);
 
    printf("GPU memory allocated...\n");
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    clock_t end = clock();
    printf("Time copy_to_gpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void copy_to_cpu_back(float* recon, float* c_proj, float* c_recon,  Process process) {
    clock_t begin = clock();
    hipSetDevice(process.i_gpu);

    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");

    long long int N = process.n_recon;                                         //lab.nbeta * lab.nv * lab.nh;
    hipMemcpy(&recon[process.idx_recon], c_recon, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(c_proj);
    hipFree(c_recon);
    clock_t end = clock();
    printf("Time copy_to_cpu: Gpu %d ---- %f \n",process.i, double(end - begin)/CLOCKS_PER_SEC);
}}

extern "C"{
void backprojection(Lab lab, float* recon, float* proj, Process process) {
    long long int M;	
    long int n_blocks;
    int n_threads;

    M = process.n_recon;                                        //lab.nx * lab.ny * lab.nz;
    
    n_threads = NUM_THREADS;
    n_blocks  = M/n_threads + (M % n_threads == 0 ? 0:1);   
    
    hipSetDevice(process.i_gpu);

    clock_t b_begin = clock();

    backproj<<<n_blocks, n_threads>>>(recon, proj, lab, process);

    clock_t b_end = clock();
    printf("Time backproj: Gpu %d ---- %f \n",process.i_gpu, double(b_end - b_begin)/CLOCKS_PER_SEC);
}}





