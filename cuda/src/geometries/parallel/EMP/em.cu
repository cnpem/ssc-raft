#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>
// #include <stdio.h>
// #include <stdlib.h>
// #include <math.h>
// #include <hipblas.h>

#include "../../../../inc/sscraft.h"

#define PI 3.141592653589793238462643383279502884

#define TPBX 16
#define TPBY 16
#define TPBZ 4
#define TPBE 256

#define SQR(x) ((x)*(x))
#define SIGN(x) ((x > 0) ? 1 : ((x < 0) ? -1 : 0))
#define APPROXINVX(x,e) ((SIGN(x))/(sqrtf( SQR(e) + SQR(x) )))

extern "C" {
  __global__ void kernel_ones(float *output, int sizeImage, int nrays, int nangles,  int blockSize)
  {
    int tx = threadIdx.x + blockIdx.x*blockDim.x; 
    int ty = threadIdx.y + blockIdx.y*blockDim.y; 
    int tz = threadIdx.z + blockIdx.z*blockDim.z;
  
    if ( (tx<sizeImage) && (ty < sizeImage) && (tz<blockSize)  )
      {
	      int voxel = tz * sizeImage * sizeImage + ty * sizeImage + tx;
	
       	output[voxel] = 1;
      }
  }
}

extern "C" {
  __global__ void kernel_flatTimesExp(float *tmp, float *flat,
				      int sizeImage, int nrays, int nangles,  int blockSize)
  {
    int tx = threadIdx.x + blockIdx.x*blockDim.x; 
    int ty = threadIdx.y + blockIdx.y*blockDim.y; 
    int tz = threadIdx.z + blockIdx.z*blockDim.z;
  
    if ( (tx<nrays) && (ty < nangles) && (tz<blockSize)  )
      {
	      int voxel = tz * nrays * nangles + ty * nrays + tx;
	
       	tmp[voxel] = flat[voxel] * expf( - tmp[voxel]);	
      }
  }
}
    
extern "C" {
  __global__ void kernel_update(float *output, float *back, float *backcounts,
				int sizeImage, int nrays, int nangles,  int blockSize)
  {
    int tx = threadIdx.x + blockIdx.x*blockDim.x; 
    int ty = threadIdx.y + blockIdx.y*blockDim.y; 
    int tz = threadIdx.z + blockIdx.z*blockDim.z;
  
    if ( (tx<sizeImage) && (ty < sizeImage) && (tz<blockSize)  )
      {
	      int voxel = tz * sizeImage * sizeImage + ty * sizeImage + tx;
	
       	output[voxel] = output[voxel] * back[voxel] / backcounts[voxel];	
      }
  }
}


extern "C" {
  // difference in the Feature domain: square images of order 'sizeImage'
  // inplace: y = y - x
  
  __global__ void kernel_difference_F(float *y, float *x,
				      int sizeImage, int blockSize)
  {
    int tx = threadIdx.x + blockIdx.x*blockDim.x;
    int ty = threadIdx.y + blockIdx.y*blockDim.y; 
    int tz = threadIdx.z + blockIdx.z*blockDim.z;
  
    if ( (tx<sizeImage) && (ty < sizeImage) && (tz<blockSize)  )
      {
	      int voxel = tz * sizeImage * sizeImage + ty * sizeImage + tx;

	      // inplace!
       	y[voxel] = y[voxel] - x[voxel];
      }
  }
}


extern "C" {
  __global__ void kernel_backprojection(float *image, float *blocksino, float *angles,
					int sizeImage, int nrays, int nangles,  int blockSize)
  {
    int i, j, k, T, z;
    float t, cs, x, y, cosk, sink;
    float xymin = -1.0;
    float dxy = 2.0 / (sizeImage - 1);
    float dt = 2.0 / (nrays - 1);
    // float dth = PI / nangles;
    float dth;
    float tmin = -1.0;
    
    i = (blockDim.x * blockIdx.x + threadIdx.x);
    j = (blockDim.y * blockIdx.y + threadIdx.y);
    z = (blockDim.z * blockIdx.z + threadIdx.z);
    
    // printf("nangles = %d \n",nangles);

    if ( (i<sizeImage) && (j < sizeImage) && (z<blockSize)  ){
    
      cs = 0;
    
      x = xymin + i * dxy;
      y = xymin + j * dxy;

      for(k=0; k < (nangles); k++)
      {
        // __sincosf(k * dth, &sink, &cosk);
        __sincosf(angles[k], &sink, &cosk);

        if ( k == (nangles - 1) ){
          dth = angles[k] - angles[k-1];
        }else{
          dth = angles[k+1] - angles[k];
        }
        
        // printf("dth[%d] = %e \n",k,dth);

        t = x * cosk + y * sink;
      
        T = (int) ((t - tmin)/dt);	     

        if ( (T > -1) && (T<nrays) )
          {
            cs += blocksino[ z * nrays * nangles + k * nrays + T] * dth;
          }
      }
      image[z * sizeImage * sizeImage + j * sizeImage + i]  = cs; 
    }
  }
}

extern "C" {
  __global__ void kernel_radon(float *output, float *input, float *angles,
			       int sizeImage, int nrays, int nangles,
			       int blockSize, float a)
  {
    int tx = threadIdx.x + blockIdx.x*blockDim.x; 
    int ty = threadIdx.y + blockIdx.y*blockDim.y; 
    int tz = threadIdx.z + blockIdx.z*blockDim.z;
    

    if ( (tx<nrays) && (ty < nangles) && (tz<blockSize)  ){
 
      int k, X, Y;

      float s, x, y, linesum, ctheta, stheta, t;  
      float dt = 2.0*a/(nrays-1);
      // float dtheta = PI/(nangles-1);
 
      // theta = ty * dtheta;
      ctheta =cosf(angles[ty]);
      stheta =sinf(angles[ty]);
      
      t = - a + tx * dt; 
      
      linesum = 0;
      for( k = 0; k < nrays; k++ ) {

        s = - a + k * dt;
        x = t * ctheta - s * stheta;
        y = t * stheta + s * ctheta;
        X = (int) ((x + 1)/dt);
        Y = (int) ((y + 1)/dt);	 

        if ((X >= 0) & (X<sizeImage) & (Y>=0) & (Y<sizeImage) )
	        linesum += input[ tz * sizeImage * sizeImage + Y * sizeImage + X ];
      }

      output[tz * nrays * nangles + ty * nrays + tx] = linesum * dt;	
      // printf("out[%d] = %e \n",tz * nrays * nangles + ty * nrays + tx,output[tz * nrays * nangles + ty * nrays + tx]);
    }
  }
}

//---------------------------
// transmission-EM algorithm
//---------------------------

extern "C" {

  void tEM(float *output, float *count, float *flat, float *angles, 
	  int sizeImage, int nrays, int nangles, int blockSize, int device, int niter)
  {
    hipSetDevice(device);
    int k;
    float *d_output, *d_count, *d_flat, *d_backcounts, *d_temp, *d_back, *d_angles;
    
    // Allocate GPU memory for the output image
    hipMalloc(&d_output, sizeof(float) *sizeImage *sizeImage*blockSize);
    hipMalloc(&d_temp, sizeof(float)  * nrays * nangles*blockSize);
    hipMalloc(&d_back, sizeof(float) *sizeImage *sizeImage*blockSize);
    hipMalloc(&d_backcounts, sizeof(float) * sizeImage * sizeImage*blockSize);
    
    // Allocate GPU memory for input image and copy
    hipMalloc(&d_count, sizeof(float) * nrays * nangles*blockSize);
    hipMemcpy(d_count, count, sizeof(float) * nrays * nangles*blockSize, hipMemcpyHostToDevice);	

    hipMalloc(&d_flat, sizeof(float) * nrays * nangles*blockSize);
    hipMemcpy(d_flat, flat, sizeof(float) * nrays * nangles*blockSize, hipMemcpyHostToDevice);	

    hipMalloc(&d_angles, sizeof(float) * nangles);
    hipMemcpy(d_angles, angles, sizeof(float) * nangles, hipMemcpyHostToDevice);	

    // printf("Printando essa porra + %d, %d, %e\n",device, nangles, count[1024]);
    // for(int i=0; i < nangles; i++) printf("Angle[%d] = %e \n",i,angles[i]);

    //GRID and BLOCKS SIZE
    dim3 threadsPerBlock(TPBX,TPBY,TPBZ);
    dim3 gridBlockD((int)ceil((nrays)/threadsPerBlock.x)+1,
		   (int)ceil((nangles)/threadsPerBlock.y)+1,
		   (int)ceil(blockSize/threadsPerBlock.z)+1);

    dim3 gridBlockF((int)ceil((sizeImage)/threadsPerBlock.x)+1,
		   (int)ceil((sizeImage)/threadsPerBlock.y)+1,
		   (int)ceil(blockSize/threadsPerBlock.z)+1);

    
    kernel_ones<<<gridBlockF, threadsPerBlock>>>(d_output, sizeImage, nrays, nangles, blockSize);

    kernel_backprojection<<<gridBlockF, threadsPerBlock>>>(d_backcounts, d_count, d_angles, sizeImage, nrays, nangles, blockSize);
 
    for( k=0; k < niter; k++ )
      {
      // printf("Aqui iter = %d \n",k);
      kernel_radon<<<gridBlockD, threadsPerBlock>>>(d_temp, d_output, d_angles, sizeImage, nrays, nangles, blockSize, 1.0);
      
      kernel_flatTimesExp<<<gridBlockD, threadsPerBlock>>>(d_temp, d_flat, sizeImage, nrays, nangles, blockSize);
      
      kernel_backprojection<<<gridBlockF, threadsPerBlock>>>(d_back, d_temp, d_angles, sizeImage, nrays, nangles, blockSize);
      
      kernel_update<<<gridBlockF, threadsPerBlock>>>(d_output, d_back, d_backcounts, sizeImage, nrays, nangles, blockSize);
      
      hipDeviceSynchronize();
    }
    
    //Copy the output image from device memory to host memory
    hipMemcpy (output , d_output , blockSize*sizeImage*sizeImage*sizeof(float) , hipMemcpyDeviceToHost);
    
    // printf("Printando out + %d, %d, %e\n",device, nangles, output[1024]);

    hipFree(d_output);
    hipFree(d_temp);
    hipFree(d_back);
    hipFree(d_count);
    hipFree(d_flat);
    hipFree(d_backcounts);
    hipFree(d_angles);
    
    hipDeviceSynchronize();
    // hipDeviceReset();
    
  }
}

//----------------------
// emission-EM algorithm
//----------------------

extern "C" {
  __global__ void kernel_radonWithDivision(float *output, float *input, float *sino,  float *angles,
					   int sizeImage, int nrays, int nangles,
					   int blockSize, float a)
  {
    float TOLZERO;
    
    int tx = threadIdx.x + blockIdx.x*blockDim.x; 
    int ty = threadIdx.y + blockIdx.y*blockDim.y; 
    int tz = threadIdx.z + blockIdx.z*blockDim.z;
  
    if ( (tx<nrays) && (ty < nangles) && (tz<blockSize)  ){
 
      int k, X, Y, voxel;

      float s, x, y, linesum, ctheta, stheta, t;  
      float dt = 2.0*a/(nrays-1);
      // float dtheta = PI/(nangles-1);
      float value;
      
      // theta = ty * dtheta;
      ctheta =cosf(angles[ty]);
      stheta =sinf(angles[ty]);
      
      t = - a + tx * dt; 
      
      linesum = 0;
      for( k = 0; k < nrays; k++ ) {

        s = - a + k * dt;
        x = t * ctheta - s * stheta;
        y = t * stheta + s * ctheta;
        X = (int) ((x + 1)/dt);
        Y = (int) ((y + 1)/dt);	 

        if ((X >= 0) & (X<sizeImage) & (Y>=0) & (Y<sizeImage) )
          linesum += input[ tz * sizeImage * sizeImage + Y * sizeImage + X ];
      }

      value = linesum * dt;

      voxel = tz * nrays * nangles + ty * nrays + tx;

      TOLZERO = 0.0001;
      output[ voxel ] = sino[ voxel] * APPROXINVX(value, TOLZERO);  

      //enforcing positivity
      if (output[voxel] < 0)
	      output[voxel] = 0.0;

      
      /*
      if ( fabs(value) > TOLZERO ) 
	      output[ voxel ] = sino[voxel] / value;	
      else
	      output[ voxel ] = 0.0;
      */
    }
  }
}

extern "C" {
  __global__ void kernel_backprojectionWithUpdate(float *image, float *blocksino, float *backones, float *angles,
						  int sizeImage, int nrays, int nangles,  int blockSize)
  {
    int i, j, k, T, z, voxel;
    float t, cs, x, y, cosk, sink;
    float xymin = -1.0;
    float dxy = 2.0 / (sizeImage - 1);
    float dt = 2.0 / (nrays - 1);
    float dth; // = PI / nangles;
    float tmin = -1.0;
    float value; 
    
    i = (blockDim.x * blockIdx.x + threadIdx.x);
    j = (blockDim.y * blockIdx.y + threadIdx.y);
    z = (blockDim.z * blockIdx.z + threadIdx.z);
  
    if ( (i<sizeImage) && (j < sizeImage) && (z<blockSize)  ){
    
      cs = 0;
    
      x = xymin + i * dxy;
      y = xymin + j * dxy;
    
      for(k=0; k < (nangles); k++)
      {
        // __sincosf(k * dth, &sink, &cosk);
        __sincosf(angles[k], &sink, &cosk);
        
        if ( k == (nangles - 1) ){
          dth = angles[k] - angles[k-1];
        }else{
          dth = angles[k+1] - angles[k];
        }

        t = x * cosk + y * sink;
      
        T = (int)((t - tmin)/dt);	     
         
        if ( (T > -1) && (T<nrays) )
        {
          cs += blocksino[ z * nrays * nangles + k * nrays + T] * dth;
        }
      }

      voxel = z * sizeImage * sizeImage + j * sizeImage + i;

      value = cs;
      
      image[ voxel ]  = image[ voxel ] * value / backones[ voxel ]; 
    }
  }
}

extern "C" {
  __global__ void kernel_backprojectionOfOnes(float *backones, float *angles,
					      int sizeImage, int nrays, int nangles,  int blockSize)
  {
    int i, j, k, T, z, voxel;
    float t, cs, x, y, cosk, sink;
    float xymin = -1.0;
    float dxy = 2.0 / (sizeImage - 1);
    float dt = 2.0 / (nrays - 1);
    float dth; // = PI / nangles;
    float tmin = -1.0;
    
    i = (blockDim.x * blockIdx.x + threadIdx.x);
    j = (blockDim.y * blockIdx.y + threadIdx.y);
    z = (blockDim.z * blockIdx.z + threadIdx.z);
  
    if ( (i<sizeImage) && (j < sizeImage) && (z<blockSize)  ){
    
      cs = 0;
    
      x = xymin + i * dxy;
      y = xymin + j * dxy;
    
      for(k=0; k < (nangles); k++)
      {
        // __sincosf(k * dth, &sink, &cosk);
        __sincosf(angles[k], &sink, &cosk);
        
        if ( k == (nangles - 1) ){
          dth = angles[k] - angles[k-1];
        }else{
          dth = angles[k+1] - angles[k];
        }

        t = x * cosk + y * sink;
      
        T = (int) ((t - tmin)/dt);	     

        if ( (T > -1) && (T<nrays) )
          {
            cs += dth; //blocksino[ z * nrays * nangles+ k * nrays + T];
          }
      }
      
      voxel = z * sizeImage * sizeImage + j * sizeImage + i;
      
      backones[ voxel ]  = cs; 
    }
  }
}

extern "C" {

  void eEM(float *output, float *sino, float *angles, 
	   int sizeImage, int nrays, int nangles, int blockSize, int device, int niter)
  {
    hipSetDevice(device);
    int k;
    float *d_output, *d_sino, *d_backones, *d_temp, *d_ones, *d_angles;
    
    // Allocate GPU memory for the output image
    hipMalloc(&d_output, sizeof(float) *sizeImage *sizeImage*blockSize);
    hipMalloc(&d_temp, sizeof(float)  * nrays * nangles * blockSize);
    hipMalloc(&d_backones, sizeof(float) * sizeImage * sizeImage*blockSize);
    hipMalloc(&d_ones, sizeof(float) * nrays * nangles * blockSize);
    
    // Allocate GPU memory for input image and copy
    hipMalloc(&d_sino, sizeof(float) * nrays * nangles*blockSize);
    hipMemcpy(d_sino, sino, sizeof(float) * nrays * nangles*blockSize, hipMemcpyHostToDevice);	

    hipMalloc(&d_angles, sizeof(float) * nangles);
    hipMemcpy(d_angles, angles, sizeof(float) * nangles, hipMemcpyHostToDevice);	
    
    // printf("Printando essa porra + %d, %d, %e\n",device, nangles, sino[1024]);

    //GRID and BLOCKS SIZE
    dim3 threadsPerBlock(TPBX,TPBY,TPBZ);
    dim3 gridBlockD((int)ceil((nrays)/threadsPerBlock.x)+1,
		   (int)ceil((nangles)/threadsPerBlock.y)+1,
		   (int)ceil(blockSize/threadsPerBlock.z)+1);

    dim3 gridBlockF((int)ceil((sizeImage)/threadsPerBlock.x)+1,
		   (int)ceil((sizeImage)/threadsPerBlock.y)+1,
		   (int)ceil(blockSize/threadsPerBlock.z)+1);

    kernel_ones<<<gridBlockF, threadsPerBlock>>>(d_output, sizeImage, nrays, nangles, blockSize);

    kernel_backprojectionOfOnes<<<gridBlockF, threadsPerBlock>>>(d_backones, d_angles, sizeImage, nrays, nangles, blockSize);
 
    for( k=0; k < niter; k++ ) {

      kernel_radonWithDivision<<<gridBlockD, threadsPerBlock>>>(d_temp, d_output, d_sino, d_angles, sizeImage, nrays, nangles, blockSize, 1.0);
            
      kernel_backprojectionWithUpdate<<<gridBlockF, threadsPerBlock>>>(d_output, d_temp, d_backones, d_angles, sizeImage, nrays, nangles, blockSize);
      
      hipDeviceSynchronize();
    }
    
    //Copy the output image from device memory to host memory
    hipMemcpy (output , d_output , blockSize*sizeImage*sizeImage*sizeof(float) , hipMemcpyDeviceToHost);
    
    hipFree(d_output);
    hipFree(d_temp);
    hipFree(d_sino);
    hipFree(d_backones);
    hipFree(d_angles);
    
    hipDeviceSynchronize();
    // hipDeviceReset();
    
  }
}

//-----------------------------------------
// emission-EM + Total Variation  algorithm
//------------------------------------------

extern "C" {
  // y : next iterate
  // x : current iterate
  
  __global__ void kernel_updateTV(float *y, float *x, float *backones,
				  int sizeImage, int blockSize,
				  float reg, float epsilon)
  {
    int i, j, z, v, vip1, vjp1, vjm1, vim1, vjm1ip1, vjp1im1;
    float sqrtA, sqrtB, sqrtD;
    float A, B, C, D, rhs;
    
    i = (blockDim.x * blockIdx.x + threadIdx.x);
    j = (blockDim.y * blockIdx.y + threadIdx.y);
    z = (blockDim.z * blockIdx.z + threadIdx.z);
  
    if ( ((i+1)<sizeImage) && ((j+1) < sizeImage) && (z<blockSize) &&
	 ((i-1)<sizeImage) && ((j-1) < sizeImage) ){

      // i: column (axis=1 from python)
      // j: row    (axis=0 from python)

      v    = z * sizeImage * sizeImage + j * sizeImage + i;
      
      vip1 = z * sizeImage * sizeImage + j * sizeImage + (i+1);
      vjp1 = z * sizeImage * sizeImage + (j+1) * sizeImage + i;
      
      sqrtA = epsilon + SQR( y[vjp1] - y[v]) + SQR( y[vip1] - y[v] ); //ok
      A     = - reg * backones[v] * sqrtf(sqrtA);

      vjm1  = z * sizeImage * sizeImage + (j-1) * sizeImage + i;
      vjm1ip1 = z * sizeImage * sizeImage + (j-1) * sizeImage + (i+1);
   
      sqrtB = epsilon * SQR( y[v] - y[vjm1]) + SQR( y[vjm1ip1] - y[vjm1]); //ok
      B     = reg * backones[v] * sqrtf(sqrtB);
      C     = A;

      vjp1im1 =  z * sizeImage * sizeImage + (j+1) * sizeImage + (i-1);
      vjm1    =  z * sizeImage * sizeImage + (j-1) * sizeImage + i;
      vim1    = z * sizeImage * sizeImage + j * sizeImage + (i-1);

      sqrtD = epsilon * SQR( y[vjp1im1] - y[vim1]) + SQR(y[v] - y[vim1]); //ok
      D     = reg * backones[v] * sqrtf(sqrtD); 
      
      rhs = x[v] - y[v] * ( y[vjp1]/A - y[vjm1]/B + y[vip1]/C - y[vim1]/D );
      
      //update!
      float TOLZERO = 1e-6; 
      y[ v ] =  rhs * APPROXINVX( y[v] * ( -1.0/A + 1.0/B - 1.0/C + 1.0/D ) + 1.0, TOLZERO );
    }
    
  }
}

extern "C" {
  // L2-error in the feature domain
  void getError_F(float *error, float *x, float *y, int N, int blockSize, int device)
  {
    hipblasHandle_t handle;
    hipblasStatus_t stat;

    //GRID and BLOCKS SIZE
    dim3 threads(TPBX,TPBY,TPBZ);
    dim3 gridF((int)ceil((N)/threads.x)+1,
	       (int)ceil((N)/threads.y)+1,
	       (int)ceil(blockSize/threads.z)+1);
    
    
    // inplace: x = x - y
    kernel_difference_F<<<gridF, threads>>>(x, y, N, blockSize);

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("ssc-raft: CUBLAS initialization failed\n");
        exit(EXIT_FAILURE);
    }
    
    stat = hipblasSnrm2(handle, N * N * blockSize, x, 1, error);
    if(stat != HIPBLAS_STATUS_SUCCESS){
      printf("ssc-raft: Error code %d, line(%d)\n", stat, __LINE__);
      exit(EXIT_FAILURE);
    }
    
    hipblasDestroy(handle);
  }
  
}

extern "C" {
  void iterEM( float *em,
	       float *sino, float *sinotmp, float *backones, float *angles,
	       int sizeImage, int nrays, int nangles, int blockSize, int device )
  {
    //GRID and BLOCKS SIZE
    dim3 threads(TPBX,TPBY,TPBZ);
    dim3 gridD((int)ceil((nrays)/threads.x)+1,
	       (int)ceil((nangles)/threads.y)+1,
	       (int)ceil(blockSize/threads.z)+1);

    dim3 gridF((int)ceil((sizeImage)/threads.x)+1,
	       (int)ceil((sizeImage)/threads.y)+1,
	       (int)ceil(blockSize/threads.z)+1);
    
    kernel_radonWithDivision<<<gridD, threads>>>(sinotmp, em, sino, angles, sizeImage, nrays, nangles, blockSize, 1.0);	

    kernel_backprojectionWithUpdate<<<gridF, threads>>>(em, sinotmp, backones, angles, sizeImage, nrays, nangles, blockSize);
  }
}

extern "C" {
  void iterTV( float *y, float *x, float *backones,
	       int sizeImage, int blockSize, int device, float reg, float epsilon)
  {
    //GRID and BLOCKS SIZE
    dim3 threads(TPBX,TPBY,TPBZ);
    dim3 gridF((int)ceil((sizeImage)/threads.x)+1,
		    (int)ceil((sizeImage)/threads.y)+1,
		    (int)ceil(blockSize/threads.z)+1);
    
    kernel_updateTV<<<gridF, threads>>>(y, x, backones, sizeImage, blockSize, reg, epsilon);
  }
}

extern "C" {
  void EMTV(float *output, float *sino, float *angles, 
	    int sizeImage, int nrays, int nangles, int blockSize, int device, int niter,
	    int niter_em, int niter_tv, float reg, float epsilon)
  {
    hipSetDevice(device);
 
    int m, k;
    float *d_em, *d_x, *d_y, *d_backones, *d_sino, *d_sinotmp, *d_angles;
    float error, _error_;
    
    // Allocate GPU memory for the output image
    hipMalloc(&d_em, sizeof(float) *sizeImage *sizeImage*blockSize);
    hipMalloc(&d_x,  sizeof(float) *sizeImage *sizeImage*blockSize);
    hipMalloc(&d_y,  sizeof(float) *sizeImage *sizeImage*blockSize);
    hipMalloc(&d_backones, sizeof(float) * sizeImage * sizeImage*blockSize);
    
    hipMalloc(&d_sinotmp, sizeof(float)  * nrays * nangles * blockSize);
    hipMalloc(&d_sino, sizeof(float) * nrays * nangles*blockSize);
    hipMemcpy(d_sino, sino, sizeof(float) * nrays * nangles*blockSize, hipMemcpyHostToDevice);	

    hipMalloc(&d_angles, sizeof(float) * nangles);
    hipMemcpy(d_angles, angles, sizeof(float) * nangles, hipMemcpyHostToDevice);	
    
    //GRID and BLOCKS SIZE
    dim3 threadsPerBlock(TPBX,TPBY,TPBZ);
    dim3 gridBlockD((int)ceil((nrays)/threadsPerBlock.x)+1,
		    (int)ceil((nangles)/threadsPerBlock.y)+1,
		    (int)ceil(blockSize/threadsPerBlock.z)+1);

    dim3 gridBlockF((int)ceil((sizeImage)/threadsPerBlock.x)+1,
		    (int)ceil((sizeImage)/threadsPerBlock.y)+1,
		    (int)ceil(blockSize/threadsPerBlock.z)+1);
    
    
    kernel_ones<<<gridBlockF, threadsPerBlock>>>(d_em, sizeImage, nrays, nangles, blockSize);

    // temp assignment to d_x in order to computer initial error estimate!
    hipMemcpy(d_x, d_em, sizeof(float) * sizeImage * sizeImage * blockSize, hipMemcpyDeviceToDevice);
    
    kernel_backprojectionOfOnes<<<gridBlockF, threadsPerBlock>>>(d_backones, d_angles, sizeImage, nrays, nangles, blockSize);

    iterEM( d_em, d_sino, d_sinotmp, d_backones, d_angles, 
	    sizeImage, nrays, nangles, blockSize, device);

    getError_F(&error, d_x, d_em, sizeImage, blockSize, device);
    
    for (m = 0; m < niter; m++)
      {
	//get d_x pointer
	hipMemcpy(d_x, d_em, sizeof(float) * sizeImage * sizeImage * blockSize, hipMemcpyDeviceToDevice);

	//EM iterations
	for( k = 0; k < niter_em; k++ )
	  {
	    iterEM( d_em, d_sino, d_sinotmp, d_backones, d_angles, 
		    sizeImage, nrays, nangles, blockSize, device);
	    
	    hipDeviceSynchronize();
	  }
	
	//TV iterations
	hipMemcpy(d_y, d_em, sizeof(float) * sizeImage * sizeImage * blockSize, hipMemcpyDeviceToDevice);
	
	for (k = 0; k < niter_tv; k++ )
	  {
	    iterTV( d_y, d_em, d_backones, 
		    sizeImage, blockSize, device, reg, epsilon);

	    hipDeviceSynchronize();
	  }

	hipMemcpy( d_em, d_y, sizeof(float) * sizeImage * sizeImage * blockSize, hipMemcpyDeviceToDevice);
       
	getError_F(&_error_, d_x, d_em, sizeImage, blockSize, device);
       
	fprintf(stdout,"EM+TV: %lf %lf\n", _error_, error);
	if (_error_ < error)
	  error = _error_;
	else
	  break;
	
    }
    
    //Copy the output image from device memory to host memory
    hipMemcpy (output , d_em , blockSize*sizeImage*sizeImage*sizeof(float) , hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_em);
    hipFree(d_backones);
    hipFree(d_sinotmp);
    hipFree(d_sino);
    hipFree(d_angles);
    
    hipDeviceSynchronize();
    // hipDeviceReset();
    
  }
}

//----------------------
// emission-EM Threads Block algorithm
//----------------------

extern "C"{   

  void eEMgpu(float *output, float *sino, float *angles, 
        int sizeImage, int nrays, int nangles, int nslices, int gpu, int niter)
  {

      size_t blocksize = min((size_t)nslices,32ul);

      for(size_t b = 0; b < nslices; b += blocksize){
          
          blocksize = min(size_t(nslices) - b, blocksize);
          // printf("Nslices: %d, blocksize: %ld, Iter: %ld \n", nslices,blocksize,b);


          eEM(output + (size_t)b*sizeImage*sizeImage, sino + (size_t)b*nrays*nangles, angles, sizeImage, nrays, nangles, blocksize, gpu, niter);          
      }

      hipDeviceSynchronize();

  }

  void eEMblock(float *output, float *sino, float *angles, 
    int sizeImage, int nrays, int nangles, int nslices, int ngpus, int niter, int* gpus)
  {
      int t;
      int blockgpu = (nslices + ngpus - 1) / ngpus;
      
      std::vector<std::future<void>> threads;

      for(t = 0; t < ngpus; t++){ 
          
          blockgpu = min(nslices - blockgpu * t, blockgpu);

          threads.push_back(std::async( std::launch::async, eEMgpu, output + (size_t)t * blockgpu * sizeImage*sizeImage, 
          sino + (size_t)t * blockgpu * nrays * nangles, angles, sizeImage, nrays, nangles, blockgpu,  gpus[t], niter
          ));
      }
  
      for(auto& t : threads)
          t.get();
  }

}

//----------------------
// transmission-EM Threads Block algorithm
//----------------------

extern "C"{   

  void tEMgpu(float *output, float *count, float *flat, float *angles, 
        int sizeImage, int nrays, int nangles, int nslices, int gpu, int niter)
  {

      size_t blocksize = min((size_t)nslices,32ul);

      for(size_t b = 0; b < nslices; b += blocksize){
          blocksize = min(size_t(nslices) - b, blocksize);
          // printf("Nslices: %d, blocksize: %ld, Iter: %ld \n", nslices,blocksize,b);

          tEM(output + (size_t)b*sizeImage*sizeImage, count + (size_t)b*nrays*nangles, flat + (size_t)b*nrays*nangles, angles, sizeImage, nrays, nangles, blocksize, gpu, niter);          
      }

      hipDeviceSynchronize();

  }

  void tEMblock(float *output, float *count, float *flat, float *angles, 
    int sizeImage, int nrays, int nangles, int nslices, int ngpus, int niter, int* gpus)
  {
      int t;
      int blockgpu = (nslices + ngpus - 1) / ngpus;
      
      std::vector<std::future<void>> threads;

      for(t = 0; t < ngpus; t++){ 
          
          blockgpu = min(nslices - blockgpu * t, blockgpu);

          threads.push_back(std::async( std::launch::async, tEMgpu, output + (size_t)t * blockgpu * sizeImage*sizeImage, 
          count + (size_t)t * blockgpu * nrays * nangles, flat + (size_t)t * blockgpu * nrays * nangles, angles, 
          sizeImage, nrays, nangles, blockgpu,  gpus[t], niter
          ));
      }
  
      for(auto& t : threads)
          t.get();
  }

}