#include "hip/hip_runtime.h"
// Authors: Gilberto Martinez, Eduardo X Miqueles, Giovanni Baraldi, Paola Ferraz

#include "../../../../inc/sscraft.h"


extern "C"{
    void getFBP(CFG configs, GPU gpus, float *recon, float *tomogram, float *angles, dim3 tomo_size, dim3 tomo_pad, dim3 recon_size)
    {
        int filter_type      = configs.reconstruction_filter_type;
        float paganin_reg    = configs.reconstruction_paganin_reg;
        float regularization = configs.reconstruction_reg;
        int axis_offset      = configs.rotation_axis_offset;

        Filter filter(filter_type, regularization, paganin_reg, axis_offset);
        
        hipfftComplex *filter_kernel;
        HANDLE_ERROR(hipMalloc((void **)&filter_kernel, sizeof(hipfftComplex) * tomo_pad.x ));

        float *sintable, *costable;
        HANDLE_ERROR(hipMalloc((void **)&sintable, configs.tomo.nangles * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void **)&costable, configs.tomo.nangles * sizeof(float)));

        setSinCosTable<<<gpus.Grd.y,gpus.BT.y>>>(sintable, costable, angles, configs.tomo.nangles);

        filterFBP(gpus, filter, tomogram, filter_kernel, tomo_size, tomo_pad);

        BackProjection_RT<<<gpus.Grd,gpus.BT>>>(recon, tomogram, sintable, costable, recon_size, tomo_size);

        hipDeviceSynchronize();
        
        hipFree(sintable);
        hipFree(costable);
        hipFree(filter_kernel);
    }

    void getFBP_thresh(CFG configs, GPU gpus, float *recon, float *tomogram, float *angles, dim3 tomo_size, dim3 tomo_pad, dim3 recon_size)
    {
        int filter_type      = configs.reconstruction_filter_type;
        float paganin_reg    = configs.reconstruction_paganin_reg;
        float regularization = configs.reconstruction_reg;
        int axis_offset      = configs.rotation_axis_offset;

        Filter filter(filter_type, regularization, paganin_reg, axis_offset);
        EType datatype = EType((EType::TypeEnum)configs.datatype);

        hipfftComplex *filter_kernel;
        HANDLE_ERROR(hipMalloc((void **)&filter_kernel, sizeof(hipfftComplex) * tomo_pad.x ));

        float *sintable, *costable;
        HANDLE_ERROR(hipMalloc((void **)&sintable, configs.tomo.nangles * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void **)&costable, configs.tomo.nangles * sizeof(float)));

        setSinCosTable<<<gpus.Grd.y,gpus.BT.y>>>(sintable, costable, angles, configs.tomo.nangles);

        filterFBP(gpus, filter, tomogram, filter_kernel, tomo_size, tomo_pad);

        BackProjection_RT_thresh<<<gpus.Grd,gpus.BT>>>(recon, tomogram, sintable, costable, recon_size, tomo_size, configs.threshold, datatype.type);

        hipDeviceSynchronize();

        hipFree(sintable);
        hipFree(costable);
        hipFree(filter_kernel);
    }

    __global__ void setSinCosTable(float *sintable, float *costable, float *angles, int nangles)
    {
        size_t k = blockIdx.x*blockDim.x + threadIdx.x;

        if ( (k >= nangles) ) return;

        sintable[k] = __asinf(angles[k]);
        costable[k] = __acosf(angles[k]);
    }
}

extern "C"{   

    void getFBPGPU(CFG configs, GPU gpus, float *recon, float *tomogram, float *angles, 
    int sizez, int ngpu)
    {
        hipSetDevice(ngpu);

        int i; 
		int blocksize = min(sizez,32);
        int ind_block = (int)ceil( (float) sizez / blocksize );

        float *dtomo, *dangles, *drecon;

        HANDLE_ERROR(hipMalloc((void **)&dangles, sizeof(float) * configs.tomo.nangles )); 
		HANDLE_ERROR(hipMalloc((void **)&dtomo  , sizeof(float) * (size_t)configs.tomo.size.x * configs.tomo.size.y * blocksize )); 
		HANDLE_ERROR(hipMalloc((void **)&drecon , sizeof(float) * (size_t)configs.recon.size.x * configs.recon.size.y * blocksize )); 

        HANDLE_ERROR(hipMemcpy(dangles, angles, configs.tomo.nangles * sizeof(float), hipMemcpyHostToDevice));

        /* Loop for each batch of size 'batch' in threads */
		int ptr = 0, subblock; size_t ptr_block_tomo = 0, ptr_block_recon = 0;

        for (i = 0; i < ind_block; i++){

			subblock        = min(sizez - ptr, blocksize);

			ptr_block_tomo  = (size_t)configs.tomo.size.x  * configs.tomo.size.y  * ptr;
            ptr_block_recon = (size_t)configs.recon.size.x * configs.recon.size.y * ptr;

			/* Update pointer */
			ptr = ptr + subblock;
			
            HANDLE_ERROR(hipMemcpy(dtomo, tomogram + ptr_block_tomo, (size_t)configs.tomo.size.x * configs.tomo.size.y * subblock * sizeof(float), hipMemcpyHostToDevice));

            getFBP( configs, gpus, drecon, dtomo, dangles, 
                    dim3(configs.tomo.size.x , configs.tomo.size.y , subblock), 
                    dim3(configs.tomo.npad.x , configs.tomo.size.y , subblock),
                    dim3(configs.recon.size.x, configs.recon.size.y, subblock)
                  );
  
            HANDLE_ERROR(hipMemcpy(recon + ptr_block_recon, drecon, (size_t)configs.recon.size.x * configs.recon.size.y * subblock * sizeof(float), hipMemcpyDeviceToHost));                 

        }
        hipDeviceSynchronize();
        hipFree(dangles);
        hipFree(dtomo);
        hipFree(drecon);
    }

    void getFBPMultiGPU(float* recon, float* tomogram, float* angles, float *paramf, int *parami, int* gpus, int ngpus)
    {
        int i, Maxgpudev;
		
		/* Multiples devices */
		hipGetDeviceCount(&Maxgpudev);

		/* If devices input are larger than actual devices on GPU, exit */
		for(i = 0; i < ngpus; i++) 
			assert(gpus[i] < Maxgpudev && "Invalid device number.");

		CFG configs; GPU gpu_parameters;

        setFBPParameters(&configs, paramf, parami);

        setGPUParameters(&gpu_parameters, configs.tomo.npad, ngpus, gpus);

		int subvolume = (tomo.size.z + ngpus - 1) / ngpus;
		int subblock, ptr = 0; 

		if (ngpus == 1){ /* 1 device */

			getFBPGPU(configs, gpu_parameters, recon, tomogram, angles, gpus[0]);

		}else{
		/* Launch async Threads for each device.
			Each device solves a block of 'nrays * nangles' size.
		*/
			// See future c++ async launch
			std::vector<std::future<void>> threads = {};

			for (i = 0; i < ngpus; i++){
				
				subblock   = min(tomo.size.z - ptr, subvolume);


				threads.push_back( std::async( std::launch::async, 
												getFBPGPU, 
												configs, gpu_parameters, 
                                                recon    + (size_t)configs.recon.size.x * configs.recon.size.y * ptr,
												tomogram + (size_t)configs.tomo.size.x  * configs.tomo.size.y  * ptr, 
												angles, 
                                                subblock,
												gpus[i]
												));

                /* Update pointer */
				ptr = ptr + subblock;		

			}
		
			// Log("Synchronizing all threads...\n");
		
			for (i = 0; i < ngpus; i++)
				threads[i].get();
		}	

		hipDeviceSynchronize();
    }

}

extern "C"{

    __global__ void BackProjection_RT(float* recon, const float *tomo, const float* sintable, const float* costable,
    dim3 recon_size, dim3 tomo_size)
    {  
        size_t i = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k = blockIdx.z*blockDim.z + threadIdx.z;
        size_t ind   = recon_size.x * j + i;
        size_t index = recon_size.y * k * recon_size.x + ind;

        float sum = 0, frac; 
        float x, y, t, norm;
        int T, angle;
        
        if ( (i >= recon_size.x) || (j >= recon_size.y) || (k >= recon_size.z) ) return;

        norm  = ( 0.5f * float(M_PI) ) / ( float(tomo_size.y) * float(tomo_size.x) ); 

        x     = - (float)recon_size.x/2.0f + i;
        y     = - (float)recon_size.y/2.0f + j;

        for(angle = 0; angle < (tomo_size.y); angle++){
        
            t = ( x * costable[angle] + y * sintable[angle] + tomo_size.x/2 );
            T = int(t);
        
            if ( ( T >= 0 ) && ( T < ( tomo_size.x - 1 ) ) ){
                frac = t-T;

                sum += sino[tomo_size.y * tomo_size.x * k + angle * tomo_size.x + T] * (1.0f - frac) + sino[angle * tomo_size.x + T + 1] * frac;
            }
        }        

        recon[index] = sum * norm;

    } 

    __global__ void BackProjection_RT_thresh(float* recon, const float *tomo, const float* sintable, const float* costable,
    dim3 recon_size, dim3 tomo_size, float threshold, EType datatype)
    {  
        size_t i   = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j   = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k   = blockIdx.z*blockDim.z + threadIdx.z;
        size_t ind = j + k * recon_size.y;

        float sum = 0, frac; 
        float x, y, t, norm;
        int T, angle;
        
        if ( (i >= recon_size.x) || (j >= recon_size.y) || (k >= recon_size.z) ) return;

        norm  = ( 0.5f * float(M_PI) ) / ( float(tomo_size.y) * float(tomo_size.x) ); 

        x     = - (float)recon_size.x/2.0f + i;
        y     = - (float)recon_size.y/2.0f + j;

        for(angle = 0; angle < (tomo_size.y); angle++){
        
            t = ( x * costable[angle] + y * sintable[angle] + tomo_size.x / 2 );
            T = int(t);
        
            if ( ( T >= 0 ) && ( T < ( tomo_size.x - 1 ) ) ){
                frac = t-T;

                sum += sino[tomo_size.y * tomo_size.x * k + angle * tomo_size.x + T] * (1.0f - frac) + sino[angle * tomo_size.x + T + 1] * frac;
            }
        }        

        /* Normalizes and transforms the recon to 
        its desired range (controled by the threshold variable)
        and to its desired data type (uint8, unint16, and etc...)
        */
        BasicOps::set_pixel(recon, sum*norm, i, ind, recon_size.x, threshold, datatype);
    } 

    __global__ void KRadon_RT(float* restrict frames, const float* image, int nrays, int nangles)
    {
        int ray = (blockDim.x * blockIdx.x + threadIdx.x);
        int ang = (blockDim.y * blockIdx.y + threadIdx.y);
        const size_t sizef = nrays*nrays;
        
        if ( ray>=nrays || ang >= nangles )
            return;
        
        float btheta = float(M_PI)*float(ang + nangles/2)/nangles;
        
        float ct = nrays/2;
        float cos_t = cosf(btheta + 1E-5f);
        float sin_t = sinf(btheta + 1E-5f);

        float x = ct - 2.0f*ct*cos_t + (ray-nrays/2)*sin_t;
        float y = ct - 2.0f*ct*sin_t - (ray-nrays/2)*cos_t;

        float tx0 = -x/cos_t;
        float tx1 = (nrays-x)/cos_t;

        float ty0 = -y/sin_t;
        float ty1 = (nrays-y)/sin_t;

        float d1 = fmaxf(fminf(tx0,tx1), fminf(ty0,ty1));
        int d2 = int(ceil(fminf(fmaxf(tx0,tx1), fmaxf(ty0,ty1)) - d1)+0.1f);

        x += d1*cos_t;
        y += d1*sin_t;
        
        float radon = 0;
        for(int s=0; s<d2; s++)
        {
            radon += image[(int(y+0.5f)*nrays + int(x+0.5f))%sizef + blockIdx.z*sizef];

            x += cos_t;
            y += sin_t;
        }

        frames[blockIdx.z*size_t(nrays*nangles) + nrays*ang + ray] = radon;
    }

}

extern "C"
{
	void GRadon(int device, float* _frames, float* _image, int nrays, int nangles, int blocksize)
	{
		hipSetDevice(device);

		rImage frames(nrays,nangles,blocksize);
		rImage image(_image,nrays,nrays,blocksize);

		KRadon_RT<<<dim3(nrays/64,nangles,blocksize),64>>>(frames.gpuptr, image.gpuptr, nrays, nangles);

		frames.CopyTo(_frames);
		hipDeviceSynchronize();
	}

	void GBackprojection(int device, float* _recon, float* _sino, int nrays, int nangles, int blocksize)
	{
		hipSetDevice(device);

		rImage sintable(nangles,1);
		rImage costable(nangles,1);

		for(int a=0; a<nangles; a++)
		{
			sintable.cpuptr[a] = sinf(float(M_PI)*a/float(nangles));
			costable.cpuptr[a] = cosf(float(M_PI)*a/float(nangles));
		}

		sintable.LoadToGPU();
		costable.LoadToGPU();

		rImage recon(nrays,nrays,blocksize);
		rImage sino(_sino,nrays,nangles,blocksize);

		KBackProjection_RT<<<dim3(nrays/64,nrays,blocksize),64>>>(
			(char*)recon.gpuptr, sino.gpuptr, nrays, nrays, nangles, EType::TypeEnum::FLOAT32, 0, sintable.gpuptr, costable.gpuptr);

		recon.CopyTo(_recon);
		hipDeviceSynchronize();

	}
}