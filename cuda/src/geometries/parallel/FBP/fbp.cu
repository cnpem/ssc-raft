#include "hip/hip_runtime.h"
// Authors: Gilberto Martinez, Eduardo X Miqueles, Giovanni Baraldi, Paola Ferraz

#include "../../../../inc/filters.h"
#include "../../../../inc/reconstructions.h"

extern "C"{
    void getFBP(CFG configs, GPU gpus, float *recon, float *tomogram, float *angles, dim3 tomo_size, dim3 tomo_pad, dim3 recon_size)
    {
        int filter_type      = configs.reconstruction_filter_type;
        float paganin_reg    = configs.reconstruction_paganin_reg;
        float regularization = configs.reconstruction_reg;
        int axis_offset      = configs.rotation_axis_offset;

        Filter filter(filter_type, regularization, paganin_reg, axis_offset);
        
        hipfftComplex *filter_kernel;
        HANDLE_ERROR(hipMalloc((void **)&filter_kernel, sizeof(hipfftComplex) * tomo_pad.x ));

        float *sintable, *costable;
        HANDLE_ERROR(hipMalloc((void **)&sintable, configs.tomo.nangles * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void **)&costable, configs.tomo.nangles * sizeof(float)));

        setSinCosTable<<<gpus.Grd.y,gpus.BT.y>>>(sintable, costable, angles, configs.tomo.nangles);

        filterFBP(gpus, filter, tomogram, filter_kernel, tomo_size, tomo_pad);

        BackProjection_RT<<<gpus.Grd,gpus.BT>>>(recon, tomogram, sintable, costable, recon_size, tomo_size);
        
        hipFree(sintable);
        hipFree(costable);
        hipFree(filter_kernel);
        hipDeviceSynchronize();
    }

    void getFBP_thresh(CFG configs, GPU gpus, float *recon, float *tomogram, float *angles, dim3 tomo_size, dim3 tomo_pad, dim3 recon_size)
    {
        int filter_type      = configs.reconstruction_filter_type;
        float paganin_reg    = configs.reconstruction_paganin_reg;
        float regularization = configs.reconstruction_reg;
        int axis_offset      = configs.rotation_axis_offset;

        Filter filter(filter_type, regularization, paganin_reg, axis_offset);
        EType datatype = EType((EType::TypeEnum)configs.datatype);

        hipfftComplex *filter_kernel;
        HANDLE_ERROR(hipMalloc((void **)&filter_kernel, sizeof(hipfftComplex) * tomo_pad.x ));

        float *sintable, *costable;
        HANDLE_ERROR(hipMalloc((void **)&sintable, configs.tomo.nangles * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void **)&costable, configs.tomo.nangles * sizeof(float)));

        setSinCosTable<<<gpus.Grd.y,gpus.BT.y>>>(sintable, costable, angles, configs.tomo.nangles);

        filterFBP(gpus, filter, tomogram, filter_kernel, tomo_size, tomo_pad);

        BackProjection_RT_thresh<<<gpus.Grd,gpus.BT>>>(recon, tomogram, sintable, costable, recon_size, tomo_size, configs.threshold, datatype.type);

        hipDeviceSynchronize();

        hipFree(sintable);
        hipFree(costable);
        hipFree(filter_kernel);
    }

    __global__ void setSinCosTable(float *sintable, float *costable, float *angles, int nangles)
    {
        size_t k = blockIdx.x*blockDim.x + threadIdx.x;

        if ( (k >= nangles) ) return;

        sintable[k] = asinf(angles[k]);
        costable[k] = acosf(angles[k]);
    }
}

extern "C"{   

    void getFBPGPU(CFG configs, GPU gpus, float *recon, float *tomogram, float *angles, 
    int sizez, int ngpu)
    {
        hipSetDevice(ngpu);

        int i; 
		int blocksize = min(sizez,32);
        int ind_block = (int)ceil( (float) sizez / blocksize );

        float *dtomo, *dangles, *drecon;

        HANDLE_ERROR(hipMalloc((void **)&dangles, sizeof(float) * configs.tomo.nangles )); 
		HANDLE_ERROR(hipMalloc((void **)&dtomo  , sizeof(float) * (size_t)configs.tomo.size.x * configs.tomo.size.y * blocksize )); 
		HANDLE_ERROR(hipMalloc((void **)&drecon , sizeof(float) * (size_t)configs.recon.size.x * configs.recon.size.y * blocksize )); 

        HANDLE_ERROR(hipMemcpy(dangles, angles, configs.tomo.nangles * sizeof(float), hipMemcpyHostToDevice));

        /* Loop for each batch of size 'batch' in threads */
		int ptr = 0, subblock; size_t ptr_block_tomo = 0, ptr_block_recon = 0;

        for (i = 0; i < ind_block; i++){

			subblock        = min(sizez - ptr, blocksize);

			ptr_block_tomo  = (size_t)configs.tomo.size.x  * configs.tomo.size.y  * ptr;
            ptr_block_recon = (size_t)configs.recon.size.x * configs.recon.size.y * ptr;

			/* Update pointer */
			ptr = ptr + subblock;
			
            HANDLE_ERROR(hipMemcpy(dtomo, tomogram + ptr_block_tomo, (size_t)configs.tomo.size.x * configs.tomo.size.y * subblock * sizeof(float), hipMemcpyHostToDevice));

            getFBP( configs, gpus, drecon, dtomo, dangles, 
                    dim3(configs.tomo.size.x , configs.tomo.size.y , subblock), 
                    dim3(configs.tomo.npad.x , configs.tomo.size.y , subblock),
                    dim3(configs.recon.size.x, configs.recon.size.y, subblock)
                  );
  
            HANDLE_ERROR(hipMemcpy(recon + ptr_block_recon, drecon, (size_t)configs.recon.size.x * configs.recon.size.y * subblock * sizeof(float), hipMemcpyDeviceToHost));                 
        }
        hipFree(dangles);
        hipFree(dtomo);
        hipFree(drecon);
        hipDeviceSynchronize();
    }

    void getFBPMultiGPU(float* recon, float* tomogram, float* angles, float *paramf, int *parami, int* gpus, int ngpus)
    {
        int i, Maxgpudev;
		
		/* Multiples devices */
		HANDLE_ERROR(hipGetDeviceCount(&Maxgpudev));

		/* If devices input are larger than actual devices on GPU, exit */
		for(i = 0; i < ngpus; i++) 
			assert(gpus[i] < Maxgpudev && "Invalid device number.");

		CFG configs; GPU gpu_parameters;

        setFBPParameters(&configs, paramf, parami);

        setGPUParameters(&gpu_parameters, configs.tomo.npad, ngpus, gpus);

		int subvolume = (configs.tomo.size.z + ngpus - 1) / ngpus;
		int subblock, ptr = 0; 

		if (ngpus == 1){ /* 1 device */

			getFBPGPU(configs, gpu_parameters, recon, tomogram, angles, subvolume, gpus[0]);

		}else{
		/* Launch async Threads for each device.
			Each device solves a block of 'nrays * nangles' size.
		*/
			// See future c++ async launch
			std::vector<std::future<void>> threads = {};

			for (i = 0; i < ngpus; i++){
				
				subblock   = min(configs.tomo.size.z - ptr, subvolume);


				threads.push_back( std::async( std::launch::async, 
												getFBPGPU, 
												configs, gpu_parameters, 
                                                recon    + (size_t)configs.recon.size.x * configs.recon.size.y * ptr,
												tomogram + (size_t)configs.tomo.size.x  * configs.tomo.size.y  * ptr, 
												angles, 
                                                subblock,
												gpus[i]
												));

                /* Update pointer */
				ptr = ptr + subblock;		

			}
		
			// Log("Synchronizing all threads...\n");
		
			for (i = 0; i < ngpus; i++)
				threads[i].get();
		}	

		hipDeviceSynchronize();
    }

}

extern "C"{

    __global__ void BackProjection_RT(float* recon, const float *tomo, const float* sintable, const float* costable,
    dim3 recon_size, dim3 tomo_size)
    {  
        size_t i = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k = blockIdx.z*blockDim.z + threadIdx.z;
        size_t ind   = recon_size.x * j + i;
        size_t index = recon_size.y * k * recon_size.x + ind;

        float sum = 0, frac; 
        float x, y, t, norm;
        int T, angle;
        
        if ( (i >= recon_size.x) || (j >= recon_size.y) || (k >= recon_size.z) ) return;

        norm  = ( 0.5f * float(M_PI) ) / ( float(tomo_size.y) * float(tomo_size.x) ); 

        x     = - (float)recon_size.x/2.0f + i;
        y     = - (float)recon_size.y/2.0f + j;

        for(angle = 0; angle < (tomo_size.y); angle++){
        
            t = ( x * costable[angle] + y * sintable[angle] + tomo_size.x/2 );
            T = int(t);
        
            if ( ( T >= 0 ) && ( T < ( tomo_size.x - 1 ) ) ){
                frac = t-T;

                sum += tomo[tomo_size.y * tomo_size.x * k + angle * tomo_size.x + T] * (1.0f - frac) + tomo[angle * tomo_size.x + T + 1] * frac;
            }
        }        

        recon[index] = sum * norm;

    } 

    __global__ void BackProjection_RT_thresh(float* recon, const float *tomo, const float* sintable, const float* costable,
    dim3 recon_size, dim3 tomo_size, float threshold, EType datatype)
    {  
        size_t i   = blockIdx.x*blockDim.x + threadIdx.x;
        size_t j   = blockIdx.y*blockDim.y + threadIdx.y;
        size_t k   = blockIdx.z*blockDim.z + threadIdx.z;
        size_t ind = j + k * recon_size.y;

        float sum = 0, frac; 
        float x, y, t, norm;
        int T, angle;
        
        if ( (i >= recon_size.x) || (j >= recon_size.y) || (k >= recon_size.z) ) return;

        norm  = ( 0.5f * float(M_PI) ) / ( float(tomo_size.y) * float(tomo_size.x) ); 

        x     = - (float)recon_size.x/2.0f + i;
        y     = - (float)recon_size.y/2.0f + j;

        for(angle = 0; angle < (tomo_size.y); angle++){
        
            t = ( x * costable[angle] + y * sintable[angle] + tomo_size.x / 2 );
            T = int(t);
        
            if ( ( T >= 0 ) && ( T < ( tomo_size.x - 1 ) ) ){
                frac = t-T;

                sum += tomo[tomo_size.y * tomo_size.x * k + angle * tomo_size.x + T] * (1.0f - frac) + tomo[angle * tomo_size.x + T + 1] * frac;
            }
        }        

        /* Normalizes and transforms the recon to 
        its desired range (controled by the threshold variable)
        and to its desired data type (uint8, unint16, and etc...)
        */
        // BasicOps::set_pixel((void*)recon, sum*norm, (int)i, (int)ind, (int)recon_size.x, threshold, datatype);
    } 

}
