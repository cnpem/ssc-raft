#include "hip/hip_runtime.h"
// Authors: Gilberto Martinez, Eduardo X Miqueles, Giovanni Baraldi, Paola Ferraz

#include "common/opt.hpp"
#include "processing/filters.hpp"
#include "geometries/parallel/fbp.hpp"

extern "C"{
    __global__ void BackProjection_SS(float *object, float *tomogram, 
    float *angles, float *sine, float *cosine, 
    float pixel_size_x, float pixel_size_y,
    dim3 obj_size, dim3 tomo_size)
    {
        int i, j, k;
        float x, y, scale;
        int t_index, angle_index;
        float t, sum;  
        // float cosk, sink;

        int nrays   = tomo_size.x;
        int nangles = tomo_size.y;

        if ( angles[nangles - 1] > float(M_PI) ){
            scale = float(M_PI) / angles[nangles - 1];
        }else{  
            scale = 1.0f;
        }

        // float xmin = -1.0;
        // float ymin = -1.0;
        // float dx   = 2.0 / (obj_size.x - 1);
        // float dy   = 2.0 / (obj_size.y - 1);

        // float tmin = -1.0;
        // float dt   = 2.0 / (nrays - 1);

        float xmin = - pixel_size_x * obj_size.x / 2.0f;
        float ymin = - pixel_size_y * obj_size.y / 2.0f;
        float dx   =   pixel_size_x;
        float dy   =   pixel_size_y;

        float tmin = - pixel_size_x * nrays / 2.0f;
        float dt   = pixel_size_x;
        
        float dangle; // = angles[1] - angles[0];
        
        i = (blockDim.x * blockIdx.x + threadIdx.x);
        j = (blockDim.y * blockIdx.y + threadIdx.y);
        k = (blockDim.z * blockIdx.z + threadIdx.z);
    
        if ( ( i < obj_size.x ) && ( j < obj_size.y ) && ( k < obj_size.z ) ){
        
            sum = 0;
            
            x = xmin + i * dx;
            y = ymin + j * dy;
            
            for(angle_index = 0; angle_index < nangles; angle_index++){

                // __sincosf(angles[angle_index], &sink, &cosk);

                /* Compute angle step size (dangle)*/
                if ( angle_index == (nangles - 1) )

                    dangle = angles[angle_index] - angles[angle_index - 1];

                else

                    dangle = angles[angle_index + 1] - angles[angle_index];
                
                t = x * cosine[angle_index] - y * sine[angle_index]; // here
                
                // t = x * cosk + y * sink;
                
                t_index = (int) ( ( t - tmin ) / dt);	     

                if ( ( t_index > -1 ) && ( t_index < nrays) )
                    sum += tomogram[ k * nrays * nangles  + angle_index * nrays + t_index] * dangle;
                
            }
        
            object[k * obj_size.y * obj_size.x + j * obj_size.x + i]  = sum * scale;
        }
    }

}

extern "C"{
    void getFBP(CFG configs, GPU gpus, 
    float *obj, float *tomogram, float *dataPadded, float *angles, 
    dim3 tomo_size, dim3 tomo_pad, dim3 obj_size)
    {
        int filter_type      = configs.reconstruction_filter_type;
        float paganin_reg    = configs.reconstruction_paganin;
        float regularization = configs.reconstruction_reg;
        float axis_offset    = configs.rotation_axis_offset;
        float pixel_x        = configs.geometry.obj_pixel_x;
        float pixel_y        = configs.geometry.obj_pixel_y;

        int nangles          = configs.tomo.size.y;

        Filter filter(filter_type, paganin_reg, regularization, axis_offset, pixel_x);
        
        float *sintable = opt::allocGPU<float>(nangles);
        float *costable = opt::allocGPU<float>(nangles);

        int grid = (int)ceil( nangles / TPBY ) + 1;
        setSinCosTable<<<grid,TPBY>>>(sintable, costable, angles, nangles);

        dim3 threadsPerBlock(TPBX,TPBY,TPBZ);
        dim3 gridBlock( (int)ceil( tomo_pad.x / TPBX ) + 1,
                        (int)ceil( tomo_pad.y / TPBY ) + 1,
                        (int)ceil( tomo_pad.z / TPBZ ) + 1);

        opt::paddR2R<<<gridBlock,threadsPerBlock>>>(tomogram, dataPadded, tomo_size, configs.tomo.pad, 0.0f);

        /* Filter */
        if (filter.type != Filter::EType::none)
            filterFBP(gpus, filter, dataPadded, tomo_pad);

        /* Backproection */
        // BackProjection_SS<<<gridBlock,threadsPerBlock>>>(obj, dataPadded, angles,
        //                                                 sintable, costable, 
        //                                                 pixel_x, pixel_y,
        //                                                 obj_size, tomo_pad);

        // opt::remove_paddR2R<<<gridBlock,threadsPerBlock>>>(dataPadded, tomogram, size, pad);


        HANDLE_ERROR(hipDeviceSynchronize());
        
        HANDLE_ERROR(hipFree(sintable));
        HANDLE_ERROR(hipFree(costable));    
    }
}

extern "C"{   

    void getFBPGPU(CFG configs, GPU gpus, 
    float *obj, float *tomogram, float *angles, 
    int sizez, int ngpu)
    {
        HANDLE_ERROR(hipSetDevice(ngpu));

        /* Projection data sizes */
        int nrays    = configs.tomo.size.x;
        int nangles  = configs.tomo.size.y;
        int nrayspad = configs.tomo.padsize.x;

        /* Reconstruction sizes */
        int sizeImagex = configs.tomo.padsize.x; //configs.obj.size.x;
        int sizeImagey = configs.tomo.padsize.x; //configs.obj.size.y;

        int i;

        int blocksize = configs.blocksize;

        if ( blocksize == 0 ){
            int blocksize_aux  = compute_GPU_blocksize(sizez, configs.total_required_mem_per_slice_bytes, true, A100_MEM);
            blocksize          = min(sizez, blocksize_aux);
        }

        int ind_block = (int)ceil( (float) sizez / blocksize );

        float *dtomo   = opt::allocGPU<float>((size_t)     nrays *    nangles * blocksize);
        float *dobj    = opt::allocGPU<float>((size_t)sizeImagex * sizeImagey * blocksize);
        float *dangles = opt::allocGPU<float>( nangles );

        /* Padding */
        float *dataPadded = opt::allocGPU<float>((size_t)nrayspad * nangles * blocksize);

        opt::CPUToGPU<float>(angles, dangles, nangles);

        /* Loop for each batch of size 'batch' in threads */
		int ptr = 0, subblock; size_t ptr_block_tomo = 0, ptr_block_obj = 0;

        printf("Size image %d, %d \n", sizeImagex, sizeImagey);
        printf("Size image %d, %d, %d \n", configs.tomo.size.z, configs.tomo.size.y,configs.tomo.size.x);
        printf("Size image %d, %d, %d \n", configs.tomo.padsize.z, configs.tomo.padsize.y,configs.tomo.padsize.x);

        fflush(stdout);

        for (i = 0; i < ind_block; i++){

			subblock       = min(sizez - ptr, blocksize);

			ptr_block_tomo = (size_t)     nrays *    nangles * ptr;
            // ptr_block_obj  = (size_t)sizeImagex * sizeImagey * ptr;
            ptr_block_obj  = (size_t)nrayspad * nangles * ptr;

			/* Update pointer */
			ptr = ptr + subblock;
			
            opt::CPUToGPU<float>(tomogram + ptr_block_tomo, dtomo, 
                                (size_t)nrays * nangles * subblock);

            getFBP( configs, gpus, dobj, dtomo, dataPadded, dangles, 
                    dim3(nrays     ,    nangles, subblock),  /* Tomogram size */
                    dim3(nrayspad  ,    nangles, subblock),  /* Tomogram padded size */
                    dim3(sizeImagex, sizeImagey, subblock)); /* Object (reconstruction) size */

            // opt::GPUToCPU<float>(obj + ptr_block_obj, dobj, 
            //                     (size_t)sizeImagex * sizeImagey * subblock);

            opt::GPUToCPU<float>(obj + ptr_block_obj, dataPadded, 
                                (size_t)nrayspad * nangles * subblock);

        }
        HANDLE_ERROR(hipDeviceSynchronize());

        HANDLE_ERROR(hipFree(dangles));
        HANDLE_ERROR(hipFree(dtomo));
        HANDLE_ERROR(hipFree(dobj));
        HANDLE_ERROR(hipFree(dataPadded));
    }

    void getFBPMultiGPU(int* gpus, int ngpus, 
    float* obj, float* tomogram, float* angles, 
    float *paramf, int *parami)
    {
        int i, Maxgpudev;

		/* Multiples devices */
		HANDLE_ERROR(hipGetDeviceCount(&Maxgpudev));

		/* If devices input are larger than actual devices on GPU, exit */
		for(i = 0; i < ngpus; i++) 
			assert(gpus[i] < Maxgpudev && "Invalid device number.");

		CFG configs; GPU gpu_parameters;

        setFBPParameters(&configs, paramf, parami);
        // printFBPParameters(&configs);

        setGPUParameters(&gpu_parameters, configs.obj.size, ngpus, gpus);

        /* Projection data sizes */
        int nrays    = configs.tomo.size.x;
        int nangles  = configs.tomo.size.y;
        int nslices  = configs.tomo.size.z;

        /* Reconstruction sizes */
        int sizeImagex = configs.tomo.padsize.x; //configs.obj.size.x;
        int sizeImagey = configs.tomo.padsize.x; //configs.obj.size.y;

		int subvolume = (nslices + ngpus - 1) / ngpus;
		int subblock, ptr = 0; 

		if (ngpus == 1){ /* 1 device */

			getFBPGPU(configs, gpu_parameters, obj, tomogram, angles, nslices, gpus[0]);

		}else{
		/* Launch async Threads for each device.
			Each device solves a block of 'nrays * nangles' size.
		*/
			// See future c++ async launch
			std::vector<std::future<void>> threads = {};
            threads.reserve(ngpus);

			for (i = 0; i < ngpus; i++){
				
				subblock   = min(nslices - ptr, subvolume);

				threads.push_back( std::async( std::launch::async, 
                    getFBPGPU, 
                    configs, gpu_parameters, 
                    obj      + (size_t)sizeImagex * sizeImagey * ptr,
                    tomogram + (size_t)     nrays *    nangles * ptr, 
                    angles, 
                    subblock,
                    gpus[i]));

                /* Update pointer */
				ptr = ptr + subblock;		

			}
			for (i = 0; i < ngpus; i++)
				threads[i].get();
		}
    }

}

