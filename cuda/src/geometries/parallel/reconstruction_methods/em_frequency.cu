#include "hip/hip_runtime.h"
#include "common/configs.hpp"
#include "common/complex.hpp"
#include "common/types.hpp"
#include "common/operations.hpp"
#include "common/opt.hpp"
#include "common/logerror.hpp"
#include "geometries/parallel/em.hpp"
#include "geometries/parallel/radon.hpp" /* FST is found here!! */
#include "geometries/parallel/bst.hpp"
#include "common10/cufft_utils.h"

#define NITER_MIN_REG 20 // must be greater than 1 so back_cu and recon_cu are all meaningful.

extern "C" {
void get_tEM_FQ(
float *sino_cu, float *recon_cu, float *angles_cu, float *flat_cu,
float *backcounts_cu, float *back_cu, hipfftComplex *ft_sino_cu, hipfftComplex *ft_recon_cu,
int nrays, int nangles, int blocksize, size_t recon_size, size_t ft_recon_size,
float scale, hipStream_t stream, hipfftHandle plan_2D_forward, hipfftHandle plan_1D_inverse,
cImage& cartesianblock, cImage& polarblock, cImage& realpolar,
hipfftHandle plan1d, hipfftHandle plan2d, int blocksize_bst, 
int zpad, int interpolation, float dx, float tv_param, int niter)
{
    int sizeimage = nrays;  /* Size of reconstruction */
    int pad0      = zpad+1; /* Padding value for FST (zpad) and BST (pad0)*/

    /* Initialize variables with BST */
    EMFQ_BST(backcounts_cu, sino_cu, angles_cu, nrays, nangles, blocksize, nrays, zpad+1);

    /* Compute (1.0/backcounts_cu) */
    calc_reciprocal_element_wise<<<recon_size/NUM_THREADS, NUM_THREADS>>>(backcounts_cu,recon_size);

    // set_value<<<recon_size/NUM_THREADS, NUM_THREADS>>>(recon_cu, 1.0, recon_size);
    // CUDA_RT_CALL(hipPeekAtLastError());
    // CUDA_RT_CALL(hipDeviceSynchronize());

    /* Begin EM Frequency iterations */
    for (int k = 0; k < niter; ++k) {
        
        /* FST call */

        /* Set ft_recon_cu to zero */
        CUDA_RT_CALL(hipMemsetAsync(ft_recon_cu, 0, sizeof(data_type)*ft_recon_size, stream));
        
        /* FST function for counts */
        pst_counts_real(ft_sino_cu, ft_recon_cu, sino_cu, recon_cu,
                        angles_cu, flat_cu,
                        plan_2D_forward, plan_1D_inverse,
                        nrays, nangles, blocksize,
                        zpad, interpolation, scale,
                        stream);

        /* BST call */
        EMFQ_BST_ITER(  back_cu, sino_cu, angles_cu, 
                        cartesianblock, polarblock, realpolar,
                        plan1d, plan2d,
                        nrays, nangles, blocksize, 
                        blocksize_bst, sizeimage, pad0);

        multiply_element_wise<<<recon_size/NUM_THREADS, NUM_THREADS>>>(back_cu, backcounts_cu, recon_size);
        
        CUDA_RT_CALL(hipPeekAtLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());
        
        /* Call for Total variation regularization (TV)
        Fev 2024 - Memory bugs (or something else) persist! 
        
        'tv_param' is the regularization parameter od TV.

        If 'tv_param == 0', then it does not enter the TV computation
        */
        if (0.0 < tv_param && NITER_MIN_REG < k) {
            
            total_variation_2d<<<recon_size/NUM_THREADS, NUM_THREADS>>>(back_cu, recon_cu, backcounts_cu,
                                                                        recon_size, nrays, nrays, blocksize,tv_param);
            
            CUDA_RT_CALL(hipPeekAtLastError());
            CUDA_RT_CALL(hipDeviceSynchronize());
        }
        /* End call for Total variation (TV) */

        multiply_element_wise<<<recon_size/NUM_THREADS, NUM_THREADS>>>(recon_cu, back_cu, recon_size);
        
        CUDA_RT_CALL(hipPeekAtLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());
    }

}

void get_tEM_FQ_GPU(CFG configs,
float *sino, float *recon, float *angles, float *flat,
int blocksize, int gpu)
{
    /*
    Definition of variables:
    blocksize:
        - Value of vertical block to be computed (fraction of nslices)
    */

    CUDA_RT_CALL(hipSetDevice(gpu));

    /* Projection data sizes */
    int nrays     = configs.tomo.size.x;
    int nangles   = configs.tomo.size.y;

    /*zpad:
        - Padding number
        - Integer values: 0, 1, 2, 3
        - total padding = zpad * nrays
    */
    int zpad = configs.tomo.pad.x;  /* Padding value for FST (zpad) */

    /*interpolation:
        - Options: 'bilinear' and 'nearest'
        - 'nearest'  = 0 (see Python function)
        - 'bilinear' = 1 (see Python function)
    */
    int interpolation = configs.interpolation;

    /*dx: 
        - Detector pixel size in [X] units (can be any)
        - My preference is to always use METERS 
        - Be consistent with the units!!
    */
    float dx = configs.geometry.detector_pixel_x;

    /* tv_param:
        - Regularization parameter for total variation (TV) regularization
        - If 'tv_param' =< 0.0, there is no application of TV regularization
    */
    float tv_param = configs.reconstruction_tv;

    /* niter:
        - Number of iterations for EM
    */
    int niter = configs.em_iterations;


    /* Declaration of FST variables */

    /* Cuda kernels parameters */
    hipStream_t stream, stream_H2D, stream_D2H; // 'stream' variable is for cuFFT and kernels.

    /* cuFFT parameters */
    hipfftHandle plan_2D_forward;
    hipfftHandle plan_1D_inverse;
    hipfftComplex *ft_recon_cu = nullptr; // pointer to 2D Fourier transform of recon on GPU.
    hipfftComplex *ft_sino_cu  = nullptr;  // pointer to 1D Fourier transform of sino on GPU.

    /* cuFFT Dimensions */
    std::array<int, FT_PST_RANK_FORWARD> forward_fft_dim = {nrays*(1+zpad), nrays*(1+zpad)};
    std::array<int, FT_PST_RANK_INVERSE> inverse_fft_dim = {nrays*(1+zpad)};

    /* GPU ptr */
    float *recon_cu      = nullptr; // pointer to recon on GPU.
    float *back_cu       = nullptr;  // Auxiliar pointer to iterations backprojection on GPU.
	float *sino_cu       = nullptr;  // pointer to tomogram (sinogram) on GPU.
    float *backcounts_cu = nullptr;  // Auxiliar pointer to iterations backcounts on GPU.
    float *angles_cu     = nullptr;  // pointer to angles list on GPU.
    float *flat_cu       = nullptr;  // pointer to flat on GPU.
    
    /* Datas Dimensions */
    size_t recon_size    = static_cast<size_t>(blocksize * nrays) * nrays;
	size_t sino_size     = static_cast<size_t>(blocksize * nrays) * nangles;
    size_t ft_sino_size  = static_cast<size_t>(blocksize * nangles) * nrays * (1 + zpad);
    size_t ft_recon_size = static_cast<size_t>(blocksize * nrays * (1 + zpad)) * nrays * (1 + zpad);

    float scale = (0 < dx)? dx/(float)inverse_fft_dim[0] : 1.0/(float)inverse_fft_dim[0];

    /* Begin of FST initialization */ 

    ssc_event_start("get_tEM_FQ_GPU()", {
            ssc_param_int("GPU device number", gpu),
            ssc_param_int("nrays", nrays),
            ssc_param_int("nangles", nangles),
            ssc_param_int("Padding", ( 1 + zpad )),
            ssc_param_int("Computed sub blocks", blocksize),
            ssc_param_float("Detector pixel in meters", dx)
    });

    /* cuFFT parameters */
    CUFFT_CALL(hipfftCreate(&plan_2D_forward));
    CUFFT_CALL(hipfftPlanMany(
		&plan_2D_forward, FT_PST_RANK_FORWARD, forward_fft_dim.data(),  // *plan, rank, *n,
		nullptr, 1, forward_fft_dim[0] * forward_fft_dim[1],    		// *inembed, istride, idist,
        nullptr, 1, forward_fft_dim[0] * forward_fft_dim[1],    		// *onembed, ostride, odist,
        HIPFFT_C2C, blocksize));							                // type, batch.

    CUFFT_CALL(hipfftCreate(&plan_1D_inverse));
    CUFFT_CALL(hipfftPlanMany( 
        &plan_1D_inverse, FT_PST_RANK_INVERSE, inverse_fft_dim.data(), 	// *plan, rank, *n,
        nullptr, 1, inverse_fft_dim[0],  			                    // *inembed, istride, idist,
        nullptr, 1, inverse_fft_dim[0],  			                    // *onembed, ostride, odist,
        HIPFFT_C2C, nangles * blocksize));	                            // type, batch. 
    
    /* Cuda kernels parameters - too advanced for me right now */
    CUDA_RT_CALL(hipStreamCreate(&stream_H2D));
    CUDA_RT_CALL(hipStreamCreate(&stream_D2H));
    CUDA_RT_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUFFT_CALL(hipfftSetStream(plan_2D_forward, stream));
    CUFFT_CALL(hipfftSetStream(plan_1D_inverse, stream));

    /* GPU ptr allocation */
    CUDA_RT_CALL(hipHostRegister(recon,
        sizeof(real_data_type) * static_cast<size_t>(blocksize) * nrays * nrays,
        hipHostRegisterDefault));

    CUDA_RT_CALL(hipHostRegister(sino,
        sizeof(real_data_type) * static_cast<size_t>(blocksize) * nangles * nrays,
        hipHostRegisterDefault));

    CUDA_RT_CALL(hipMalloc(&recon_cu     ,sizeof(real_data_type) * recon_size     ));
    CUDA_RT_CALL(hipMalloc(&back_cu      ,sizeof(real_data_type) * recon_size     ));
    CUDA_RT_CALL(hipMalloc(&backcounts_cu,sizeof(real_data_type) * recon_size     ));
    CUDA_RT_CALL(hipMalloc(&sino_cu      ,sizeof(real_data_type) * sino_size      ));
    CUDA_RT_CALL(hipMalloc(&ft_recon_cu  ,sizeof(data_type)      * ft_recon_size  ));
    CUDA_RT_CALL(hipMalloc(&ft_sino_cu   ,sizeof(data_type)      * ft_sino_size   ));
    CUDA_RT_CALL(hipMalloc(&flat_cu      ,sizeof(float)          * nrays*blocksize));
    CUDA_RT_CALL(hipMalloc(&angles_cu    ,sizeof(float)          * nangles        ));
    
    /* Copy data to GPU */
    CUDA_RT_CALL(hipMemcpyAsync(angles_cu,angles,sizeof(float) * nangles        ,hipMemcpyHostToDevice,stream_H2D));
    CUDA_RT_CALL(hipMemcpyAsync(flat_cu  ,flat  ,sizeof(float) * nrays*blocksize,hipMemcpyHostToDevice,stream_H2D));
    CUDA_RT_CALL(hipMemcpyAsync(sino_cu  ,sino  ,sizeof(float) * sino_size      ,hipMemcpyHostToDevice,stream_H2D));
    CUDA_RT_CALL(hipMemcpyAsync(recon_cu ,recon ,sizeof(float) * recon_size     ,hipMemcpyHostToDevice,stream_H2D));
    
    CUDA_RT_CALL(hipStreamSynchronize(stream_H2D));
    /* End of FST initialization */ 

    /* Begin of BST initialization */ 
    int blocksize_bst = 1;
    int sizeimage     = nrays;
    int pad0          = zpad+1;

    /* CImage complex C++ Giovanni struct pointers 
    for BST implementation - See 'inc/commons/types.hpp'
    Change near future!! */
    /* GPU ptr */
    cImage cartesianblock_bst(sizeimage   , sizeimage * blocksize_bst);
    cImage     polarblock_bst(nrays * pad0, nangles   * blocksize_bst);
    cImage      realpolar_bst(nrays * pad0, nangles   * blocksize_bst);

    /* cuFFT parameters */    
    hipfftHandle plan1d_bst;
    hipfftHandle plan2d_bst;

    int dimms1d[] = {(int)nrays*pad0/2};
    int dimms2d[] = {(int)sizeimage,(int)sizeimage};
    int beds[]    = {nrays*pad0/2};

    HANDLE_FFTERROR( hipfftPlanMany(&plan1d_bst, 1, dimms1d, beds, 1, 
        nrays*pad0/2, beds, 1, nrays*pad0/2, HIPFFT_C2C, nangles*blocksize_bst*2) );

    HANDLE_FFTERROR( hipfftPlanMany(&plan2d_bst, 2, dimms2d, nullptr, 
        0, 0, nullptr, 0, 0, HIPFFT_C2C, blocksize_bst) );

    /* End of BST initialization */  

    /* Calls EM Frequency iterations */
    get_tEM_FQ( sino_cu, recon_cu, angles_cu, flat_cu,
                backcounts_cu, back_cu, ft_sino_cu, ft_recon_cu,
                nrays, nangles, blocksize, recon_size, ft_recon_size,
                scale, stream, plan_2D_forward, plan_1D_inverse,
                cartesianblock_bst, polarblock_bst, realpolar_bst,
                plan1d_bst, plan2d_bst, blocksize_bst, 
                zpad, interpolation, dx, tv_param, niter);

    /* Copy computed reconstruction back to CPU */
    CUDA_RT_CALL(hipMemcpyAsync(   recon, recon_cu, sizeof(real_data_type) * recon_size,
                                    hipMemcpyDeviceToHost, stream_D2H));
    
    CUDA_RT_CALL(hipStreamSynchronize(stream_D2H));

    /* Free ptr - Dealocation */    
    CUDA_RT_CALL(hipHostUnregister(recon));
    CUDA_RT_CALL(hipHostUnregister(sino));
    
    /* Free and Destroy FST allocations */
    free_cuda_counts_real(
        angles_cu, flat_cu, recon_cu, sino_cu,
        ft_recon_cu, ft_sino_cu,
        plan_2D_forward, plan_1D_inverse,
        stream, stream_H2D, stream_D2H);

    /* Free FST auxiliar allocations */
    CUDA_RT_CALL(hipFree(back_cu));
    CUDA_RT_CALL(hipFree(backcounts_cu));
    
    /* Destroy BST plans */
    CUFFT_CALL(hipfftDestroy(plan1d_bst));
    CUFFT_CALL(hipfftDestroy(plan2d_bst));
    
    
    // Apagar (só coloquei pra ver em qual crasha):
    // cartesianblock_bst.~cImage();
    // polarblock_bst.~cImage();
    // realpolar_bst.~cImage();
    // Apagar (não deveria ser necessário):
    // CUDA_RT_CALL(hipDeviceReset());

    ssc_event_stop(); /* get_tEM_FQ_GPU() */
}
}

//----------------------
// EM on frequency Threads Block algorithm
//----------------------

extern "C"{

    void _get_tEM_FQ_GPU(CFG configs,
    float *count, float *obj, float *angles, float *flat, 
    int blockgpu, int gpu)
    {
        /* Projection data sizes */
        int nrays     = configs.tomo.size.x;
        int nangles   = configs.tomo.size.y;
        int pad       = configs.tomo.pad.x;

        int blocksize = configs.blocksize;

        if ( blocksize == 0 ){
            int blocksize_aux = calc_blocksize(blockgpu, nangles, nrays, pad, true); 
            blocksize     = min(blockgpu, blocksize_aux);
        }

        /* Indexes and pointers for subBlocks */
        int ind_block = (int)ceil( (float) blockgpu / blocksize );
        int subblock, ptr = 0;

        for (int i = 0; i < ind_block; i++){

            subblock = min(blockgpu - ptr, blocksize);
            // printf("Subblock of get_tEM_FQ_GPU on block %d: %d \n",i,subblock);

            get_tEM_FQ_GPU( configs,
                            count + (size_t)ptr*nrays*nangles, 
                            obj   + (size_t)ptr*nrays*nrays,
                            angles, 
                            flat + (size_t)ptr*nrays,
                            subblock, gpu);

            /* Update pointer */
            ptr = ptr + subblock;
        }
        HANDLE_ERROR(hipDeviceSynchronize());
    }

    void get_tEM_FQ_MultiGPU(int* gpus, int ngpus, 
    float *count, float *obj, float *angles, float *flat,
    float *paramf, int *parami)
    {
        ssc_event_start("get_tEM_FQ_MultiGPU()", {ssc_param_int("ngpus", ngpus)});

        int i, Maxgpudev;
		
		/* Multiples devices */
		HANDLE_ERROR(hipGetDeviceCount(&Maxgpudev));

		/* If devices input are larger than actual devices on GPU, exit */
		for(i = 0; i < ngpus; i++) 
			assert(gpus[i] < Maxgpudev && "Invalid device number.");

        /* General struct found on inc/common/configs.hpp */
        CFG configs;

        /* Found on src/geometries/parallel/reconstruction_methods/parameters.cu */
        setEMFQParameters(&configs, paramf, parami);
        // printEMFQParameters(&configs);

        /* Projection data sizes */
        int nrays    = configs.tomo.size.x;
        int nangles  = configs.tomo.size.y;
        int nslices  = configs.tomo.size.z;
        
        /* Indexes and pointers for GPUs blocks */    
        int t;
        int blockgpu = (nslices + ngpus - 1) / ngpus;
        int subblock, ptr = 0;
        
        std::vector<std::future<void>> threads;
        threads.reserve(ngpus);

        if (ngpus == 1){

            _get_tEM_FQ_GPU(configs, count, obj, angles, flat, nslices, gpus[0]);

        }else{
            for(t = 0; t < ngpus; t++){ 
                
                subblock = min(nslices - ptr, blockgpu);

                threads.push_back(std::async( std::launch::async, _get_tEM_FQ_GPU, 
                    configs,
                    count + (size_t)ptr * nrays * nangles, 
                    obj   + (size_t)ptr * nrays * nrays, 
                    angles, 
                    flat + (size_t)ptr * nrays, 
                    subblock, gpus[t]));

                /* Update pointer */
                ptr = ptr + subblock;
            }

            for(auto& t : threads)
                t.get();
        }
        ssc_event_stop(); /* get_tEM_FQ_MultiGPU */
    }
}

