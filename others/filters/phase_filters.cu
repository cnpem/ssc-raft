#include "hip/hip_runtime.h"
#include "processing/filters.hpp"

extern "C" {

void setPhaseFilterParameters(GEO *geometry, DIM *tomo, 
	float *parameters_float, int *parameters_int)
	{
		/* Set Geometry */
		geometry->geometry = parameters_int[0];
		
		/* Set Tomogram (or detector) variables (h for horizontal (nrays) and v for vertical (nslices)) */
		tomo->size         = dim3(parameters_int[1],parameters_int[2],parameters_int[3]); 

		tomo->x            = parameters_float[0];
		tomo->y            = parameters_float[1];
		tomo->z            = parameters_float[2];
		tomo->dx           = parameters_float[3];
		tomo->dy           = parameters_float[4];
		tomo->dz           = parameters_float[5];

		/* Set Padding */
		tomo->pad          = dim3(parameters_int[4],parameters_int[5],parameters_int[6]); 

		int npadx          = tomo->size.x * ( 1 + 2 * tomo->pad.x ); 
		int npady          = tomo->size.y * ( 1 + 2 * tomo->pad.y ); 
		int npadz          = tomo->size.z * ( 1 + 2 * tomo->pad.z );

		tomo->padsize         = dim3(npadx,npady,npadz); 

		/* Set General reconstruction variables*/
		geometry->energy   = parameters_float[6];

		geometry->lambda   = ( plank * vc          ) / geometry->energy;
		geometry->wave     = ( 2.0   * float(M_PI) ) / geometry->lambda;

		geometry->z1x      = parameters_float[7];
		geometry->z1y      = parameters_float[8];
		geometry->z2x      = parameters_float[9];
		geometry->z2y      = parameters_float[10];

		/* Set magnitude [(z1+z2)/z1] according to the beam geometry */
		switch (geometry->geometry){
			case 0: /* Parallel */	
				geometry->magnitude_x = 1.0;
				geometry->magnitude_y = 1.0;
				break;
			case 1: /* Conebeam */
				geometry->magnitude_x = ( geometry->z1x + geometry->z2x ) / geometry->z1x;
				geometry->magnitude_y = ( geometry->z1y + geometry->z2y ) / geometry->z1y;
				break;
			case 2: /* Fanbeam */		
				geometry->magnitude_x = ( geometry->z1x + geometry->z2x ) / geometry->z1x;
				geometry->magnitude_y = 1.0;
				break;
			default:
				printf("Parallel case as default! \n");
				geometry->magnitude_x = 1.0;
				geometry->magnitude_y = 1.0;
				break;
		}

	}


	void getPhaseFilterMultiGPU(int *gpus, int ngpus, 
    float *projections, float *paramf, int *parami, 
	int phase_type, float phase_reg)
	{	
		int i, Maxgpudev;
		
		/* Multiples devices */
		HANDLE_ERROR(hipGetDeviceCount(&Maxgpudev));

		/* If devices input are larger than actual devices on GPU, exit */
		for(i = 0; i < ngpus; i++) 
			assert(gpus[i] < Maxgpudev && "Invalid device number.");

		GEO geometry; DIM tomo; GPU gpu_parameters;

        setPhaseFilterParameters(&geometry, &tomo, paramf, parami);

        setGPUParameters(&gpu_parameters, tomo.padsize, ngpus, gpus);

		int subvolume = (tomo.size.z + ngpus - 1) / ngpus;
		int subblock, ptr = 0; size_t ptr_volume = 0;

		if (ngpus == 1){ /* 1 device */

			getPhaseFilterGPU(  gpu_parameters, geometry, tomo, 
                                projections, 
                                phase_type, phase_reg, gpus[0]);

		}else{
		/* Launch async Threads for each device.
			Each device solves a block of 'nrays * nangles' size.
		*/
			// See future c++ async launch
			std::vector<std::future<void>> threads = {};

			for (i = 0; i < ngpus; i++){
				
				subblock   = min(tomo.size.z - ptr, subvolume);
				ptr_volume = (size_t)tomo.size.x * tomo.size.y * ptr;

				/* Update pointer */
				ptr = ptr + subblock;
				
				threads.push_back( std::async(  std::launch::async, 
												getPhaseFilterGPU, 
												gpu_parameters, 
                                                geometry, tomo,
												projections + ptr_volume, 
												phase_type, phase_reg, 
												gpus[i]
												));		

			}
		
			// Log("Synchronizing all threads...\n");
		
			for (i = 0; i < ngpus; i++)
				threads[i].get();
		}	

		HANDLE_ERROR(hipDeviceSynchronize());
	}

	void getPhaseFilterGPU(GPU gpus, GEO geometry, DIM tomo,
	float *projections, int phase_type, float phase_reg, int ngpu)
	{	
		/* Initialize GPU device */
		HANDLE_ERROR(hipSetDevice(ngpu));

		int i; 
		int blocksize = min(tomo.size.z,32);
		int ind_block = (int)ceil( (float) tomo.size.z / blocksize );

		float *dprojections; 
		HANDLE_ERROR(hipMalloc((void **)&dprojections, sizeof(float) * (size_t)tomo.size.x * tomo.size.y * blocksize )); 

		/* Loop for each batch of size 'batch' in threads */
		int ptr = 0, subblock; size_t ptr_block = 0;

		for (i = 0; i < ind_block; i++){

			subblock    = min(tomo.size.z - ptr, blocksize);
			ptr_block = (size_t)tomo.size.x * tomo.size.y * ptr;

			/* Update pointer */
			ptr = ptr + subblock;

			getPhaseFilter( gpus, geometry, projections, phase_type, phase_reg,
							 dim3(tomo.size.x, tomo.size.y, subblock), 
							 dim3(tomo.npad.x, tomo.npad.y, subblock)
							 );

			HANDLE_ERROR(hipMemcpy(projections, dprojections + ptr_block, (size_t)tomo.size.x * tomo.size.y * subblock * sizeof(float), hipMemcpyDeviceToHost));

		}
		HANDLE_ERROR(hipDeviceSynchronize());

		HANDLE_ERROR(hipFree(dprojections));

	}

	void getPhaseFilter(GPU gpus, GEO geometry, float *projections, 
	int phase_type, int phase_reg, dim3 size, dim3 size_pad)
	{	
		float *phase_kernel;
		HANDLE_ERROR(hipMalloc((void **)&phase_kernel, sizeof(float) * (size_t)size_pad.x * size_pad.y ));
		
		setPhaseFilterKernel(gpus, geometry, phase_kernel, size_pad, phase_type, phase_reg);

		/* Plan for Fourier transform - cufft */
		int n[] = {(int)size_pad.x,(int)size_pad.x};
		HANDLE_FFTERROR(hipfftPlanMany(&gpus.mplan, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, size.z));

		applyPhaseFilter(gpus, projections, phase_kernel, phase_type, size, size_pad);
	
		// hipDeviceSynchronize();

		/* Destroy plan */
		HANDLE_FFTERROR(hipfftDestroy(gpus.mplan));
		HANDLE_ERROR(hipFree(phase_kernel));
	}

	void applyPhaseFilter(GPU gpus, float *projections, float *kernel, 
	int phase_type, dim3 tomo, dim3 tomo_pad)
	{
		switch (phase_type){
			case 0:
				/* code */
				printf("No filter was selected!");
				break;
			case 1:
				/* code */
				_paganin_gpu(gpus, projections, kernel, tomo, tomo_pad);
				break;
			case 2:
				/* code */
				_bronnikov_gpu(gpus, projections, kernel, tomo, tomo_pad);
				break;
			case 3:
				/* code */
				_born_gpu(gpus, projections, kernel, tomo, tomo_pad);
				break;
			case 4:
				/* code */
				_rytov_gpu(gpus, projections, kernel, tomo, tomo_pad);
				break;

			default:
				_paganin_gpu(gpus, projections, kernel, tomo, tomo_pad);
				break;
		}	

	}

	void setPhaseFilterKernel(GPU gpus, GEO geometry, float *kernel, 
	dim3 size_pad, int phase_type, float phase_reg)
	{	
		hipblasHandle_t handle = NULL;
        hipblasCreate(&handle);
        hipblasStatus_t stat;

		/* Compute phase filter kernel */ 
		switch (phase_type){
				case 0:
					/* code */
					printf("No filter was selected!");
					break;
				case 1:
					/* code */
					paganinKernel<<<gpus.Grd,gpus.BT>>>(geometry, kernel, size_pad, phase_reg);
					break;
				case 2:
					/* code */
					bronnikovKernel<<<gpus.Grd,gpus.BT>>>(geometry, kernel, size_pad, phase_reg);
					break;
				case 3:
					/* code */
					bornKernel<<<gpus.Grd,gpus.BT>>>(geometry, kernel, size_pad, phase_reg);
					break;
				case 4:
					/* code */
					rytovKernel<<<gpus.Grd,gpus.BT>>>(geometry, kernel, size_pad, phase_reg);
					break;
				default:
					printf("Using default Paganin phase filter. \n");
					paganinKernel<<<gpus.Grd,gpus.BT>>>(geometry, kernel, size_pad, phase_reg);
					break;
			}

        /* Normalize kernel by maximum value */ 
 		int max;
        stat = hipblasIsamax(handle, (int)size_pad.x * size_pad.y, kernel, 1, &max);

        if (stat != HIPBLAS_STATUS_SUCCESS)
            printf("Cublas Max failed\n");

		float maximum;
		HANDLE_ERROR(hipMemcpy(&maximum, kernel + max, sizeof(float), hipMemcpyDeviceToHost));
        Normalize<<<gpus.Grd,gpus.BT>>>(kernel, maximum, size_pad);

		hipblasDestroy(handle);
	}

}

